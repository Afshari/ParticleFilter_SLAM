#include "hip/hip_runtime.h"
﻿
#include "headers.h"

// #define CORRELATION_EXEC
#define BRESENHAM_EXEC


#ifdef CORRELATION_EXEC
#include "data/correlation/combined_3423.h"
#endif

#ifdef BRESENHAM_EXEC
#include "data/bresenham/500.h"
#endif




__global__ void kernel_correlation(const int* d_grid_map, const int* d_Y_io_x, const int* d_Y_io_y,
                                    const int* d_Y_io_idx, int* result, int numElements);

__global__ void kernel_bresenham(const int* arr_start_x, const int* arr_start_y, const int end_x, const int end_y,
                                    int* result_array_x, int* result_array_y, const int result_len, const int* index_array);

void host_correlation();
void host_bresenham();


int main() {


#ifdef CORRELATION_EXEC
    host_correlation();
#endif

#ifdef BRESENHAM_EXEC
    host_bresenham();
#endif
    

    return 0;
}


/*
* Host Functions
*/

#ifdef CORRELATION_EXEC
void host_correlation() {

    hipError_t cudaStatus;
    float time_total, time_memory_copy, time_kernel, time_result_copy;
    hipEvent_t start_total, stop_total, stop_memory_copy, start_kernel, stop_kernel, start_result_copy;
    gpuErrchk(hipEventCreate(&start_total));
    gpuErrchk(hipEventCreate(&stop_total));
    gpuErrchk(hipEventCreate(&stop_memory_copy));
    gpuErrchk(hipEventCreate(&start_kernel));
    gpuErrchk(hipEventCreate(&stop_kernel));
    gpuErrchk(hipEventCreate(&start_result_copy));


    const int num_elements_of_grid_map = GRID_WIDTH * GRID_HEIGHT;
    size_t size_of_grid_map = num_elements_of_grid_map * sizeof(int);

    printf("Elements of Grid_Map: %d,  Size of Grid_Map: %d\n", (int)num_elements_of_grid_map, (int)size_of_grid_map);

    const int num_elements_of_Y = Y_LENGTH;
    size_t size_of_Y_x_y = num_elements_of_Y * sizeof(int);
    size_t size_of_Y_idx = num_elements_of_Y * sizeof(int);

    printf("Elements of Y_io_x: %d,  Size of Y_io_x: %d\n", (int)num_elements_of_Y, (int)size_of_Y_x_y);
    printf("Elements of Y_io_y: %d,  Size of Y_io_y: %d\n", (int)num_elements_of_Y, (int)size_of_Y_x_y);
    printf("Elements of Y_io_idx: %d,  Size of Y_io_idx: %d\n", (int)num_elements_of_Y, (int)size_of_Y_idx);


    gpuErrchk(hipEventRecord(start_total, 0));

    int* d_grid_map = NULL;
    int* d_Y_io_x = NULL;
    int* d_Y_io_y = NULL;
    int* d_Y_io_idx = NULL;


    gpuErrchk(hipMalloc((void**)&d_grid_map, size_of_grid_map));
    gpuErrchk(hipMalloc((void**)&d_Y_io_x, size_of_Y_x_y));
    gpuErrchk(hipMalloc((void**)&d_Y_io_y, size_of_Y_x_y));
    gpuErrchk(hipMalloc((void**)&d_Y_io_idx, size_of_Y_idx));


    hipMemcpy(d_grid_map, grid_map, size_of_grid_map, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_io_x, Y_io_x, size_of_Y_x_y, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_io_y, Y_io_y, size_of_Y_x_y, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_io_idx, Y_io_idx, size_of_Y_idx, hipMemcpyHostToDevice);

    const int num_elements_of_particles = 100;
    size_t size_of_result = 25 * num_elements_of_particles * sizeof(int);
    int* result = (int*)malloc(size_of_result);
    memset(result, 0, size_of_result);
    int* d_result = NULL;

    gpuErrchk(hipMalloc((void**)&d_result, size_of_result));
    gpuErrchk(hipMemcpy(d_result, result, size_of_result, hipMemcpyHostToDevice));


    gpuErrchk(hipEventRecord(stop_memory_copy, 0));
    gpuErrchk(hipEventSynchronize(stop_memory_copy));
    gpuErrchk(hipEventElapsedTime(&time_memory_copy, start_total, stop_memory_copy));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_elements_of_Y + threadsPerBlock - 1) / threadsPerBlock;
    // printf("CUDA kernel launch with %d blocks of %d threads, All Threads: %d\n", blocksPerGrid, threadsPerBlock, blocksPerGrid * threadsPerBlock);

    gpuErrchk(hipEventRecord(start_kernel, 0));

    kernel_correlation << <blocksPerGrid, threadsPerBlock >> > (d_grid_map, d_Y_io_x, d_Y_io_y, d_Y_io_idx, d_result, num_elements_of_Y);

    hipDeviceSynchronize();

    gpuErrchk(hipEventRecord(stop_kernel, 0));
    gpuErrchk(hipEventSynchronize(stop_kernel));
    gpuErrchk(hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel));


    gpuErrchk(hipEventRecord(start_result_copy, 0));

    gpuErrchk(hipMemcpy(result, d_result, size_of_result, hipMemcpyDeviceToHost));


    int final_result[num_elements_of_particles] = { 0 };

    for (int i = 0; i < num_elements_of_particles; i++) {
        int curr_max_value = result[i];
        for (int j = 0; j < 25; j++) {
            int curr_value = result[j * 100 + i];
            // printf("curr_value: %d\n", curr_value);
            if (curr_value > curr_max_value) {
                curr_max_value = curr_value;
            }
        }
        final_result[i] = curr_max_value;
    }

    bool all_equal = true;
    for (int i = 0; i < num_elements_of_particles; i++) {
        // printf("index: %d --> %d, %d\n", i, final_result[i], corr[i]);
        if (final_result[i] != corr[i])
            all_equal = false;
    }

    gpuErrchk(hipEventRecord(stop_total, 0));
    gpuErrchk(hipEventSynchronize(stop_total));
    gpuErrchk(hipEventElapsedTime(&time_result_copy, start_result_copy, stop_total));
    gpuErrchk(hipEventElapsedTime(&time_total, start_total, stop_total));


    printf("Memory Copy: %7.3f ms\t Kernel Execution: %7.3f ms\t Result Copy: %7.3f ms\n", time_memory_copy, time_kernel, time_result_copy);
    printf("Total Time of Execution:  %3.1f ms - Python Execution Time: %7.3f ms \n", time_total, EXEC_TIME * 1000);
    printf("All Equal: %s\n", all_equal ? "true" : "false");

    printf("Program Finished\n");

    gpuErrchk(hipFree(d_grid_map));
    gpuErrchk(hipFree(d_Y_io_x));
    gpuErrchk(hipFree(d_Y_io_y));
    gpuErrchk(hipFree(d_Y_io_idx));
    gpuErrchk(hipFree(d_result));
}
#endif


#ifdef BRESENHAM_EXEC
void host_bresenham() {

    float time_total;
    hipEvent_t start_total, stop_total;
    gpuErrchk(hipEventCreate(&start_total));
    gpuErrchk(hipEventCreate(&stop_total));

    size_t size_of_array_start = Y_io_shape * sizeof(int);
    size_t size_of_result = Y_if_shape * sizeof(int);

    int* d_start_x = NULL;
    int* d_start_y = NULL;
    int* d_index_array = NULL;

    int* d_result_array_x = NULL;
    int* d_result_array_y = NULL;

    gpuErrchk(hipMalloc((void**)&d_start_x, size_of_array_start));
    gpuErrchk(hipMalloc((void**)&d_start_y, size_of_array_start));
    gpuErrchk(hipMalloc((void**)&d_index_array, size_of_array_start));

    gpuErrchk(hipMalloc((void**)&d_result_array_x, size_of_result));
    gpuErrchk(hipMalloc((void**)&d_result_array_y, size_of_result));

    int* result_x = (int*)malloc(size_of_result);
    int* result_y = (int*)malloc(size_of_result);
    memset(result_x, 0, size_of_result);
    memset(result_y, 0, size_of_result);


    hipMemcpy(d_start_x, Y_io_x, size_of_array_start, hipMemcpyHostToDevice);
    hipMemcpy(d_start_y, Y_io_y, size_of_array_start, hipMemcpyHostToDevice);
    hipMemcpy(d_index_array, free_idx, size_of_array_start, hipMemcpyHostToDevice);


    hipMemcpy(d_result_array_x, result_x, size_of_result, hipMemcpyHostToDevice);
    hipMemcpy(d_result_array_y, result_y, size_of_result, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (Y_io_shape + threadsPerBlock - 1) / threadsPerBlock;
    // printf("CUDA kernel launch with %d blocks of %d threads, All Threads: %d\n", blocksPerGrid, threadsPerBlock, blocksPerGrid * threadsPerBlock);

    gpuErrchk(hipEventRecord(start_total, 0));

    kernel_bresenham << <blocksPerGrid, threadsPerBlock >> > (d_start_x, d_start_y, p_ib[0], p_ib[1], d_result_array_x, d_result_array_y, Y_io_shape, d_index_array);
    hipDeviceSynchronize();

    gpuErrchk(hipEventRecord(stop_total, 0));
    gpuErrchk(hipEventSynchronize(stop_total));
    gpuErrchk(hipEventElapsedTime(&time_total, start_total, stop_total));

    printf("Total Time of Execution:  %3.1f ms\n", time_total);

    gpuErrchk(hipMemcpy(result_x, d_result_array_x, size_of_result, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(result_y, d_result_array_y, size_of_result, hipMemcpyDeviceToHost));

    bool all_equal = true;
    int errors = 0;
    printf("Start\n");
    for (int i = 0; i < Y_if_shape; i++) {
        if (result_x[i] != free_x[i] || result_y[i] != free_y[i]) {
            all_equal = false;
            errors += 1;
            printf("%d -- %d, %d | %d, %d\n", i, result_x[i], free_x[i], result_y[i], free_y[i]);
        }
    }

    printf("All Equal: %s\n", all_equal ? "true" : "false");
    printf("Errors: %d\n", errors);


    printf("Program Finished\n");

    gpuErrchk(hipFree(d_start_x));
    gpuErrchk(hipFree(d_start_y));
    gpuErrchk(hipFree(d_index_array));
    gpuErrchk(hipFree(d_result_array_x));
    gpuErrchk(hipFree(d_result_array_y));
}
#endif


/*
* Kernel Functions
*/

#ifdef BRESENHAM_EXEC
__global__ void kernel_bresenham(const int* arr_start_x, const int* arr_start_y, const int end_x, const int end_y,
                                    int* result_array_x, int* result_array_y, const int result_len, const int* index_array) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < result_len) {

        int x = arr_start_x[i];
        int y = arr_start_y[i];
        int x1 = x;
        int y1 = y;
        int x2 = end_x;
        int y2 = end_y;

        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);

        int start_index = index_array[i];

        if (dx == 0) {
        
            int sign = (y2 - y1) > 0 ? 1 : -1;
            result_array_x[start_index] = x;
            result_array_y[start_index] = y;

            for (int j = 1; j <= dy; j++) {
                result_array_x[start_index + j] = x;
                result_array_y[start_index + j] = y + sign * j;
            }
        }
        else {

            float gradient = dy / float(dx);
            bool should_reverse = false;

            if (gradient > 1) {

                swap(dx, dy);
                swap(x, y);
                swap(x1, y1);
                swap(x2, y2);
                should_reverse = true;
            }

            int p = 2 * dy - dx;
            if (should_reverse == false) {
                result_array_x[start_index] = x;
                result_array_y[start_index] = y;
            }
            else {
                result_array_x[start_index] = y;
                result_array_y[start_index] = x;
            }

            for (int j = 1; j <= dx; j++) {

                if (p > 0) {
                    y = (y < y2) ? y + 1 : y - 1;
                    p = p + 2 * (dy - dx);
                }
                else {
                    p = p + 2 * dy;
                }

                x = (x < x2) ? x + 1 : x - 1;

                if (should_reverse == false) {
                    result_array_x[start_index + j] = x;
                    result_array_y[start_index + j] = y;
                }
                else {
                    result_array_x[start_index + j] = y;
                    result_array_y[start_index + j] = x;
                }
            }
        }
    }
}
#endif

#ifdef CORRELATION_EXEC
__global__ void kernel_correlation(const int* d_grid_map, const int* d_Y_io_x, const int* d_Y_io_y,
                                    const int* d_Y_io_idx, int* result, int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        int loop_counter = 0;
        for (int x_offset = -2; x_offset <= 2; x_offset++) {

            for (int y_offset = -2; y_offset <= 2; y_offset++) {

                int idx = d_Y_io_idx[i];
                int x = d_Y_io_x[i] + x_offset;
                int y = d_Y_io_y[i] + y_offset;

                if (x >= 0 && y >= 0 && x < GRID_WIDTH && y < GRID_HEIGHT) {

                    int grid_map_idx = x * GRID_HEIGHT + y;
                    int value = d_grid_map[grid_map_idx];

                    if (value != 0)
                        atomicAdd(&result[loop_counter * 100 + idx], value);
                }
                loop_counter++;
            }
        }
    }
}
#endif