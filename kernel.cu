#include "hip/hip_runtime.h"
﻿
#include "headers.h"

// #define CORRELATION_EXEC
// #define BRESENHAM_EXEC
// #define UPDATE_MAP_EXEC
// #define UPDATE_STATE_EXEC
// #define UPDATE_PARTICLE_WEIGHTS_EXEC
// #define RESAMPLING_EXEC



#ifdef CORRELATION_EXEC
#include "data/correlation/combined_3423.h"
#endif

#ifdef BRESENHAM_EXEC
#include "data/bresenham/500.h"
#endif

#ifdef UPDATE_MAP_EXEC
#include "data/log_odds/100.h"
#endif

#ifdef UPDATE_STATE_EXEC
#include "data/state_update/100.h"
#endif

#ifdef UPDATE_PARTICLE_WEIGHTS_EXEC
#include "data/particle_weights/200.h"
#endif

#ifdef RESAMPLING_EXEC
#include "data/resampling/70.h"
#endif

#include "data/update_particles/10.h"


__global__ void kernel_correlation(const int* d_grid_map, const int* d_Y_io_x, const int* d_Y_io_y,
                                    const int* d_Y_io_idx, int* result, const int _GRID_WIDTH, const int _GRID_HEIGHT, int numElements);

__global__ void kernel_bresenham(const int* arr_start_x, const int* arr_start_y, const int end_x, const int end_y,
                                    int* result_array_x, int* result_array_y, const int result_len, const int* index_array);

__global__ void kernel_update_log_odds(float *log_odds, int *f_x, int *f_y, const float _log_t,
                                        const int _GRID_WIDTH, const int _GRID_HEIGHT, const int numElements);

__global__ void kernel_update_map(int* grid_map, const float* log_odds, const float _LOG_ODD_PRIOR, const int _WALL, const int _FREE, const int numElements);


__global__ void kernel_resampling(const float* weights, int* js, const float* rnd, const int numElements);


__global__ void kernel_update_particles(const float* xs, const float* ys, const float* thetas,
                                            float* T_wb, const float* T_bl, float* T_wl, int numElements);

__device__ void kernel_matrix_mul_3x3(const float* A, const float* B, float* C, int start_i);

void host_correlation();
void host_bresenham();
void host_update_map();
void host_update_state();
void host_update_particle_weights();
void host_resampling();



int main() {


#ifdef CORRELATION_EXEC
    host_correlation();
#endif

#ifdef BRESENHAM_EXEC
    host_bresenham();
#endif

#ifdef UPDATE_MAP_EXEC
    host_update_map();
#endif

#ifdef UPDATE_STATE_EXEC
    host_update_state();
#endif

#ifdef __cplusplus 
    printf("C++\n");
#endif

#ifdef UPDATE_PARTICLE_WEIGHTS_EXEC
    host_update_particle_weights();
#endif

#ifdef RESAMPLING_EXEC
    host_resampling();
#endif

    // [✓] - Create a kernel with Input (xs, ys, thetas, numElements)
    // [✓] - Add Output to kernel(p_wb, R_wb) --> len of parameters (2x, 4x)
    // [✓] - Create device variables (d_xs, d_ys, d_thetas) & (d_p_wb, d_R_wb) & (res_p_wb, res_R_wb)
    // [✓] - Copy data from (xs, ys, thetas) -> (d_xs, d_ys, d_thetas)
    // [✓] - Initialize to zero (res_p_wb, res_R_wb) & (d_p_wb, d_R_wb)
    // [✓] - Change (p_wb, R_wb) to (T_wb)
    // [ ] - Add 'T_bl' to Input Parameters
    // [✓] - Create a custom MatrixMultiplication function
    // [ ] - Add 'lidar_coords' to function Input parameters
    // [ ] - Add Send length of 'lidar_coords' to the function
    // [ ] - Calculate Length of Result


    const int STATES_LEN = NUM_PARTICLES;

    size_t size_of_states = STATES_LEN * sizeof(float);

    float* d_xs       = NULL;
    float* d_ys       = NULL;
    float* d_thetas   = NULL;

    gpuErrchk(hipMalloc((void**)&d_xs,     size_of_states));
    gpuErrchk(hipMalloc((void**)&d_ys,     size_of_states));
    gpuErrchk(hipMalloc((void**)&d_thetas, size_of_states));

    hipMemcpy(d_xs, xs, size_of_states, hipMemcpyHostToDevice);
    hipMemcpy(d_ys, ys, size_of_states, hipMemcpyHostToDevice);
    hipMemcpy(d_thetas, thetas, size_of_states, hipMemcpyHostToDevice);


    //size_t size_of_p_wb = 2 * STATES_LEN * sizeof(float);
    //size_t size_of_R_wb = 4 * STATES_LEN * sizeof(float);
    //float* d_p_wb = NULL;
    //float* d_R_wb = NULL;
    //gpuErrchk(hipMalloc((void**)&d_p_wb, size_of_p_wb));
    //gpuErrchk(hipMalloc((void**)&d_R_wb, size_of_R_wb));

    size_t size_of_T_wb = 9 * STATES_LEN * sizeof(float);
    size_t size_of_T_bl = 9 * sizeof(float);
    size_t size_of_T_wl = 9 * NUM_PARTICLES * sizeof(float);
    size_t size_of_lidar_coords = 2 * lidar_coords_LEN * sizeof(float);
    float* d_T_wb = NULL;
    float* d_T_bl = NULL;
    float* d_T_wl = NULL;
    float* d_lidar_coords = NULL;
    gpuErrchk(hipMalloc((void**)&d_T_wb, size_of_T_wb));
    gpuErrchk(hipMalloc((void**)&d_T_bl, size_of_T_bl));
    gpuErrchk(hipMalloc((void**)&d_T_wl, size_of_T_wl));
    gpuErrchk(hipMalloc((void**)&d_lidar_coords, size_of_lidar_coords));

    //float* res_p_wb = (float*)malloc(size_of_p_wb);
    //float* res_R_wb = (float*)malloc(size_of_R_wb);
    //memset(res_p_wb, 0, size_of_p_wb);
    //memset(res_R_wb, 0, size_of_R_wb);
    //hipMemcpy(d_p_wb, res_p_wb, size_of_p_wb, hipMemcpyHostToDevice);
    //hipMemcpy(d_R_wb, res_R_wb, size_of_R_wb, hipMemcpyHostToDevice);

    float* res_T_wb = (float*)malloc(size_of_T_wb);
    float* res_T_wl = (float*)malloc(size_of_T_wl);
    memset(res_T_wb, 0, size_of_T_wb);
    memset(res_T_wl, 0, size_of_T_wl);
    hipMemcpy(d_T_wb, res_T_wb, size_of_T_wb, hipMemcpyHostToDevice);
    hipMemcpy(d_T_wl, res_T_wl, size_of_T_wl, hipMemcpyHostToDevice);
    hipMemcpy(d_T_bl, T_bl, size_of_T_bl, hipMemcpyHostToDevice);
    hipMemcpy(d_lidar_coords, lidar_coords, size_of_lidar_coords, hipMemcpyHostToDevice);

    int threadsPerBlock = NUM_PARTICLES;
    int blocksPerGrid = 1;

    // const float* xs, const float* ys, const float* thetas, float* T_wb, const float* T_bl, float* T_wl, int numElements
    kernel_update_particles << <blocksPerGrid, threadsPerBlock >> > (d_xs, d_ys, d_thetas, d_T_wb, d_T_bl, d_T_wl, NUM_PARTICLES);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(res_T_wb, d_T_wb, size_of_T_wb, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_T_wl, d_T_wl, size_of_T_wl, hipMemcpyDeviceToHost));

    for (int i = 0; i < 9 * NUM_PARTICLES; i++) {
        // printf("%f, %f | ", res_T_wb[i], T_wb[i]);
        assert( abs(res_T_wb[i] - T_wb[i]) < 1e-5 );
    }
    for (int i = 0; i < 9 * NUM_PARTICLES; i++) {
        printf("%f, %f |  ", res_T_wl[i], T_wl[i]);
        assert(abs(res_T_wl[i] - T_wl[i]) < 1e-5);
    }
    //printf("\n\n");
    //for (int i = 0; i < 4 * NUM_PARTICLES; i++) {
    //    printf("%e, %e  |  ", res_R_wb[i], R_wb[i]);
    //    assert( abs(res_R_wb[i] - R_wb[i]) < 1e-6 );
    //}

    return 0;
}


/*
* Host Functions
*/

#ifdef CORRELATION_EXEC
void host_correlation() {

    hipError_t cudaStatus;
    float time_total, time_memory_copy, time_kernel, time_result_copy;
    hipEvent_t start_total, stop_total, stop_memory_copy, start_kernel, stop_kernel, start_result_copy;
    gpuErrchk(hipEventCreate(&start_total));
    gpuErrchk(hipEventCreate(&stop_total));
    gpuErrchk(hipEventCreate(&stop_memory_copy));
    gpuErrchk(hipEventCreate(&start_kernel));
    gpuErrchk(hipEventCreate(&stop_kernel));
    gpuErrchk(hipEventCreate(&start_result_copy));


    const int num_elements_of_grid_map = GRID_WIDTH * GRID_HEIGHT;
    size_t size_of_grid_map = num_elements_of_grid_map * sizeof(int);

    printf("Elements of Grid_Map: %d,  Size of Grid_Map: %d\n", (int)num_elements_of_grid_map, (int)size_of_grid_map);

    const int num_elements_of_Y = Y_LENGTH;
    size_t size_of_Y_x_y = num_elements_of_Y * sizeof(int);
    size_t size_of_Y_idx = num_elements_of_Y * sizeof(int);

    printf("Elements of Y_io_x: %d,  Size of Y_io_x: %d\n", (int)num_elements_of_Y, (int)size_of_Y_x_y);
    printf("Elements of Y_io_y: %d,  Size of Y_io_y: %d\n", (int)num_elements_of_Y, (int)size_of_Y_x_y);
    printf("Elements of Y_io_idx: %d,  Size of Y_io_idx: %d\n", (int)num_elements_of_Y, (int)size_of_Y_idx);


    gpuErrchk(hipEventRecord(start_total, 0));

    int* d_grid_map = NULL;
    int* d_Y_io_x = NULL;
    int* d_Y_io_y = NULL;
    int* d_Y_io_idx = NULL;


    gpuErrchk(hipMalloc((void**)&d_grid_map, size_of_grid_map));
    gpuErrchk(hipMalloc((void**)&d_Y_io_x, size_of_Y_x_y));
    gpuErrchk(hipMalloc((void**)&d_Y_io_y, size_of_Y_x_y));
    gpuErrchk(hipMalloc((void**)&d_Y_io_idx, size_of_Y_idx));


    hipMemcpy(d_grid_map, grid_map, size_of_grid_map, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_io_x, Y_io_x, size_of_Y_x_y, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_io_y, Y_io_y, size_of_Y_x_y, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_io_idx, Y_io_idx, size_of_Y_idx, hipMemcpyHostToDevice);

    const int num_elements_of_particles = 100;
    size_t size_of_result = 25 * num_elements_of_particles * sizeof(int);
    int* result = (int*)malloc(size_of_result);
    memset(result, 0, size_of_result);
    int* d_result = NULL;

    gpuErrchk(hipMalloc((void**)&d_result, size_of_result));
    gpuErrchk(hipMemcpy(d_result, result, size_of_result, hipMemcpyHostToDevice));


    gpuErrchk(hipEventRecord(stop_memory_copy, 0));
    gpuErrchk(hipEventSynchronize(stop_memory_copy));
    gpuErrchk(hipEventElapsedTime(&time_memory_copy, start_total, stop_memory_copy));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_elements_of_Y + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads, All Threads: %d\n", blocksPerGrid, threadsPerBlock, blocksPerGrid * threadsPerBlock);

    gpuErrchk(hipEventRecord(start_kernel, 0));

    kernel_correlation << <blocksPerGrid, threadsPerBlock >> > (d_grid_map, d_Y_io_x, d_Y_io_y, d_Y_io_idx, d_result, GRID_WIDTH, GRID_HEIGHT, num_elements_of_Y);

    hipDeviceSynchronize();

    gpuErrchk(hipEventRecord(stop_kernel, 0));
    gpuErrchk(hipEventSynchronize(stop_kernel));
    gpuErrchk(hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel));


    gpuErrchk(hipEventRecord(start_result_copy, 0));

    gpuErrchk(hipMemcpy(result, d_result, size_of_result, hipMemcpyDeviceToHost));


    int final_result[num_elements_of_particles] = { 0 };

    for (int i = 0; i < num_elements_of_particles; i++) {
        int curr_max_value = result[i];
        for (int j = 0; j < 25; j++) {
            int curr_value = result[j * 100 + i];
            // printf("curr_value: %d\n", curr_value);
            if (curr_value > curr_max_value) {
                curr_max_value = curr_value;
            }
        }
        final_result[i] = curr_max_value;
    }

    bool all_equal = true;
    for (int i = 0; i < num_elements_of_particles; i++) {
        // printf("index: %d --> %d, %d\n", i, final_result[i], corr[i]);
        if (final_result[i] != corr[i])
            all_equal = false;
    }

    gpuErrchk(hipEventRecord(stop_total, 0));
    gpuErrchk(hipEventSynchronize(stop_total));
    gpuErrchk(hipEventElapsedTime(&time_result_copy, start_result_copy, stop_total));
    gpuErrchk(hipEventElapsedTime(&time_total, start_total, stop_total));


    printf("Memory Copy: %7.3f ms\t Kernel Execution: %7.3f ms\t Result Copy: %7.3f ms\n", time_memory_copy, time_kernel, time_result_copy);
    printf("Total Time of Execution:  %3.1f ms - Python Execution Time: %7.3f ms \n", time_total, EXEC_TIME * 1000);
    printf("All Equal: %s\n", all_equal ? "true" : "false");

    printf("Program Finished\n");

    gpuErrchk(hipFree(d_grid_map));
    gpuErrchk(hipFree(d_Y_io_x));
    gpuErrchk(hipFree(d_Y_io_y));
    gpuErrchk(hipFree(d_Y_io_idx));
    gpuErrchk(hipFree(d_result));
}
#endif

#ifdef BRESENHAM_EXEC
void host_bresenham() {

    float time_total;
    hipEvent_t start_total, stop_total;
    gpuErrchk(hipEventCreate(&start_total));
    gpuErrchk(hipEventCreate(&stop_total));

    size_t size_of_array_start = Y_io_shape * sizeof(int);
    size_t size_of_result = Y_if_shape * sizeof(int);

    int* d_start_x = NULL;
    int* d_start_y = NULL;
    int* d_index_array = NULL;

    int* d_result_array_x = NULL;
    int* d_result_array_y = NULL;

    gpuErrchk(hipMalloc((void**)&d_start_x, size_of_array_start));
    gpuErrchk(hipMalloc((void**)&d_start_y, size_of_array_start));
    gpuErrchk(hipMalloc((void**)&d_index_array, size_of_array_start));

    gpuErrchk(hipMalloc((void**)&d_result_array_x, size_of_result));
    gpuErrchk(hipMalloc((void**)&d_result_array_y, size_of_result));

    int* result_x = (int*)malloc(size_of_result);
    int* result_y = (int*)malloc(size_of_result);
    memset(result_x, 0, size_of_result);
    memset(result_y, 0, size_of_result);


    hipMemcpy(d_start_x, Y_io_x, size_of_array_start, hipMemcpyHostToDevice);
    hipMemcpy(d_start_y, Y_io_y, size_of_array_start, hipMemcpyHostToDevice);
    hipMemcpy(d_index_array, free_idx, size_of_array_start, hipMemcpyHostToDevice);


    hipMemcpy(d_result_array_x, result_x, size_of_result, hipMemcpyHostToDevice);
    hipMemcpy(d_result_array_y, result_y, size_of_result, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (Y_io_shape + threadsPerBlock - 1) / threadsPerBlock;
    // printf("CUDA kernel launch with %d blocks of %d threads, All Threads: %d\n", blocksPerGrid, threadsPerBlock, blocksPerGrid * threadsPerBlock);

    gpuErrchk(hipEventRecord(start_total, 0));

    kernel_bresenham << <blocksPerGrid, threadsPerBlock >> > (d_start_x, d_start_y, p_ib[0], p_ib[1], d_result_array_x, d_result_array_y, Y_io_shape, d_index_array);
    hipDeviceSynchronize();

    gpuErrchk(hipEventRecord(stop_total, 0));
    gpuErrchk(hipEventSynchronize(stop_total));
    gpuErrchk(hipEventElapsedTime(&time_total, start_total, stop_total));

    printf("Total Time of Execution:  %3.1f ms\n", time_total);

    gpuErrchk(hipMemcpy(result_x, d_result_array_x, size_of_result, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(result_y, d_result_array_y, size_of_result, hipMemcpyDeviceToHost));

    bool all_equal = true;
    int errors = 0;
    printf("Start\n");
    for (int i = 0; i < Y_if_shape; i++) {
        if (result_x[i] != free_x[i] || result_y[i] != free_y[i]) {
            all_equal = false;
            errors += 1;
            printf("%d -- %d, %d | %d, %d\n", i, result_x[i], free_x[i], result_y[i], free_y[i]);
        }
    }

    printf("All Equal: %s\n", all_equal ? "true" : "false");
    printf("Errors: %d\n", errors);


    printf("Program Finished\n");

    gpuErrchk(hipFree(d_start_x));
    gpuErrchk(hipFree(d_start_y));
    gpuErrchk(hipFree(d_index_array));
    gpuErrchk(hipFree(d_result_array_x));
    gpuErrchk(hipFree(d_result_array_y));
}
#endif

#ifdef UPDATE_MAP_EXEC

void host_update_map() {

    size_t size_of_io = Y_io_LEN * sizeof(int);
    size_t size_of_if = Y_if_LEN * sizeof(int);
    size_t size_of_log_odds = (GRID_WIDTH * GRID_HEIGHT) * sizeof(float);
    size_t size_of_map = (GRID_WIDTH * GRID_HEIGHT) * sizeof(int);

    float* result_log_odds = (float*)malloc(size_of_log_odds);
    memset(result_log_odds, 0, size_of_log_odds);

    int* result_grid_map = (int*)malloc(size_of_map);

    int* d_Y_io_x = NULL;
    int* d_Y_io_y = NULL;
    int* d_Y_if_x = NULL;
    int* d_Y_if_y = NULL;

    float* d_log_odds = NULL;
    int* d_grid_map = NULL;

    gpuErrchk(hipMalloc((void**)&d_Y_io_x, size_of_io));
    gpuErrchk(hipMalloc((void**)&d_Y_io_y, size_of_io));
    gpuErrchk(hipMalloc((void**)&d_Y_if_x, size_of_if));
    gpuErrchk(hipMalloc((void**)&d_Y_if_y, size_of_if));

    gpuErrchk(hipMalloc((void**)&d_log_odds, size_of_log_odds));
    gpuErrchk(hipMalloc((void**)&d_grid_map, size_of_map));

    hipMemcpy(d_Y_io_x, Y_io_x, size_of_io, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_io_y, Y_io_y, size_of_io, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_if_x, Y_if_x, size_of_if, hipMemcpyHostToDevice);
    hipMemcpy(d_Y_if_y, Y_if_y, size_of_if, hipMemcpyHostToDevice);

    hipMemcpy(d_log_odds, pre_log_odds, size_of_log_odds, hipMemcpyHostToDevice);
    hipMemcpy(d_grid_map, pre_grid_map, size_of_map, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (Y_io_LEN + threadsPerBlock - 1) / threadsPerBlock;

    kernel_update_log_odds << <blocksPerGrid, threadsPerBlock >> > (d_log_odds, d_Y_io_x, d_Y_io_y, 2 * log_t, GRID_WIDTH, GRID_HEIGHT, Y_io_LEN);
    hipDeviceSynchronize();

    threadsPerBlock = 256;
    blocksPerGrid = (Y_if_LEN + threadsPerBlock - 1) / threadsPerBlock;

    kernel_update_log_odds << <blocksPerGrid, threadsPerBlock >> > (d_log_odds, d_Y_if_x, d_Y_if_y, (-1) * log_t, GRID_WIDTH, GRID_HEIGHT, Y_if_LEN);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(result_log_odds, d_log_odds, size_of_log_odds, hipMemcpyDeviceToHost));


    threadsPerBlock = 256;
    blocksPerGrid = ((GRID_WIDTH * GRID_HEIGHT) + threadsPerBlock - 1) / threadsPerBlock;

    // int* grid_map, const float* log_odds, const int _LOG_ODD_PRIOR, const int _WALL, const int _FREE, const int numElements
    kernel_update_map << <blocksPerGrid, threadsPerBlock >> > (d_grid_map, d_log_odds, LOG_ODD_PRIOR, WALL, FREE, GRID_WIDTH * GRID_HEIGHT);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(result_grid_map, d_grid_map, size_of_map, hipMemcpyDeviceToHost));


    int numError = 0;
    int numCorrect = 0;
    for (int i = 0; i < (GRID_WIDTH * GRID_HEIGHT); i++) {

        if (abs(result_log_odds[i] - post_log_odds[i]) > 0.1) {
            printf("%d: %f, %f, %f\n", i, result_log_odds[i], post_log_odds[i], pre_log_odds[i]);
            numError += 1;
        }
        else if (post_log_odds[i] != pre_log_odds[i]) {
            numCorrect += 1;
        }
    }
    printf("Error: %d, Correct: %d\n", numError, numCorrect);



    numError = 0;
    numCorrect = 0;
    for (int i = 0; i < (GRID_WIDTH * GRID_HEIGHT); i++) {

        if (abs(result_grid_map[i] - post_grid_map[i]) > 0.1) {
            printf("%d: %d, %d, %d\n", i, result_grid_map[i], pre_grid_map[i], post_grid_map[i]);
            numError += 1;
        }
        else {
            numCorrect += 1;
        }
    }

    printf("Error: %d, Correct: %d\n", numError, numCorrect);

}

#endif


#ifdef UPDATE_STATE_EXEC
void host_update_state() {

    // [ ] - Create std::map with key:float and value:int --> states
    // [ ] - Create std::vector<float> for xs --> vec_xs
    // [ ] - Iterate over vec_xs:   If it is not in states add to it with value 1
    //                              If it is already in states then increase value by 1
    // [ ] - Change std::map keys to tuple<float, float, float>
    // [ ] - Find max value in std::map

    // [ ] - First create arrays in thrust and then copy them to the std::vec


    int N = 100;

    thrust::device_vector<float> d_xs(xs, xs + N);
    thrust::device_vector<float> d_ys(ys, ys + N);
    thrust::device_vector<float> d_thetas(thetas, thetas + N);

    float time_total;
    hipEvent_t start_total, stop_total;
    gpuErrchk(hipEventCreate(&start_total));
    gpuErrchk(hipEventCreate(&stop_total));

    gpuErrchk(hipEventRecord(start_total, 0));

    thrust::host_vector<float> h_xs(d_xs.begin(), d_xs.end());
    thrust::host_vector<float> h_ys(d_ys.begin(), d_ys.end());
    thrust::host_vector<float> h_thetas(d_thetas.begin(), d_thetas.end());

    std::vector<float> vec_xs(h_xs.begin(), h_xs.end());
    std::vector<float> vec_ys(h_ys.begin(), h_ys.end());
    std::vector<float> vec_thetas(h_thetas.begin(), h_thetas.end());


    std::map<std::tuple<float, float, float>, int> states;

    for (int i = 0; i < N; i++) {

        if (states.find(std::make_tuple(vec_xs[i], vec_ys[i], vec_thetas[i])) == states.end()) {
            states.insert({ std::make_tuple(vec_xs[i], vec_ys[i], vec_thetas[i]), 1 });
        }
        else {
            states[std::make_tuple(vec_xs[i], vec_ys[i], vec_thetas[i])] += 1;
        }
    }

    std::map<std::tuple<float, float, float>, int>::iterator best
        = std::max_element(states.begin(), states.end(), [](const std::pair<std::tuple<float, float, float>, int>& a,
            const std::pair<std::tuple<float, float, float>, int>& b)->bool { return a.second < b.second; });

    auto key = best->first;
    std::cout << std::get<0>(key) << " " << std::get<1>(key) << " " << std::get<2>(key) << " " << best->second << "\n";

    float theta = std::get<2>(key);
    float _T_wb[] = { cos(theta), -sin(theta), std::get<0>(key),
                        sin(theta),  cos(theta), std::get<1>(key),
                        0, 0, 1 };

    gpuErrchk(hipEventRecord(stop_total, 0));
    gpuErrchk(hipEventSynchronize(stop_total));
    gpuErrchk(hipEventElapsedTime(&time_total, start_total, stop_total));

    printf("Total Time of Execution:  %3.1f ms\n", time_total);


    for (int i = 0; i < 9; i++) {
        printf("%f  ", _T_wb[i]);
    }
    printf("\n");

}
#endif

#ifdef UPDATE_PARTICLE_WEIGHTS_EXEC
void host_update_particle_weights() {

    int N = 100;
    thrust::device_vector<double> d_pre_weights(pre_weights, pre_weights + N);

    float time_total;
    hipEvent_t start_total, stop_total;
    gpuErrchk(hipEventCreate(&start_total));
    gpuErrchk(hipEventCreate(&stop_total));

    gpuErrchk(hipEventRecord(start_total, 0));


    thrust::host_vector<double> h_pre_weights(d_pre_weights.begin(), d_pre_weights.end());
    std::vector<double> vec_weights(h_pre_weights.begin(), h_pre_weights.end());
    double max_val = *max_element(vec_weights.begin(), vec_weights.end());

    thrust::for_each(d_pre_weights.begin(), d_pre_weights.end(), _1 -= max_val - 50);
    thrust::transform(d_pre_weights.begin(), d_pre_weights.end(), d_pre_weights.begin(), thrust_exp());

    h_pre_weights.assign(d_pre_weights.begin(), d_pre_weights.end());
    vec_weights.assign(h_pre_weights.begin(), h_pre_weights.end());
    auto sum = std::accumulate(vec_weights.begin(), vec_weights.end(), 0.0, std::plus<double>());

    thrust::transform(d_pre_weights.begin(), d_pre_weights.end(), d_pre_weights.begin(), thrust_div_sum(sum));

    gpuErrchk(hipEventRecord(stop_total, 0));
    gpuErrchk(hipEventSynchronize(stop_total));
    gpuErrchk(hipEventElapsedTime(&time_total, start_total, stop_total));

    printf("Total Time of Execution:  %3.1f ms\n", time_total);

    for (int i = 0; i < N; i++) {
        printf("%.10e ", (double)d_pre_weights[i]);
    }
    printf("\n");

    printf("Max value: %f, %f\n", max_val, sum);
    // printf("Sum: %f\n", sum);
}
#endif

#ifdef RESAMPLING_EXEC
void host_resampling() {

    // [✓] - Create a new kernel 'kernel_resampling'
    // [✓] - Inputs to this kernel are --> (weights, 'j' as output, u)
    // [✓] - Must launch kernel with Grid: 1 & threadPerBlocks: 100
    // [✓] - Each thread has a for-loop. from 0 to 
    // [ ] - Try to Add New Particles with new Resampling

    float time_total;
    hipEvent_t start_total, stop_total;
    gpuErrchk(hipEventCreate(&start_total));
    gpuErrchk(hipEventCreate(&stop_total));

    float* d_weights = NULL;
    int* d_js = NULL;
    float* d_rnd = NULL;

    size_t size_of_weights = NUM_PARTICLES * sizeof(float);
    size_t size_of_js = NUM_PARTICLES * sizeof(int);
    size_t size_of_rnd = NUM_PARTICLES * sizeof(float);

    int js_result[100] = { 0 };

    gpuErrchk(hipMalloc((void**)&d_weights, size_of_weights));
    gpuErrchk(hipMalloc((void**)&d_js, size_of_js));
    gpuErrchk(hipMalloc((void**)&d_rnd, size_of_rnd));

    hipMemcpy(d_weights, weights, size_of_weights, hipMemcpyHostToDevice);
    hipMemcpy(d_js, js_result, size_of_js, hipMemcpyHostToDevice);
    hipMemcpy(d_rnd, rnds, size_of_rnd, hipMemcpyHostToDevice);

    int threadsPerBlock = NUM_PARTICLES;
    int blocksPerGrid = 1;

    gpuErrchk(hipEventRecord(start_total, 0));

    kernel_resampling << <blocksPerGrid, threadsPerBlock >> > (d_weights, d_js, d_rnd, NUM_PARTICLES);
    hipDeviceSynchronize();

    gpuErrchk(hipEventRecord(stop_total, 0));
    gpuErrchk(hipEventSynchronize(stop_total));
    gpuErrchk(hipEventElapsedTime(&time_total, start_total, stop_total));

    printf("Total Time of Execution:  %3.1f ms\n", time_total);

    gpuErrchk(hipMemcpy(js_result, d_js, size_of_js, hipMemcpyDeviceToHost));

    for (int i = 0; i < 100; i++) {
        printf("%d, %d | ", js_result[i], js[i]);
        assert(js_result[i] == js[i]);
    }
    printf("\n");
}
#endif

/*
* Kernel Functions
*/

__global__ void kernel_bresenham(const int* arr_start_x, const int* arr_start_y, const int end_x, const int end_y,
                                    int* result_array_x, int* result_array_y, const int result_len, const int* index_array) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < result_len) {

        int x = arr_start_x[i];
        int y = arr_start_y[i];
        int x1 = x;
        int y1 = y;
        int x2 = end_x;
        int y2 = end_y;

        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);

        int start_index = index_array[i];

        if (dx == 0) {
        
            int sign = (y2 - y1) > 0 ? 1 : -1;
            result_array_x[start_index] = x;
            result_array_y[start_index] = y;

            for (int j = 1; j <= dy; j++) {
                result_array_x[start_index + j] = x;
                result_array_y[start_index + j] = y + sign * j;
            }
        }
        else {

            float gradient = dy / float(dx);
            bool should_reverse = false;

            if (gradient > 1) {

                swap(dx, dy);
                swap(x, y);
                swap(x1, y1);
                swap(x2, y2);
                should_reverse = true;
            }

            int p = 2 * dy - dx;
            if (should_reverse == false) {
                result_array_x[start_index] = x;
                result_array_y[start_index] = y;
            }
            else {
                result_array_x[start_index] = y;
                result_array_y[start_index] = x;
            }

            for (int j = 1; j <= dx; j++) {

                if (p > 0) {
                    y = (y < y2) ? y + 1 : y - 1;
                    p = p + 2 * (dy - dx);
                }
                else {
                    p = p + 2 * dy;
                }

                x = (x < x2) ? x + 1 : x - 1;

                if (should_reverse == false) {
                    result_array_x[start_index + j] = x;
                    result_array_y[start_index + j] = y;
                }
                else {
                    result_array_x[start_index + j] = y;
                    result_array_y[start_index + j] = x;
                }
            }
        }
    }
}


__global__ void kernel_correlation(const int* d_grid_map, const int* d_Y_io_x, const int* d_Y_io_y,
                                    const int* d_Y_io_idx, int* result, const int _GRID_WIDTH, const int _GRID_HEIGHT, int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        int loop_counter = 0;
        for (int x_offset = -2; x_offset <= 2; x_offset++) {

            for (int y_offset = -2; y_offset <= 2; y_offset++) {

                int idx = d_Y_io_idx[i];
                int x = d_Y_io_x[i] + x_offset;
                int y = d_Y_io_y[i] + y_offset;

                if (x >= 0 && y >= 0 && x < _GRID_WIDTH && y < _GRID_HEIGHT) {

                    int grid_map_idx = x * _GRID_HEIGHT + y;
                    int value = d_grid_map[grid_map_idx];

                    if (value != 0)
                        atomicAdd(&result[loop_counter * 100 + idx], value);
                }
                loop_counter++;
            }
        }
    }
}


__global__ void kernel_update_map(int* grid_map, const float* log_odds, const float _LOG_ODD_PRIOR, const int _WALL, const int _FREE, const int numElements) {


    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        if (log_odds[i] > 0)
            grid_map[i] = _WALL;

        if (log_odds[i] < _LOG_ODD_PRIOR)
            grid_map[i] = _FREE;
    }
}


__global__ void kernel_update_log_odds(float* log_odds, int* f_x, int* f_y, const float _log_t,
    const int _GRID_WIDTH, const int _GRID_HEIGHT, const int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        int x = f_x[i];
        int y = f_y[i];

        if (x >= 0 && y >= 0 && x < _GRID_WIDTH && y < _GRID_HEIGHT) {

            int grid_map_idx = x * _GRID_HEIGHT + y;

            log_odds[grid_map_idx] = log_odds[grid_map_idx] + _log_t;
        }
    }
}


__global__ void kernel_resampling(const float* weights, int* js, const float* rnd, const int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        float u = rnd[i] / numElements;
        int j = 0;
        float beta = u + float(i) / numElements;

        float accum = 0;
        for (int idx = 0; idx <= i; idx++) {
            accum += weights[idx];

            while (beta > accum) {
                j += 1;
                accum += weights[j];
            }
        }
        js[i] = j;
    }
}


__global__ void kernel_update_particles(const float* xs, const float* ys, const float* thetas, 
                                        float* T_wb, const float *T_bl, float *T_wl, int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        int T_idx = i * 9;

        float p_wb_0 = xs[i];
        float p_wb_1 = ys[i];

        float R_wb_0 =  cos(thetas[i]);
        float R_wb_1 = -sin(thetas[i]);
        float R_wb_2 =  sin(thetas[i]);
        float R_wb_3 =  cos(thetas[i]);

        T_wb[T_idx + 0] = R_wb_0;   T_wb[T_idx + 1] = R_wb_1;   T_wb[T_idx + 2] = p_wb_0;
        T_wb[T_idx + 3] = R_wb_2;   T_wb[T_idx + 4] = R_wb_3;   T_wb[T_idx + 5] = p_wb_1;
        T_wb[T_idx + 6] = 0;        T_wb[T_idx + 7] = 0;        T_wb[T_idx + 8] = 1;

        kernel_matrix_mul_3x3(T_wb, T_bl, T_wl, T_idx);
    }
}

__device__ void kernel_matrix_mul_3x3(const float* A, const float* B, float* C, int start_i) {

    // A[i, j] --> A[i*3 + j]

    for (int i = 0; i < 3; i++) {

        for (int j = 0; j < 3; j++) {

            float currVal = 0;
            for (int k = 0; k < 3; k++) {
                currVal += A[start_i + (i * 3) + k] * B[k * 3 + j];
            }
            C[start_i + (i * 3) + j] = currVal;
        }
    }
}