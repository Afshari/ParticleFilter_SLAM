#include "hip/hip_runtime.h"
﻿
#include "headers.h"
#include "host_asserts.h"
#include "host_utils.h"
#include "device_utils.cuh"

// ✓



#define BRESENHAM_EXEC
// #define UPDATE_MAP_EXEC
// #define UPDATE_MAP_INIT_EXEC
// #define MAP_FUNC_EXEC

/**************************************************/
// #define UPDATE_STATE_EXEC
// #define UPDATE_PARTICLE_WEIGHTS_EXEC
// #define CORRELATION_EXEC
// #define RESAMPLING_EXEC
// #define UPDATE_PARTICLES_EXEC
// #define UPDATE_UNIQUE_EXEC
// #define UPDATE_LOOP_EXEC
// #define UPDATE_FUNC_EXEC



#ifdef BRESENHAM_EXEC
#include "data/bresenham/1400.h"
#endif

#ifdef UPDATE_MAP_EXEC
#include "data/log_odds/2000.h"
#endif

#ifdef UPDATE_MAP_INIT_EXEC
#include "data/update_map_init/900.h"
#endif

#ifdef MAP_FUNC_EXEC
#include "data/map/100.h"
#endif

/**************************************************/
#ifdef CORRELATION_EXEC
#include "data/map_correlation/4300.h"
#endif

#ifdef UPDATE_STATE_EXEC
#include "data/state_update/300.h"
#endif

#ifdef UPDATE_PARTICLE_WEIGHTS_EXEC
#include "data/particle_weights/1400.h"
#endif

#ifdef RESAMPLING_EXEC
#include "data/resampling/70.h"
#endif

#ifdef UPDATE_PARTICLES_EXEC
#include "data/update_particles/100.h"
// #include "data/update_loop/100.h"
#endif

#ifdef UPDATE_UNIQUE_EXEC
//#include "data/update_unique/200.h"
#include "data/update_unique/4800.h"
#endif

#ifdef UPDATE_LOOP_EXEC
#include "data/update_loop/4500.h"
#endif

#ifdef UPDATE_FUNC_EXEC
#include "data/update_func/4800.h"
#endif

__global__ void kernel_bresenham_rearrange(int* particles_free_x, int* particles_free_y, int* particles_free_x_max, int* particles_free_y_max,
    int* particles_free_idx, const int MAX_DIST_IN_MAP, const int NUM_ELEMS);

__global__ void kernel_bresenham(const int* particles_occupied_x, const int* particles_occupied_y, const int* position_image_body,
    int* particles_free_x, int* particles_free_y, int* particles_free_counter, const int PARTICLES_LEN, const int MAX_DIST_IN_MAP);

__global__ void kernel_bresenham(const int* particles_occupied_x, const int* particles_occupied_y,
    const int* position_image_body, int* particles_free_x, int* particles_free_y, const int* particles_free_idx, const int PARTICLES_LEN);

__global__ void kernel_index_init_const(int* indices, const int value);

__global__ void kernel_index_expansion(const int* idx, int* extended_idx, const int numElements);
__global__ void kernel_correlation_max(const float* correlation_raw, float* correlation, const int _NUM_PARTICLES);
__global__ void kernel_correlation(const int* grid_map, const int* states_x, const int* states_y,
                                    const int* states_idx, float* result, const int _GRID_WIDTH, const int _GRID_HEIGHT, int numElements);


__global__ void kernel_update_log_odds(float *log_odds, int *f_x, int *f_y, const float _log_t,
                                        const int _GRID_WIDTH, const int _GRID_HEIGHT, const int numElements);

__global__ void kernel_update_map(int* grid_map, const float* log_odds, const float _LOG_ODD_PRIOR, const int _WALL, const int _FREE, const int numElements);

__global__ void kernel_resampling(const float* weights, int* js, const float* rnd, const int numElements);

__global__ void kernel_update_particles_states(const float* states_x, const float* states_y, const float* states_theta,
                                                float* transition_body_frame, const float* transition_lidar_frame, float* transition_world_frame, const int numElements);

__global__ void kernel_update_particles_lidar(float* transition_world_frame, int* processed_measure_x, int* processed_measure_y, 
    float* particles_wframe_x, float* particles_wframe_y, const float* _lidar_coords, float _res, int _xmin, int _ymax, const int _LIDAR_COORDS_LEN);
__global__ void kernel_update_particles_lidar(float* transition_world_frame, int* processed_measure_x, int* processed_measure_y, const float* _lidar_coords, float _res, int _xmin, int _ymax,
                                                const int _lidar_coords_LEN, const int numElements);

__device__ void kernel_matrix_mul_3x3(const float* A, const float* B, float* C, int start_i);
__global__ void kernel_matrix_mul_3x3(const float* A, const float* B, float* C);

__global__ void kernel_2d_map_counter(uint8_t* map_2d, int* unique_counter, int* unique_counter_col, const int _GRID_WIDHT, const int _GRID_HEIGHT);

__global__ void kernel_create_2d_map(const int* particles_x, const int* particles_y, const int* particles_idx, const int IDX_LEN, uint8_t* map_2d,
     const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS);

__global__ void kernel_create_2d_map(const int* particles_x, const int* particles_y, const int* particles_idx, const int IDX_LEN, uint8_t* map_2d,
    int* unique_in_particle, int* unique_in_particle_col, const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS);

__global__ void kernel_update_2d_map_with_measure(const int* measure_x, const int* measure_y, const int* measure_idx, const int IDX_LEN, uint8_t* map_2d,
    int* unique_in_particle, int* unique_in_particle_col, const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS);

__global__ void kernel_update_unique_restructure2(uint8_t* map_2d, int* particles_x, int* particles_y, int* particles_idx, int* unique_in_particle_col,
    const int _GRID_WIDTH, const int _GRID_HEIGHT);

__global__ void kernel_update_unique_restructure(uint8_t* map_2d, int* particles_x, int* particles_y, int* particles_idx, 
    int* unique_in_each_particle, int* unique_in_each_particle_col, const int _GRID_WIDTH, const int _GRID_HEIGHT);

__global__ void kernel_position_to_image(int* position_image_body, float* transition_world_lidar, float _res, int _xmin, int _ymax);

__global__ void kernel_rearrange_particles(int* particles_x, int* particles_y, const int* particles_idx,
    const int* c_particles_x, const int* c_particles_y, const int* c_particles_idx, const int* js,
    const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS, const int IDX_LEN, const int C_IDX_LEN);

__global__ void kernel_rearrange_states(float* states_x, float* states_y, float* states_theta,
    float* c_states_x, float* c_states_y, float* c_states_theta, int* js);

__global__ void kernel_rearrange_indecies(int* particles_idx, int* c_particles_idx, int* js, int* last_len, const int ARR_LEN);
__global__ void kernel_arr_increase(int* arr, const int increase_value, const int start_index);
__global__ void kernel_arr_increase(float* arr, const float increase_value, const int start_index);
__global__ void kernel_arr_mult(float* arr, const float mult_value);
__global__ void kernel_arr_mult(float* arr, float* mult_arr);
__global__ void kernel_arr_max(float* arr, float* result, const int LEN);
__global__ void kernel_arr_sum_exp(float* arr, double* result, const int LEN);
__global__ void kernel_arr_normalize(float* arr, const double norm);
__global__ void kernel_update_unique_sum(int* unique_in_particle, const int _NUM_ELEMS);
__global__ void kernel_update_unique_sum_col(int* unique_in_particle_col, const int _GRID_WIDTH);

/**************************************************/
void host_bresenham();
void host_update_map();
void host_update_map_init();
void host_map();

/**************************************************/
void host_correlation();
void host_update_state();
void host_update_particle_weights();
void host_resampling();
void host_update_particles();
void host_update_unique();
void host_update_loop();
void host_update_func();

int main() {


#ifdef BRESENHAM_EXEC
    host_bresenham();
#endif

#ifdef UPDATE_MAP_EXEC
    host_update_map();
#endif

#ifdef UPDATE_MAP_INIT_EXEC
    host_update_map_init();
#endif

#ifdef MAP_FUNC_EXEC
    host_map();
#endif

/**************************************************/
#ifdef CORRELATION_EXEC
    host_correlation();
#endif

#ifdef UPDATE_STATE_EXEC
    host_update_state();
#endif

#ifdef UPDATE_PARTICLE_WEIGHTS_EXEC
    host_update_particle_weights();
#endif

#ifdef RESAMPLING_EXEC
    host_resampling();
#endif

#ifdef UPDATE_PARTICLES_EXEC
    host_update_particles();
#endif

#ifdef UPDATE_UNIQUE_EXEC
    host_update_unique();
#endif

#ifdef UPDATE_LOOP_EXEC
    host_update_loop();
#endif

#ifdef UPDATE_FUNC_EXEC
    host_update_func();
#endif

    return 0;
}


/*
* Host Functions
*/

#ifdef BRESENHAM_EXEC
void host_bresenham() {

    const int data_len = 6;
    int data[data_len] = { 1, 0, 2, 2, 1, 3 };
    int* d_data = NULL;
    gpuErrchk(hipMalloc((void**)&d_data, data_len * sizeof(int)));
    gpuErrchk(hipMemcpy(d_data, data, data_len * sizeof(int), hipMemcpyHostToDevice));

    thrust::exclusive_scan(thrust::host, data, data + 6, data, 0); // in-place scan
    thrust::exclusive_scan(thrust::device, d_data, d_data + 6, d_data, 0); // in-place scan


    printf("width=%d, height=%d\n", GRID_WIDTH, GRID_HEIGHT);
    int MAX_DIST_IN_MAP = sqrt(pow(GRID_WIDTH, 2) + pow(GRID_HEIGHT, 2));
    printf("MAX_DIST_IN_MAP=%d\n", MAX_DIST_IN_MAP);

    /********************************************************************/
    /************************ BRESENHAM VARIABLES ***********************/
    /********************************************************************/
    int PARTICLES_OCCUPIED_LEN = ST_PARTICLES_OCCUPIED_LEN;
    int PARTICLES_FREE_LEN = ST_PARTICLES_FREE_LEN;

    int PARTICLE_UNIQUE_COUNTER = PARTICLES_OCCUPIED_LEN + 1;

    size_t sz_particles_occupied_pos = PARTICLES_OCCUPIED_LEN * sizeof(int);
    size_t sz_particles_free_pos = 0;
    size_t sz_particles_free_pos_max = PARTICLES_OCCUPIED_LEN * MAX_DIST_IN_MAP * sizeof(int);
    size_t sz_particles_free_counter = PARTICLE_UNIQUE_COUNTER * sizeof(int);
    size_t sz_position_image_body = 2 * sizeof(int);
    
    int* d_particles_occupied_x = NULL;
    int* d_particles_occupied_y = NULL;
    int* d_particles_occupied_idx = NULL;
    int* d_particles_free_x = NULL;
    int* d_particles_free_y = NULL;
    int* d_particles_free_x_max = NULL;
    int* d_particles_free_y_max = NULL;
    int* d_particles_free_counter = NULL;
    int* d_particles_free_idx = NULL;
    int* d_position_image_body = NULL;

    gpuErrchk(hipMalloc((void**)&d_particles_occupied_x, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_occupied_y, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_x_max, sz_particles_free_pos_max));
    gpuErrchk(hipMalloc((void**)&d_particles_free_y_max, sz_particles_free_pos_max));
    gpuErrchk(hipMalloc((void**)&d_particles_free_counter, sz_particles_free_counter));
    gpuErrchk(hipMalloc((void**)&d_particles_free_idx, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_position_image_body, sz_position_image_body));

    int* res_particles_free_x = (int*)malloc(sz_particles_free_pos);
    int* res_particles_free_y = (int*)malloc(sz_particles_free_pos);
    int* res_particles_free_counter = (int*)malloc(sz_particles_free_counter);

    gpuErrchk(hipMemcpy(d_particles_occupied_x, h_particles_occupied_x, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_occupied_y, h_particles_occupied_y, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_idx, h_particles_free_idx, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_position_image_body, h_position_image_body, sz_position_image_body, hipMemcpyHostToDevice));


    memset(res_particles_free_x, 0, sz_particles_free_pos);
    memset(res_particles_free_y, 0, sz_particles_free_pos);
    gpuErrchk(hipMemset(d_particles_free_x, 0, sz_particles_free_pos));
    gpuErrchk(hipMemset(d_particles_free_y, 0, sz_particles_free_pos));
    gpuErrchk(hipMemset(d_particles_free_x_max, 0, sz_particles_free_pos_max));
    gpuErrchk(hipMemset(d_particles_free_y_max, 0, sz_particles_free_pos_max));
    gpuErrchk(hipMemset(d_particles_free_counter, 0, sz_particles_free_counter));
    


    /********************************************************************/
    /************************* BRESENHAM KERNEL *************************/
    /********************************************************************/
    auto start_bresenham = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = 256;
    int blocksPerGrid = (PARTICLES_OCCUPIED_LEN + threadsPerBlock - 1) / threadsPerBlock;
    kernel_bresenham << <blocksPerGrid, threadsPerBlock >> > (d_particles_occupied_x, d_particles_occupied_y, d_position_image_body,
        d_particles_free_x_max, d_particles_free_y_max, d_particles_free_counter, PARTICLES_OCCUPIED_LEN, MAX_DIST_IN_MAP);
    hipDeviceSynchronize();
    //kernel_update_unique_sum << <1, 1 >> > (d_particles_free_counter, PARTICLE_UNIQUE_COUNTER);
    //hipDeviceSynchronize();

    thrust::exclusive_scan(thrust::device, d_particles_free_counter, d_particles_free_counter + PARTICLE_UNIQUE_COUNTER, d_particles_free_counter, 0); // in-place scan
    auto stop_bresenham = std::chrono::high_resolution_clock::now();


    auto start_bresenham_rearrange = std::chrono::high_resolution_clock::now();
    gpuErrchk(hipMemcpy(res_particles_free_counter, d_particles_free_counter, sz_particles_free_counter, hipMemcpyDeviceToHost));

    PARTICLES_FREE_LEN = res_particles_free_counter[PARTICLE_UNIQUE_COUNTER - 1];
    sz_particles_free_pos = PARTICLES_FREE_LEN * sizeof(int);
    gpuErrchk(hipMalloc((void**)&d_particles_free_x, sz_particles_free_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_y, sz_particles_free_pos));

    kernel_bresenham_rearrange << <blocksPerGrid, threadsPerBlock >> > (d_particles_free_x, d_particles_free_y, d_particles_free_x_max, d_particles_free_y_max,
        d_particles_free_counter, MAX_DIST_IN_MAP, PARTICLES_OCCUPIED_LEN);
    hipDeviceSynchronize();
    auto stop_bresenham_rearrange = std::chrono::high_resolution_clock::now();


    auto duration_bresenham = std::chrono::duration_cast<std::chrono::microseconds>(stop_bresenham - start_bresenham);
    auto duration_bresenham_rearrange = std::chrono::duration_cast<std::chrono::microseconds>(stop_bresenham_rearrange - start_bresenham_rearrange);
    std::cout << "Time taken by function (Bresenham): " << duration_bresenham.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Bresenham Rearrange): " << duration_bresenham_rearrange.count() << " microseconds" << std::endl;

    res_particles_free_x = (int*)malloc(sz_particles_free_pos);
    res_particles_free_y = (int*)malloc(sz_particles_free_pos);

    gpuErrchk(hipMemcpy(res_particles_free_x, d_particles_free_x, sz_particles_free_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_free_y, d_particles_free_y, sz_particles_free_pos, hipMemcpyDeviceToHost));

    ASSERT_particles_free_index(res_particles_free_counter, h_particles_free_idx, PARTICLES_OCCUPIED_LEN, false);

    ASSERT_particles_free_new_len(PARTICLES_FREE_LEN, ST_PARTICLES_FREE_LEN);

    ASSERT_particles_free(res_particles_free_x, res_particles_free_y, h_particles_free_x, h_particles_free_y, PARTICLES_FREE_LEN);

    printf("Program Finished\n");

    gpuErrchk(hipFree(d_particles_occupied_x));
    gpuErrchk(hipFree(d_particles_occupied_y));
    gpuErrchk(hipFree(d_particles_free_idx));
    gpuErrchk(hipFree(d_particles_free_x));
    gpuErrchk(hipFree(d_particles_free_y));
}
#endif

#ifdef UPDATE_MAP_INIT_EXEC
void host_update_map_init() {

    /********************************************************************/
    /********************* IMAGE TRANSFORM VARIABLES ********************/
    /********************************************************************/
    //size_t sz_lidar_coords = LIDAR_COORDS_LEN * sizeof(float);
    size_t sz_transition_frames = 9 * sizeof(float);
    size_t sz_lidar_coords = 2 * LIDAR_COORDS_LEN * sizeof(float);
    size_t sz_processed_measure_pos = LIDAR_COORDS_LEN * sizeof(int);
    size_t sz_particles_wframe_pos = LIDAR_COORDS_LEN * sizeof(float);
    size_t sz_position_image = 2 * sizeof(int);

    float* d_lidar_coords = NULL;
    float* d_transition_body_lidar = NULL;
    float* d_transition_world_body = NULL;
    float* d_transition_world_lidar = NULL;
    int* d_processed_measure_x = NULL;
    int* d_processed_measure_y = NULL;
    float* d_particles_wframe_x = NULL;
    float* d_particles_wframe_y = NULL;
    int* d_position_image_body = NULL;

    float* res_transition_world_lidar = (float*)malloc(sz_transition_frames);
    int* res_processed_measure_x = (int*)malloc(sz_processed_measure_pos);
    int* res_processed_measure_y = (int*)malloc(sz_processed_measure_pos);
    float* res_particles_wframe_x = (float*)malloc(sz_particles_wframe_pos);
    float* res_particles_wframe_y = (float*)malloc(sz_particles_wframe_pos);
    int* res_position_image_body = (int*)malloc(sz_position_image);
    

    gpuErrchk(hipMalloc((void**)&d_lidar_coords, sz_lidar_coords));
    gpuErrchk(hipMalloc((void**)&d_transition_body_lidar, sz_transition_frames));
    gpuErrchk(hipMalloc((void**)&d_transition_world_body, sz_transition_frames));
    gpuErrchk(hipMalloc((void**)&d_transition_world_lidar, sz_transition_frames));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_x, sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_y, sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_wframe_x, sz_particles_wframe_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_wframe_y, sz_particles_wframe_pos));
    gpuErrchk(hipMalloc((void**)&d_position_image_body, sz_position_image));


    gpuErrchk(hipMemcpy(d_lidar_coords, lidar_coords, sz_lidar_coords, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_transition_body_lidar, h_transition_body_lidar, sz_transition_frames, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_transition_world_body,  h_transition_world_body,  sz_transition_frames, hipMemcpyHostToDevice));


    /********************************************************************/
    /********************* IMAGE TRANSFORM KERNEL ********************/
    /********************************************************************/
    auto start_world_to_image_transform = std::chrono::high_resolution_clock::now();
    kernel_matrix_mul_3x3 << < 1, 1 >> > (d_transition_world_body, d_transition_body_lidar, d_transition_world_lidar);
    hipDeviceSynchronize();

    int threadsPerBlock = 1;
    int blocksPerGrid = LIDAR_COORDS_LEN;
    kernel_update_particles_lidar << < blocksPerGrid, threadsPerBlock >> > (d_transition_world_lidar, d_processed_measure_x, d_processed_measure_y, 
        d_particles_wframe_x, d_particles_wframe_y, d_lidar_coords, res, xmin, ymax, LIDAR_COORDS_LEN);
    hipDeviceSynchronize();

    kernel_position_to_image << < 1, 1 >> > (d_position_image_body, d_transition_world_lidar, res, xmin, ymax);
    hipDeviceSynchronize();

    auto stop_world_to_image_transform = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_transition_world_lidar, d_transition_world_lidar, sz_transition_frames, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_x, d_processed_measure_x, sz_processed_measure_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_y, d_processed_measure_y, sz_processed_measure_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_wframe_x, d_particles_wframe_x, sz_particles_wframe_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_wframe_y, d_particles_wframe_y, sz_particles_wframe_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_position_image_body, d_position_image_body, sz_position_image, hipMemcpyDeviceToHost));

    ASSERT_transition_world_lidar(res_transition_world_lidar, h_transition_world_lidar, 9, false);

    ASSERT_particles_wframe(res_particles_wframe_x, res_particles_wframe_y, h_particles_wframe_x, h_particles_wframe_y, LIDAR_COORDS_LEN, false);

    ASSERT_processed_measurements(res_processed_measure_x, res_processed_measure_y, h_particles_x, h_particles_y, LIDAR_COORDS_LEN);

    ASSERT_position_image_body(res_position_image_body, h_position_image_body);

    auto duration_world_to_image_transform = std::chrono::duration_cast<std::chrono::microseconds>(stop_world_to_image_transform - start_world_to_image_transform);
    std::cout << "Time taken by function (Kernel): " << duration_world_to_image_transform.count() << " microseconds" << std::endl;


}
#endif

#ifdef UPDATE_MAP_EXEC
void host_update_map() {

    // [✓]

    int PARTICLES_OCCUPIED_LEN = ST_PARTICLES_OCCUPIED_LEN;
    int PARTICLES_OCCUPIED_UNIQUE_LEN = 0;
    int PARTICLES_FREE_LEN = ST_PARTICLES_FREE_LEN;
    int PARTICLES_FREE_UNIQUE_LEN = 0;

    /********************************************************************/
    /**************************** MAP VARIABLES *************************/
    /********************************************************************/
    size_t sz_particles_occupied_pos = PARTICLES_OCCUPIED_LEN * sizeof(int);
    size_t sz_particles_free_pos = PARTICLES_FREE_LEN * sizeof(int);
    size_t sz_map = (GRID_WIDTH * GRID_HEIGHT) * sizeof(int);

    int* d_grid_map = NULL;
    int* d_particles_occupied_x = NULL;
    int* d_particles_occupied_y = NULL;
    int* d_particles_free_x = NULL;
    int* d_particles_free_y = NULL;

    gpuErrchk(hipMalloc((void**)&d_grid_map, sz_map));
    gpuErrchk(hipMalloc((void**)&d_particles_occupied_x, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_occupied_y, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_x, sz_particles_free_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_y, sz_particles_free_pos));

    gpuErrchk(hipMemcpy(d_grid_map, pre_grid_map, sz_map, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_occupied_x, h_particles_occupied_x, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_occupied_y, h_particles_occupied_y, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_x, h_particles_free_x, sz_particles_free_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_y, h_particles_free_y, sz_particles_free_pos, hipMemcpyHostToDevice));

    /********************************************************************/
    /************************* LOG-ODDS VARIABLES ***********************/
    /********************************************************************/
    size_t   sz_map_2d = GRID_WIDTH * GRID_HEIGHT * sizeof(uint8_t);
    size_t   sz_unique_counter = 1 * sizeof(int);
    size_t   sz_unique_counter_col = (GRID_WIDTH + 1) * sizeof(int);

    uint8_t* d_map_occupied_2d = NULL;
    uint8_t* d_map_free_2d = NULL;
    int* d_unique_occupied_counter = NULL;
    int* d_unique_free_counter = NULL;
    int* d_unique_occupied_counter_col = NULL;
    int* d_unique_free_counter_col = NULL;

    int* res_unique_occupied_counter = (int*)malloc(sz_unique_counter);
    int* res_unique_free_counter = (int*)malloc(sz_unique_counter);
    int* res_unique_occupied_counter_col = (int*)malloc(sz_unique_counter_col);
    int* res_unique_free_counter_col = (int*)malloc(sz_unique_counter_col);


    gpuErrchk(hipMalloc((void**)&d_map_occupied_2d, sz_map_2d));
    gpuErrchk(hipMalloc((void**)&d_map_free_2d, sz_map_2d));
    gpuErrchk(hipMalloc((void**)&d_unique_occupied_counter, sz_unique_counter));
    gpuErrchk(hipMalloc((void**)&d_unique_occupied_counter_col, sz_unique_counter_col));
    gpuErrchk(hipMalloc((void**)&d_unique_free_counter, sz_unique_counter));
    gpuErrchk(hipMalloc((void**)&d_unique_free_counter_col, sz_unique_counter_col));

    size_t sz_particles_idx = 2 * sizeof(int);

    int h_particles_occupied_idx[] = { 0, PARTICLES_OCCUPIED_LEN };
    int h_particles_free_idx[] = { 0, PARTICLES_FREE_LEN };
    int* d_particles_occupied_idx = NULL;
    int* d_particles_free_idx = NULL;

    gpuErrchk(hipMalloc((void**)&d_particles_occupied_idx, sz_particles_idx));
    gpuErrchk(hipMalloc((void**)&d_particles_free_idx, sz_particles_idx));

    gpuErrchk(hipMemset(d_map_occupied_2d, 0, sz_map_2d));
    gpuErrchk(hipMemset(d_map_free_2d, 0, sz_map_2d));

    gpuErrchk(hipMemcpy(d_particles_occupied_idx, h_particles_occupied_idx, sz_particles_idx, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_idx, h_particles_free_idx, sz_particles_idx, hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(d_unique_occupied_counter_col, 0, sz_unique_counter_col));
    gpuErrchk(hipMemset(d_unique_free_counter_col, 0, sz_unique_counter_col));

    /********************************************************************/
    /**************************** CREATE MAP ****************************/
    /********************************************************************/
    auto start_create_map = std::chrono::high_resolution_clock::now();
    int threadsPerBlock = 256;
    int blocksPerGrid = 1;
    kernel_create_2d_map << <blocksPerGrid, threadsPerBlock >> > (d_particles_occupied_x, d_particles_occupied_y, d_particles_occupied_idx, 
        PARTICLES_OCCUPIED_LEN, d_map_occupied_2d, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES);
    kernel_create_2d_map << <blocksPerGrid, threadsPerBlock >> > (d_particles_free_x, d_particles_free_y, d_particles_free_idx, 
        PARTICLES_FREE_LEN, d_map_free_2d, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES);
    hipDeviceSynchronize();
    auto stop_create_map = std::chrono::high_resolution_clock::now();


    auto start_unique_counter = std::chrono::high_resolution_clock::now();
    threadsPerBlock = GRID_WIDTH;
    blocksPerGrid = 1;
    kernel_2d_map_counter << <blocksPerGrid, threadsPerBlock >> > (d_map_occupied_2d, d_unique_occupied_counter, d_unique_occupied_counter_col, GRID_WIDTH, GRID_HEIGHT);
    kernel_2d_map_counter << <blocksPerGrid, threadsPerBlock >> > (d_map_free_2d, d_unique_free_counter, d_unique_free_counter_col, GRID_WIDTH, GRID_HEIGHT);
    hipDeviceSynchronize();
    auto stop_unique_counter = std::chrono::high_resolution_clock::now();

    auto start_unique_sum = std::chrono::high_resolution_clock::now();
    threadsPerBlock = 1;
    blocksPerGrid = 1;
    kernel_update_unique_sum_col << <blocksPerGrid, threadsPerBlock >> > (d_unique_occupied_counter_col, GRID_WIDTH);
    kernel_update_unique_sum_col << <blocksPerGrid, threadsPerBlock >> > (d_unique_free_counter_col, GRID_WIDTH);
    hipDeviceSynchronize();
    auto stop_unique_sum = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(h_particles_occupied_idx, d_particles_occupied_idx, sz_particles_idx, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_unique_occupied_counter, d_unique_occupied_counter, sz_unique_counter, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_unique_free_counter, d_unique_free_counter, sz_unique_counter, hipMemcpyDeviceToHost));

    //gpuErrchk(hipMemcpy(res_unique_occupied_counter_col, d_unique_occupied_counter_col, sz_unique_counter_col, hipMemcpyDeviceToHost));
    //gpuErrchk(hipMemcpy(res_unique_free_counter_col, d_unique_free_counter_col, sz_unique_counter_col, hipMemcpyDeviceToHost));

    /*---------------------------------------------------------------------*/
    /*-------------------- REINITIALIZE MAP VARIABLES ---------------------*/
    /*---------------------------------------------------------------------*/
    /*---------------------------------------------------------------------*/
    PARTICLES_OCCUPIED_UNIQUE_LEN = res_unique_occupied_counter[0];
    PARTICLES_FREE_UNIQUE_LEN = res_unique_free_counter[0];

    printf("\n--> Occupied Unique: %d, %d\n", PARTICLES_OCCUPIED_UNIQUE_LEN, ST_PARTICLES_OCCUPIED_UNIQUE_LEN);
    assert(PARTICLES_OCCUPIED_UNIQUE_LEN == ST_PARTICLES_OCCUPIED_UNIQUE_LEN);
    printf("\n--> Free Unique: %d, %d\n", PARTICLES_FREE_UNIQUE_LEN, ST_PARTICLES_FREE_UNIQUE_LEN);
    assert(PARTICLES_FREE_UNIQUE_LEN == ST_PARTICLES_FREE_UNIQUE_LEN);


    gpuErrchk(hipFree(d_particles_occupied_x));
    gpuErrchk(hipFree(d_particles_occupied_y));
    gpuErrchk(hipFree(d_particles_free_x));
    gpuErrchk(hipFree(d_particles_free_y));

    sz_particles_occupied_pos = PARTICLES_OCCUPIED_UNIQUE_LEN * sizeof(int);
    sz_particles_free_pos = PARTICLES_FREE_UNIQUE_LEN * sizeof(int);

    int* res_particles_occupied_x = (int*)malloc(sz_particles_occupied_pos);
    int* res_particles_occupied_y = (int*)malloc(sz_particles_occupied_pos);
    int* res_particles_free_x = (int*)malloc(sz_particles_free_pos);
    int* res_particles_free_y = (int*)malloc(sz_particles_free_pos);


    gpuErrchk(hipMalloc((void**)&d_particles_occupied_x, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_occupied_y, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_x, sz_particles_free_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_y, sz_particles_free_pos));

    auto start_restructure_map = std::chrono::high_resolution_clock::now();
    threadsPerBlock = GRID_WIDTH;
    blocksPerGrid = 1;
    kernel_update_unique_restructure2 << <blocksPerGrid, threadsPerBlock >> > (d_map_occupied_2d, d_particles_occupied_x, d_particles_occupied_y, d_particles_occupied_idx, 
        d_unique_occupied_counter_col, GRID_WIDTH, GRID_HEIGHT);
    kernel_update_unique_restructure2 << <blocksPerGrid, threadsPerBlock >> > (d_map_free_2d, d_particles_free_x, d_particles_free_y, d_particles_free_idx,
        d_unique_free_counter_col, GRID_WIDTH, GRID_HEIGHT);
    hipDeviceSynchronize();
    auto stop_restructure_map = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_particles_occupied_x, d_particles_occupied_x, sz_particles_occupied_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_occupied_y, d_particles_occupied_y, sz_particles_occupied_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_free_x, d_particles_free_x, sz_particles_free_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_free_y, d_particles_free_y, sz_particles_free_pos, hipMemcpyDeviceToHost));


    ASSERT_particles_occupied(res_particles_occupied_x, res_particles_occupied_y, h_particles_occupied_unique_x, h_particles_occupied_unique_y,
        "Occupied", PARTICLES_OCCUPIED_UNIQUE_LEN, false);
    ASSERT_particles_occupied(res_particles_free_x, res_particles_free_y, h_particles_free_unique_x, h_particles_free_unique_y,
        "Free", PARTICLES_FREE_UNIQUE_LEN, false);

    /********************************************************************/
    /************************* LOG-ODDS VARIABLES ***********************/
    /********************************************************************/
    size_t sz_log_odds = (GRID_WIDTH * GRID_HEIGHT) * sizeof(float);

    float* d_log_odds = NULL;

    int* res_grid_map = (int*)malloc(sz_map);
    float* res_log_odds = (float*)malloc(sz_log_odds);
    memset(res_log_odds, 0, sz_log_odds);

    gpuErrchk(hipMalloc((void**)&d_log_odds, sz_log_odds));
    gpuErrchk(hipMemcpy(d_log_odds, pre_log_odds, sz_log_odds, hipMemcpyHostToDevice));


    /********************************************************************/
    /************************** LOG-ODDS KERNEL *************************/
    /********************************************************************/
    auto start_update_map = std::chrono::high_resolution_clock::now();

    threadsPerBlock = 256;
    blocksPerGrid = (PARTICLES_OCCUPIED_UNIQUE_LEN + threadsPerBlock - 1) / threadsPerBlock;
    kernel_update_log_odds << <blocksPerGrid, threadsPerBlock >> > (d_log_odds, d_particles_occupied_x, d_particles_occupied_y, 2 * log_t, GRID_WIDTH, GRID_HEIGHT, PARTICLES_OCCUPIED_UNIQUE_LEN);
    hipDeviceSynchronize();

    threadsPerBlock = 256;
    blocksPerGrid = (PARTICLES_FREE_UNIQUE_LEN + threadsPerBlock - 1) / threadsPerBlock;
    kernel_update_log_odds << <blocksPerGrid, threadsPerBlock >> > (d_log_odds, d_particles_free_x, d_particles_free_y, (-1) * log_t, GRID_WIDTH, GRID_HEIGHT, PARTICLES_FREE_UNIQUE_LEN);
    hipDeviceSynchronize();

    threadsPerBlock = 256;
    blocksPerGrid = ((GRID_WIDTH * GRID_HEIGHT) + threadsPerBlock - 1) / threadsPerBlock;
    kernel_update_map << <blocksPerGrid, threadsPerBlock >> > (d_grid_map, d_log_odds, LOG_ODD_PRIOR, WALL, FREE, GRID_WIDTH * GRID_HEIGHT);
    hipDeviceSynchronize();

    auto stop_update_map = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_log_odds, d_log_odds, sz_log_odds, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_grid_map, d_grid_map, sz_map, hipMemcpyDeviceToHost));

    ASSERT_log_odds(res_log_odds, pre_log_odds, post_log_odds, (GRID_WIDTH * GRID_HEIGHT));
    ASSERT_log_odds_maps(res_grid_map, pre_grid_map, post_grid_map, (GRID_WIDTH * GRID_HEIGHT));
    printf("\n");

    auto duration_create_map = std::chrono::duration_cast<std::chrono::microseconds>(stop_create_map - start_create_map);
    auto duration_unique_counter = std::chrono::duration_cast<std::chrono::microseconds>(stop_unique_counter - start_unique_counter);
    auto duration_unique_sum = std::chrono::duration_cast<std::chrono::microseconds>(stop_unique_sum - start_unique_sum);
    auto duration_restructure_map = std::chrono::duration_cast<std::chrono::microseconds>(stop_restructure_map - start_restructure_map);
    auto duration_update_map = std::chrono::duration_cast<std::chrono::microseconds>(stop_update_map - start_update_map);

    std::cout << "Time taken by function (Create Map): " << duration_create_map.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Unique Counter): " << duration_unique_counter.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Unique Sum): " << duration_unique_sum.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Restructure Map): " << duration_restructure_map.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Update Map): " << duration_update_map.count() << " microseconds" << std::endl;

}

#endif

#ifdef MAP_FUNC_EXEC
void host_map() {

    /********************************************************************/
    /********************* IMAGE TRANSFORM VARIABLES ********************/
    /********************************************************************/
    size_t sz_transition_frames = 9 * sizeof(float);
    size_t sz_lidar_coords = 2 * LIDAR_COORDS_LEN * sizeof(float);
    size_t sz_processed_measure_pos = LIDAR_COORDS_LEN * sizeof(int);
    size_t sz_particles_wframe_pos = LIDAR_COORDS_LEN * sizeof(float);
    size_t sz_position_image = 2 * sizeof(int);

    float* d_lidar_coords = NULL;
    float* d_transition_body_lidar = NULL;
    float* d_transition_world_body = NULL;
    float* d_transition_world_lidar = NULL;
    int* d_processed_measure_x = NULL;
    int* d_processed_measure_y = NULL;
    float* d_particles_wframe_x = NULL;
    float* d_particles_wframe_y = NULL;
    int* d_position_image_body = NULL;

    float* res_transition_world_lidar = (float*)malloc(sz_transition_frames);
    int* res_processed_measure_x = (int*)malloc(sz_processed_measure_pos);
    int* res_processed_measure_y = (int*)malloc(sz_processed_measure_pos);
    float* res_particles_wframe_x = (float*)malloc(sz_particles_wframe_pos);
    float* res_particles_wframe_y = (float*)malloc(sz_particles_wframe_pos);
    int* res_position_image_body = (int*)malloc(sz_position_image);

    gpuErrchk(hipMalloc((void**)&d_lidar_coords, sz_lidar_coords));
    gpuErrchk(hipMalloc((void**)&d_transition_body_lidar, sz_transition_frames));
    gpuErrchk(hipMalloc((void**)&d_transition_world_body, sz_transition_frames));
    gpuErrchk(hipMalloc((void**)&d_transition_world_lidar, sz_transition_frames));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_x, sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_y, sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_wframe_x, sz_particles_wframe_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_wframe_y, sz_particles_wframe_pos));
    gpuErrchk(hipMalloc((void**)&d_position_image_body, sz_position_image));

    gpuErrchk(hipMemcpy(d_lidar_coords, lidar_coords, sz_lidar_coords, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_transition_body_lidar, h_transition_body_lidar, sz_transition_frames, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_transition_world_body, h_transition_world_body, sz_transition_frames, hipMemcpyHostToDevice));


    /********************************************************************/
    /************************ BRESENHAM VARIABLES ***********************/
    /********************************************************************/
    int MAX_DIST_IN_MAP = sqrt(pow(GRID_WIDTH, 2) + pow(GRID_HEIGHT, 2));
    int PARTICLE_UNIQUE_COUNTER = PARTICLES_OCCUPIED_LEN + 1;

    size_t sz_particles_occupied_pos = PARTICLES_OCCUPIED_LEN * sizeof(int);
    size_t sz_particles_free_pos = 0;
    size_t sz_particles_free_pos_max = PARTICLES_OCCUPIED_LEN * MAX_DIST_IN_MAP * sizeof(int);
    size_t sz_particles_free_counter = PARTICLE_UNIQUE_COUNTER * sizeof(int);
    size_t sz_position_image_body = 2 * sizeof(int);

    int* d_particles_occupied_x = NULL;
    int* d_particles_occupied_y = NULL;
    int* d_particles_occupied_idx = NULL;
    int* d_particles_free_x = NULL;
    int* d_particles_free_y = NULL;
    int* d_particles_free_x_max = NULL;
    int* d_particles_free_y_max = NULL;
    int* d_particles_free_counter = NULL;
    int* d_particles_free_idx = NULL;
    //int* d_position_image_body = NULL;

    gpuErrchk(hipMalloc((void**)&d_particles_occupied_x, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_occupied_y, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_x_max, sz_particles_free_pos_max));
    gpuErrchk(hipMalloc((void**)&d_particles_free_y_max, sz_particles_free_pos_max));
    gpuErrchk(hipMalloc((void**)&d_particles_free_counter, sz_particles_free_counter));
    gpuErrchk(hipMalloc((void**)&d_particles_free_idx, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_position_image_body, sz_position_image_body));

    int* res_particles_free_x = (int*)malloc(sz_particles_free_pos);
    int* res_particles_free_y = (int*)malloc(sz_particles_free_pos);
    int* res_particles_free_counter = (int*)malloc(sz_particles_free_counter);

    gpuErrchk(hipMemcpy(d_particles_occupied_x, h_particles_occupied_x, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_occupied_y, h_particles_occupied_y, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_idx, h_particles_free_idx, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_position_image_body, h_position_image_body, sz_position_image_body, hipMemcpyHostToDevice));

    memset(res_particles_free_x, 0, sz_particles_free_pos);
    memset(res_particles_free_y, 0, sz_particles_free_pos);
    gpuErrchk(hipMemset(d_particles_free_x, 0, sz_particles_free_pos));
    gpuErrchk(hipMemset(d_particles_free_y, 0, sz_particles_free_pos));
    gpuErrchk(hipMemset(d_particles_free_x_max, 0, sz_particles_free_pos_max));
    gpuErrchk(hipMemset(d_particles_free_y_max, 0, sz_particles_free_pos_max));
    gpuErrchk(hipMemset(d_particles_free_counter, 0, sz_particles_free_counter));


    /********************************************************************/
    /**************************** MAP VARIABLES *************************/
    /********************************************************************/
    size_t sz_particles_occupied_pos = 0; // ELEMS_PARTICLES * sizeof(int);
    size_t sz_particles_free_pos = 0; // ELEMS_PARTICLES_IFRAME * sizeof(int);
    size_t sz_map = (GRID_WIDTH * GRID_HEIGHT) * sizeof(int);

    int* d_grid_map = NULL;
    int* d_particles_occupied_x = NULL;
    int* d_particles_occupied_y = NULL;
    int* d_particles_free_x = NULL;
    int* d_particles_free_y = NULL;

    gpuErrchk(hipMalloc((void**)&d_grid_map, sz_map));
    gpuErrchk(hipMalloc((void**)&d_particles_occupied_x, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_occupied_y, sz_particles_occupied_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_x, sz_particles_free_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_y, sz_particles_free_pos));

    gpuErrchk(hipMemcpy(d_grid_map, pre_grid_map, sz_map, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_occupied_x, h_particles_occupied_x, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_occupied_y, h_particles_occupied_y, sz_particles_occupied_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_x, h_particles_free_x, sz_particles_free_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_y, h_particles_free_y, sz_particles_free_pos, hipMemcpyHostToDevice));

    /********************************************************************/
    /************************* LOG-ODDS VARIABLES ***********************/
    /********************************************************************/
    size_t   sz_map_2d = GRID_WIDTH * GRID_HEIGHT * sizeof(uint8_t);
    size_t   sz_unique_counter = 1 * sizeof(int);
    size_t   sz_unique_counter_col = (GRID_WIDTH + 1) * sizeof(int);

    uint8_t* d_map_occupied_2d = NULL;
    uint8_t* d_map_free_2d = NULL;
    int* d_unique_occupied_counter = NULL;
    int* d_unique_free_counter = NULL;
    int* d_unique_occupied_counter_col = NULL;
    int* d_unique_free_counter_col = NULL;

    int* res_unique_occupied_counter = (int*)malloc(sz_unique_counter);
    int* res_unique_free_counter = (int*)malloc(sz_unique_counter);
    int* res_unique_occupied_counter_col = (int*)malloc(sz_unique_counter_col);
    int* res_unique_free_counter_col = (int*)malloc(sz_unique_counter_col);


    gpuErrchk(hipMalloc((void**)&d_map_occupied_2d, sz_map_2d));
    gpuErrchk(hipMalloc((void**)&d_map_free_2d, sz_map_2d));
    gpuErrchk(hipMalloc((void**)&d_unique_occupied_counter, sz_unique_counter));
    gpuErrchk(hipMalloc((void**)&d_unique_occupied_counter_col, sz_unique_counter_col));
    gpuErrchk(hipMalloc((void**)&d_unique_free_counter, sz_unique_counter));
    gpuErrchk(hipMalloc((void**)&d_unique_free_counter_col, sz_unique_counter_col));

    size_t sz_particles_idx = 2 * sizeof(int);

    int h_particles_occupied_idx[] = { 0, ELEMS_PARTICLES };
    int h_particles_free_idx[] = { 0, ELEMS_PARTICLES_IFRAME };
    int* d_particles_occupied_idx = NULL;
    int* d_particles_free_idx = NULL;

    gpuErrchk(hipMalloc((void**)&d_particles_occupied_idx, sz_particles_idx));
    gpuErrchk(hipMalloc((void**)&d_particles_free_idx, sz_particles_idx));

    gpuErrchk(hipMemset(d_map_occupied_2d, 0, sz_map_2d));
    gpuErrchk(hipMemset(d_map_free_2d, 0, sz_map_2d));

    gpuErrchk(hipMemcpy(d_particles_occupied_idx, h_particles_occupied_idx, sz_particles_idx, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particles_free_idx, h_particles_free_idx, sz_particles_idx, hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(d_unique_occupied_counter_col, 0, sz_unique_counter_col));
    gpuErrchk(hipMemset(d_unique_free_counter_col, 0, sz_unique_counter_col));


    /********************************************************************/
    /***************** World to IMAGE TRANSFORM KERNEL ******************/
    /********************************************************************/
    auto start_world_to_image_transform = std::chrono::high_resolution_clock::now();
    kernel_matrix_mul_3x3 << < 1, 1 >> > (d_transition_world_body, d_transition_body_lidar, d_transition_world_lidar);
    hipDeviceSynchronize();

    int threadsPerBlock = 1;
    int blocksPerGrid = LIDAR_COORDS_LEN;
    kernel_update_particles_lidar << < blocksPerGrid, threadsPerBlock >> > (d_transition_world_lidar, d_processed_measure_x, d_processed_measure_y,
        d_particles_wframe_x, d_particles_wframe_y, d_lidar_coords, res, xmin, ymax, LIDAR_COORDS_LEN);
    hipDeviceSynchronize();

    kernel_position_to_image << < 1, 1 >> > (d_position_image_body, d_transition_world_lidar, res, xmin, ymax);
    hipDeviceSynchronize();

    auto stop_world_to_image_transform = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_transition_world_lidar, d_transition_world_lidar, sz_transition_frames, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_x, d_processed_measure_x, sz_processed_measure_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_y, d_processed_measure_y, sz_processed_measure_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_wframe_x, d_particles_wframe_x, sz_particles_wframe_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_wframe_y, d_particles_wframe_y, sz_particles_wframe_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_position_image_body, d_position_image_body, sz_position_image, hipMemcpyDeviceToHost));

    //ASSERT_transition_world_lidar(res_transition_world_lidar, h_transition_world_lidar, 9, false);
    //ASSERT_particles_wframe(res_particles_wframe_x, res_particles_wframe_y, h_particles_wframe_x, h_particles_wframe_y, LIDAR_COORDS_LEN, false);
    //ASSERT_processed_measurements(res_processed_measure_x, res_processed_measure_y, h_particles_x, h_particles_y, LIDAR_COORDS_LEN);
    ASSERT_position_image_body(res_position_image_body, h_position_image_body);


    /********************************************************************/
    /************************* BRESENHAM KERNEL *************************/
    /********************************************************************/
    auto start_bresenham = std::chrono::high_resolution_clock::now();

    threadsPerBlock = 256;
    blocksPerGrid = (PARTICLES_OCCUPIED_LEN + threadsPerBlock - 1) / threadsPerBlock;
    kernel_bresenham << <blocksPerGrid, threadsPerBlock >> > (d_particles_occupied_x, d_particles_occupied_y, d_position_image_body,
        d_particles_free_x_max, d_particles_free_y_max, d_particles_free_counter, PARTICLES_OCCUPIED_LEN, MAX_DIST_IN_MAP);
    hipDeviceSynchronize();
    kernel_update_unique_sum << <1, 1 >> > (d_particles_free_counter, PARTICLE_UNIQUE_COUNTER);
    hipDeviceSynchronize();
    auto stop_bresenham = std::chrono::high_resolution_clock::now();

    auto start_bresenham_rearrange = std::chrono::high_resolution_clock::now();
    gpuErrchk(hipMemcpy(res_particles_free_counter, d_particles_free_counter, sz_particles_free_counter, hipMemcpyDeviceToHost));

    const int PARTICLES_NEW_LEN = res_particles_free_counter[PARTICLE_UNIQUE_COUNTER - 1];
    sz_particles_free_pos = PARTICLES_NEW_LEN * sizeof(int);
    gpuErrchk(hipMalloc((void**)&d_particles_free_x, sz_particles_free_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_free_y, sz_particles_free_pos));

    kernel_bresenham_rearrange << <blocksPerGrid, threadsPerBlock >> > (d_particles_free_x, d_particles_free_y, d_particles_free_x_max, d_particles_free_y_max,
        d_particles_free_counter, MAX_DIST_IN_MAP, PARTICLES_OCCUPIED_LEN);
    hipDeviceSynchronize();
    auto stop_bresenham_rearrange = std::chrono::high_resolution_clock::now();



    res_particles_free_x = (int*)malloc(sz_particles_free_pos);
    res_particles_free_y = (int*)malloc(sz_particles_free_pos);

    gpuErrchk(hipMemcpy(res_particles_free_x, d_particles_free_x, sz_particles_free_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_free_y, d_particles_free_y, sz_particles_free_pos, hipMemcpyDeviceToHost));

    ASSERT_particles_free_index(res_particles_free_counter, h_particles_free_idx, PARTICLES_OCCUPIED_LEN, false);
    ASSERT_particles_free_new_len(PARTICLES_NEW_LEN, PARTICLES_FREE_LEN);
    ASSERT_particles_free(res_particles_free_x, res_particles_free_y, h_particles_free_x, h_particles_free_y, PARTICLES_NEW_LEN);


    auto duration_world_to_image_transform = std::chrono::duration_cast<std::chrono::microseconds>(stop_world_to_image_transform - start_world_to_image_transform);
    auto duration_bresenham = std::chrono::duration_cast<std::chrono::microseconds>(stop_bresenham - start_bresenham);
    auto duration_bresenham_rearrange = std::chrono::duration_cast<std::chrono::microseconds>(stop_bresenham_rearrange - start_bresenham_rearrange);

    std::cout << "Time taken by function (World to Image Transform Kernel): " << duration_world_to_image_transform.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Bresenham): " << duration_bresenham.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Bresenham Rearrange): " << duration_bresenham_rearrange.count() << " microseconds" << std::endl;

}
#endif



#ifdef UPDATE_STATE_EXEC
void host_update_state() {

    thrust::device_vector<float> d_temp(xs, xs + NUM_PARTICLES);

    size_t sz_states_pos = NUM_PARTICLES * sizeof(float);

    float* d_states_x       = NULL;
    float* d_states_y       = NULL;
    float* d_states_theta   = NULL;

    gpuErrchk(hipMalloc((void**)&d_states_x,       sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_states_y,       sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_states_theta,   sz_states_pos));


    gpuErrchk(hipMemcpy(d_states_x, xs, sz_states_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_states_y, ys, sz_states_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_states_theta, thetas, sz_states_pos, hipMemcpyHostToDevice));


    auto start_update_states = std::chrono::high_resolution_clock::now();

    thrust::device_vector<float> d_vec_states_x(d_states_x, d_states_x + NUM_PARTICLES);
    thrust::device_vector<float> d_vec_states_y(d_states_y, d_states_y + NUM_PARTICLES);
    thrust::device_vector<float> d_vec_states_theta(d_states_theta, d_states_theta + NUM_PARTICLES);


    thrust::host_vector<float> h_vec_states_x(d_vec_states_x.begin(), d_vec_states_x.end());
    thrust::host_vector<float> h_vec_states_y(d_vec_states_y.begin(), d_vec_states_y.end());
    thrust::host_vector<float> h_vec_states_theta(d_vec_states_theta.begin(), d_vec_states_theta.end());

    std::vector<float> std_vec_states_x(h_vec_states_x.begin(), h_vec_states_x.end());
    std::vector<float> std_vec_states_y(h_vec_states_y.begin(), h_vec_states_y.end());
    std::vector<float> std_vec_states_theta(h_vec_states_theta.begin(), h_vec_states_theta.end());


    std::map<std::tuple<float, float, float>, int> states;

    for (int i = 0; i < NUM_PARTICLES; i++) {

        if (states.find(std::make_tuple(std_vec_states_x[i], std_vec_states_y[i], std_vec_states_theta[i])) == states.end()) {
            states.insert({ std::make_tuple(std_vec_states_x[i], std_vec_states_y[i], std_vec_states_theta[i]), 1 });
        }
        else {
            states[std::make_tuple(std_vec_states_x[i], std_vec_states_y[i], std_vec_states_theta[i])] += 1;
        }
    }

    std::map<std::tuple<float, float, float>, int>::iterator best
        = std::max_element(states.begin(), states.end(), [](const std::pair<std::tuple<float, float, float>, int>& a,
            const std::pair<std::tuple<float, float, float>, int>& b)->bool { return a.second < b.second; });

    auto key = best->first;
    // std::cout << std::get<0>(key) << " " << std::get<1>(key) << " " << std::get<2>(key) << " " << best->second << "\n";

    float theta = std::get<2>(key);
    float _T_wb[] = { cos(theta), -sin(theta), std::get<0>(key),
                        sin(theta),  cos(theta), std::get<1>(key),
                        0, 0, 1 };

    auto stop_update_states = std::chrono::high_resolution_clock::now();
    auto duration_update_states = std::chrono::duration_cast<std::chrono::microseconds>(stop_update_states - start_update_states);

    std::cout << "Time taken by function (Update States): " << duration_update_states.count() << " microseconds" << std::endl;

    for (int i = 0; i < 9; i++) {
        printf("%f  ", _T_wb[i]);
        assert(T_wb[i] == _T_wb[i]);
    }
    printf("\n");
    printf("%f, %f, %f\n", std::get<0>(key), std::get<1>(key), std::get<2>(key));

    
}
#endif

#ifdef UPDATE_PARTICLE_WEIGHTS_EXEC
void host_update_particle_weights() {

    /********************************************************************/
    /************************ WEIGHTS VARIABLES *************************/
    /********************************************************************/
    size_t sz_weights       = NUM_PARTICLES * sizeof(float);
    size_t sz_weights_max   = sizeof(float);
    size_t sz_sum_exp       = sizeof(double);
    
    float*  d_weights        = NULL;
    float*  d_weights_max    = NULL;
    double* d_sum_exp        = NULL;


    float*  res_weights      = (float*)malloc(sz_weights);
    float*  res_weights_max  = (float*)malloc(sz_weights_max);
    double* res_sum_exp      = (double*)malloc(sz_sum_exp);

    gpuErrchk(hipMalloc((void**)&d_weights,        sz_weights));
    gpuErrchk(hipMalloc((void**)&d_weights_max,    sz_weights_max));
    gpuErrchk(hipMalloc((void**)&d_sum_exp,        sz_sum_exp));


    gpuErrchk(hipMemcpy(d_weights, pre_weights, sz_weights, hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(d_weights_max, 0, sz_weights_max));
    gpuErrchk(hipMemset(d_sum_exp,     0, sz_sum_exp));


    /********************************************************************/
    /********************** UPDATE WEIGHTS KERNEL ***********************/
    /********************************************************************/
    auto start_update_particle_weights = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = 1;
    int blocksPerGrid = 1;

    kernel_arr_max << < blocksPerGrid, threadsPerBlock >> > (d_weights, d_weights_max, NUM_PARTICLES);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(res_weights_max, d_weights_max, sz_weights_max, hipMemcpyDeviceToHost));
    assert(res_weights_max[0] == ARR_MAX);

    float norm_value = -res_weights_max[0] + 50;

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_arr_increase << < blocksPerGrid, threadsPerBlock >> > (d_weights, norm_value, 0);
    hipDeviceSynchronize();

    threadsPerBlock = 1;
    blocksPerGrid = 1;
    kernel_arr_sum_exp << < blocksPerGrid, threadsPerBlock >> > (d_weights, d_sum_exp, NUM_PARTICLES);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(res_sum_exp, d_sum_exp, sz_sum_exp, hipMemcpyDeviceToHost));

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_arr_normalize << < blocksPerGrid, threadsPerBlock >> > (d_weights, res_sum_exp[0]);
    hipDeviceSynchronize();

    auto stop_update_particle_weights = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_weights, d_weights, sz_weights, hipMemcpyDeviceToHost));

    ASSERT_update_particle_weights(res_weights, weights, NUM_PARTICLES, false);

    auto duration_update_particle_weights = std::chrono::duration_cast<std::chrono::microseconds>(stop_update_particle_weights - start_update_particle_weights);
    std::cout << "Time taken by function (Update Particle Weights): " << duration_update_particle_weights.count() << " microseconds" << std::endl;
}
#endif

#ifdef RESAMPLING_EXEC
void host_resampling() {

    /********************************************************************/
    /*********************** RESAMPLING VARIABLES ***********************/
    /********************************************************************/
    float*  d_weights   = NULL;
    int*    d_js        = NULL;
    float*  d_rnd       = NULL;

    size_t sz_weights   = NUM_PARTICLES * sizeof(float);
    size_t sz_js        = NUM_PARTICLES * sizeof(int);
    size_t sz_rnd       = NUM_PARTICLES * sizeof(float);

    int* res_js = (int*)malloc(sz_js);

    gpuErrchk(hipMalloc((void**)&d_weights,    sz_weights));
    gpuErrchk(hipMalloc((void**)&d_js,         sz_js));
    gpuErrchk(hipMalloc((void**)&d_rnd,        sz_rnd));

    hipMemcpy(d_weights,   weights,    sz_weights,     hipMemcpyHostToDevice);
    hipMemcpy(d_rnd,       rnds,       sz_rnd,         hipMemcpyHostToDevice);
    gpuErrchk(hipMemset(d_js, 0, sz_js));

    /********************************************************************/
    /************************ RESAMPLING kerenel ************************/
    /********************************************************************/
    auto start_resampling = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = NUM_PARTICLES;
    int blocksPerGrid = 1;

    kernel_resampling << <blocksPerGrid, threadsPerBlock >> > (d_weights, d_js, d_rnd, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_resampling = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_js, d_js, sz_js, hipMemcpyDeviceToHost));

    auto duration_resampling = std::chrono::duration_cast<std::chrono::microseconds>(stop_resampling - start_resampling);
    std::cout << "Time taken by function (Kernel Resampling): " << duration_resampling.count() << " microseconds" << std::endl;

    ASSERT_resampling(res_js, js, NUM_PARTICLES, true);

}
#endif

#ifdef UPDATE_PARTICLES_EXEC
void host_update_particles() {

    // ✓
    // [✓] - Change Execution Time to 'chrono time'
    // [ ] - Create 'd_measure_idx'
    // [ ] - Create a kernel for initializing 'd_measure_idx'
    
#ifndef lidar_coords_LEN
    const int lidar_coords_LEN = LIDAR_COORDS_LEN;

#else
    int* processed_measure = (int*)malloc(2 * NUM_PARTICLES * lidar_coords_LEN * sizeof(int));
    for (int i = 0; i < 2 * NUM_PARTICLES * lidar_coords_LEN; i++)
        processed_measure[i] = h_processed_measure_pos_float[i];
#endif

    printf("lidar_coords_LEN: %d \n", lidar_coords_LEN);
    
    //const int lidar_coords_LEN = LIDAR_COORDS_LEN;

    /********************************************************************/
    /************************** PRIOR VARIABLES *************************/
    /********************************************************************/
    size_t sz_states        = NUM_PARTICLES * sizeof(float);
    size_t sz_lidar_coords  = 2 * lidar_coords_LEN * sizeof(float);

    float* d_states_x     = NULL;
    float* d_states_y     = NULL;
    float* d_states_theta = NULL;
    float* d_lidar_coords = NULL;

    gpuErrchk(hipMalloc((void**)&d_states_x,       sz_states));
    gpuErrchk(hipMalloc((void**)&d_states_y,       sz_states));
    gpuErrchk(hipMalloc((void**)&d_states_theta,   sz_states));
    gpuErrchk(hipMalloc((void**)&d_lidar_coords,   sz_lidar_coords));

    hipMemcpy(d_states_x,      h_states_x,     sz_states,          hipMemcpyHostToDevice);
    hipMemcpy(d_states_y,      h_states_y,     sz_states,          hipMemcpyHostToDevice);
    hipMemcpy(d_states_theta,  h_states_theta, sz_states,          hipMemcpyHostToDevice);
    hipMemcpy(d_lidar_coords,  lidar_coords,   sz_lidar_coords,    hipMemcpyHostToDevice);


    /********************************************************************/
    /************************* MIDDLE VARIABLES *************************/
    /********************************************************************/
    size_t sz_transition_body_frame     = 9 * NUM_PARTICLES * sizeof(float);
    size_t sz_transition_lidar_frame    = 9 * sizeof(float);
    size_t sz_transition_world_frame    = 9 * NUM_PARTICLES * sizeof(float);
    size_t sz_processed_measure_pos     = NUM_PARTICLES * lidar_coords_LEN * sizeof(int);
    size_t sz_measure_idx               = NUM_PARTICLES * lidar_coords_LEN * sizeof(int);

    float* d_transition_body_frame  = NULL;
    float* d_transition_lidar_frame = NULL;
    float* d_transition_world_frame = NULL;
    int*   d_processed_measure_x    = NULL;
    int*   d_processed_measure_y    = NULL;
    int*   d_measure_idx            = NULL;

    gpuErrchk(hipMalloc((void**)&d_transition_body_frame,  sz_transition_body_frame));
    gpuErrchk(hipMalloc((void**)&d_transition_lidar_frame, sz_transition_lidar_frame));
    gpuErrchk(hipMalloc((void**)&d_transition_world_frame, sz_transition_world_frame));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_x,    sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_y,    sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_measure_idx,            sz_measure_idx));


    /********************************************************************/
    /************************* HOST VARIABLES ***************************/
    /********************************************************************/
    float* res_transition_body_frame    = (float*)malloc(sz_transition_body_frame);
    float* res_transition_world_frame   = (float*)malloc(sz_transition_world_frame);
    int*   res_processed_measure_x      = (int*)malloc(sz_processed_measure_pos);
    int*   res_processed_measure_y      = (int*)malloc(sz_processed_measure_pos);
    int*   res_measure_idx              = (int*)malloc(sz_measure_idx);

    memset(res_transition_body_frame,   0, sz_transition_body_frame);
    memset(res_transition_world_frame,  0, sz_transition_world_frame);
    memset(res_processed_measure_x,     0, sz_processed_measure_pos);
    memset(res_processed_measure_y,     0, sz_processed_measure_pos);


    hipMemcpy(d_transition_body_frame,     res_transition_body_frame,  sz_transition_body_frame,   hipMemcpyHostToDevice);
    hipMemcpy(d_transition_world_frame,    res_transition_world_frame, sz_transition_world_frame,  hipMemcpyHostToDevice);
    hipMemcpy(d_processed_measure_x,       res_processed_measure_x,    sz_processed_measure_pos,   hipMemcpyHostToDevice);
    hipMemcpy(d_processed_measure_y,       res_processed_measure_y,    sz_processed_measure_pos,   hipMemcpyHostToDevice);
    hipMemcpy(d_transition_lidar_frame,    h_transition_lidar_frame,   sz_transition_lidar_frame,  hipMemcpyHostToDevice);


    /********************************************************************/
    /*************************** KERNEL EXEC ****************************/
    /********************************************************************/
    auto start_kernel = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = NUM_PARTICLES;
    int blocksPerGrid = 1;

    kernel_update_particles_states << <blocksPerGrid, threadsPerBlock >> > (d_states_x, d_states_y, d_states_theta, d_transition_body_frame, d_transition_lidar_frame, d_transition_world_frame, NUM_PARTICLES);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = lidar_coords_LEN;
    kernel_update_particles_lidar << < blocksPerGrid, threadsPerBlock >> > (d_transition_world_frame, d_processed_measure_x, d_processed_measure_y, d_lidar_coords, res, xmin, ymax, lidar_coords_LEN, NUM_PARTICLES);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_index_init_const << < blocksPerGrid, threadsPerBlock >> > (d_measure_idx, lidar_coords_LEN);
    hipDeviceSynchronize();

    threadsPerBlock = 1;
    blocksPerGrid = 1;
    kernel_update_unique_sum << < blocksPerGrid, threadsPerBlock >> > (d_measure_idx, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_kernel = std::chrono::high_resolution_clock::now();
    auto duration_kernel = std::chrono::duration_cast<std::chrono::microseconds>(stop_kernel - start_kernel);
    std::cout << "Time taken by function (Kernel): " << duration_kernel.count() << " microseconds" << std::endl;

    gpuErrchk(hipMemcpy(res_transition_body_frame,     d_transition_body_frame,    sz_transition_body_frame,   hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_transition_world_frame,    d_transition_world_frame,   sz_transition_world_frame,  hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_x,       d_processed_measure_x,      sz_processed_measure_pos,   hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_y,       d_processed_measure_y,      sz_processed_measure_pos,   hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_measure_idx,               d_measure_idx,              sz_measure_idx,             hipMemcpyDeviceToHost));

    bool printVerbose = false;

    for (int i = 0; i < 9 * NUM_PARTICLES; i++) {
        if (printVerbose == true) printf("%f, %f | ", res_transition_body_frame[i], h_transition_body_frame[i]);
        assert(abs(res_transition_body_frame[i] - h_transition_body_frame[i]) < 1e-5);
    }
    for (int i = 0; i < 9 * NUM_PARTICLES; i++) {
        if(printVerbose == true) printf("%f, %f |  ", res_transition_world_frame[i], h_transition_world_frame[i]);
        assert(abs(res_transition_world_frame[i] - h_transition_world_frame[i]) < 1e-5);
    }

    //for (int i = 0; i < 2 * NUM_PARTICLES * lidar_coords_LEN; i++) {
    //    if ( abs(res_Y_wo[i] - Y_wo[i]) > 1e-5 ) {
    //        // printf("%f, %f, %f, %i  |  ", res_Y_wo[i], Y_wi[i], Y_wo[i], i);
    //        printf("%f, %f, %i  |  ", res_Y_wo[i], Y_wo[i], i);
    //    }
    //    // assert( res_Y_wo[i] == Y_wi[i] );
    //}


    int notEqualCounter = 0;
    for (int i = 0; i < NUM_PARTICLES; i++) {
        int h_idx = 2 * i * lidar_coords_LEN;
        int res_idx = i * lidar_coords_LEN;
        for (int j = 0; j < lidar_coords_LEN; j++) {
            if (abs(res_processed_measure_x[j + res_idx] - processed_measure[j + h_idx]) > 1e-5) {
                printf("%d, %d, %d  |  ", (i * lidar_coords_LEN + j), res_processed_measure_x[j + res_idx], processed_measure[j + h_idx]);
                notEqualCounter += 1;
                if (notEqualCounter > 50)
                    exit(-1);
            }
        }
        h_idx += lidar_coords_LEN;
        for (int j = 0; j < lidar_coords_LEN; j++) {
            if (abs(res_processed_measure_y[j + res_idx] - processed_measure[j + h_idx]) > 1e-5) {
                printf("%d, %d, %d  |  ", (i * lidar_coords_LEN + j), res_processed_measure_y[j + res_idx], processed_measure[j + h_idx]);
                notEqualCounter += 1;
                if (notEqualCounter > 50)
                    exit(-1);
            }
        }
    }
    printf("\nProcessed Measure Error Count: %d\n", notEqualCounter);

    for (int i = 0; i < NUM_PARTICLES; i++) {
        int diff = (i == 0) ? 0 : (res_measure_idx[i] - res_measure_idx[i - 1]);
        if(printVerbose == true) printf("index %d --> value: %d, diff: %d\n", i, res_measure_idx[i], diff);
    }
}
#endif

#ifdef UPDATE_UNIQUE_EXEC
void host_update_unique() {

    int negative_before_counter = getNegativeCounter(h_particles_x_prior, h_particles_y_prior, BEFORE_LEN);
    int count_bigger_than_height = getGreaterThanCounter(h_particles_y_prior, GRID_HEIGHT, BEFORE_LEN);
    int negative_after_counter = getNegativeCounter(h_particles_x_post, h_particles_y_post, AFTER_LEN);;


    printf("GRID_WIDTH: %d, GRID_HEIGHT: %d\n", GRID_WIDTH, GRID_HEIGHT);
    printf("MEASURE_LEN: %d\n", MEASURE_LEN);
    printf("negative_before_counter: %d\n", negative_before_counter);
    printf("negative_after_counter: %d\n", negative_after_counter);
    printf("count_bigger_than_height: %d\n", count_bigger_than_height);

    // [ ] - Create a kernel for cum_sum
    // [✓] - Change name of variables to (d_particles_x, d_particles_y, d_particles_idx)
    // [ ] - Write down function input & output --> Inputs: (d_particles_x, d_particles_y, d_particles_idx) & (d_measure_x, d_measure_y, d_measure_idx)   /   Outputs: ()
    // [ ] - Write down variables that created in the function: ()
    // [ ] - print 'measure_idx'

    const int UNIQUE_COUNTER_LEN = NUM_PARTICLES + 1;

    /********************************************************************/
    /************************** PRIOR VARIABLES *************************/
    /********************************************************************/
    int* d_particles_x      = NULL;
    int* d_particles_y      = NULL;
    int* d_particles_idx    = NULL;
    size_t   sz_particles_pos = BEFORE_LEN * sizeof(int);
    size_t   sz_particles_idx = NUM_PARTICLES * sizeof(int);

    gpuErrchk(hipMalloc((void**)&d_particles_x, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_y, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_idx, sz_particles_idx));

    hipMemcpy(d_particles_x, h_particles_x_prior, sz_particles_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_y, h_particles_y_prior, sz_particles_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_idx, h_particles_idx_prior, sz_particles_idx, hipMemcpyHostToDevice);


    /********************************************************************/
    /**************************** MAP VARIABLES *************************/
    /********************************************************************/
    uint8_t* d_map_2d = NULL;
    int* d_unique_in_particle = NULL;
    int* d_unique_in_particle_col = NULL;

    size_t   sz_map_2d = GRID_WIDTH * GRID_HEIGHT * NUM_PARTICLES * sizeof(uint8_t);
    size_t   sz_unique_in_particle = UNIQUE_COUNTER_LEN * sizeof(int);
    size_t   sz_unique_in_particle_col = UNIQUE_COUNTER_LEN * GRID_WIDTH * sizeof(int);

    gpuErrchk(hipMalloc((void**)&d_map_2d,                 sz_map_2d));
    gpuErrchk(hipMalloc((void**)&d_unique_in_particle,     sz_unique_in_particle));
    gpuErrchk(hipMalloc((void**)&d_unique_in_particle_col, sz_unique_in_particle_col));

    int* h_unique_in_particle       = (int*)malloc(sz_unique_in_particle);
    int* h_unique_in_particle_col   = (int*)malloc(sz_unique_in_particle_col);
    uint8_t* h_map_2d               = (uint8_t*)malloc(sz_map_2d);

    hipMemset(d_map_2d, 0, sz_map_2d);
    hipMemset(d_unique_in_particle, 0, sz_unique_in_particle);
    hipMemset(d_unique_in_particle_col, 0, sz_unique_in_particle_col);

    /********************************************************************/
    /*********************** MEASUREMENT VARIABLES **********************/
    /********************************************************************/
    int* d_measure_x = NULL;
    int* d_measure_y = NULL;
    int* d_measure_idx = NULL;
    size_t sz_measure_pos = MEASURE_LEN * sizeof(int);
    size_t sz_measure_idx = NUM_PARTICLES * sizeof(int);

    gpuErrchk(hipMalloc((void**)&d_measure_x, sz_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_measure_y, sz_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_measure_idx, sz_measure_idx));

    hipMemcpy(d_measure_x, h_measure_x, sz_measure_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_measure_y, h_measure_y, sz_measure_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_measure_idx, h_measure_idx, sz_measure_idx, hipMemcpyHostToDevice);


    /********************************************************************/
    /**************************** CREATE MAP ****************************/
    /********************************************************************/
    int threadsPerBlock = 100; 
    int blocksPerGrid = NUM_PARTICLES; // NUM_ELEMS;

    auto start_create_map = std::chrono::high_resolution_clock::now();

    kernel_create_2d_map << <blocksPerGrid, threadsPerBlock >> > (d_particles_x, d_particles_y, d_particles_idx, BEFORE_LEN, d_map_2d, d_unique_in_particle,
        d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES); // NUM_ELEMS);
    hipDeviceSynchronize();

    auto stop_create_map = std::chrono::high_resolution_clock::now();


    /********************************************************************/
    /**************************** UPDATE MAP ****************************/
    /********************************************************************/
    auto start_update_map = std::chrono::high_resolution_clock::now();

    threadsPerBlock = NUM_PARTICLES; // NUM_ELEMS;
    blocksPerGrid = 1;

    kernel_update_2d_map_with_measure << <blocksPerGrid, threadsPerBlock >> > (d_measure_x, d_measure_y, d_measure_idx, MEASURE_LEN, d_map_2d, d_unique_in_particle,
        d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES); // NUM_ELEMS);
    hipDeviceSynchronize();

    auto stop_update_map = std::chrono::high_resolution_clock::now();


    /********************************************************************/
    /************************* CUMULATIVE SUM ***************************/
    /********************************************************************/
    threadsPerBlock = UNIQUE_COUNTER_LEN;
    blocksPerGrid = 1;

    auto start_cumulative_sum = std::chrono::high_resolution_clock::now();

    kernel_update_unique_sum << <1, 1 >> > (d_unique_in_particle, UNIQUE_COUNTER_LEN);
    kernel_update_unique_sum_col << <blocksPerGrid, threadsPerBlock >> > (d_unique_in_particle_col, GRID_WIDTH);
    hipDeviceSynchronize();

    auto stop_cumulative_sum = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(h_map_2d, d_map_2d, sz_map_2d, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_unique_in_particle, d_unique_in_particle, sz_unique_in_particle, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_unique_in_particle_col, d_unique_in_particle_col, sz_unique_in_particle_col, hipMemcpyDeviceToHost));

    int NEW_LEN = h_unique_in_particle[NUM_PARTICLES]; //[NUM_ELEMS - 1];
    gpuErrchk(hipFree(d_particles_x));
    gpuErrchk(hipFree(d_particles_y));

    sz_particles_pos = NEW_LEN * sizeof(int);
    gpuErrchk(hipMalloc((void**)&d_particles_x, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_y, sz_particles_pos));
    int* res_particles_x = (int*)malloc(sz_particles_pos);
    int* res_particles_y = (int*)malloc(sz_particles_pos);
    int* res_particles_idx = (int*)malloc(sz_particles_idx);


    /********************************************************************/
    /************************ MAP RESTRUCTURE ***************************/
    /********************************************************************/
    threadsPerBlock = GRID_WIDTH;
    blocksPerGrid = NUM_PARTICLES;

    auto start_map_restructure = std::chrono::high_resolution_clock::now();

    hipMemset(d_particles_idx, 0, sz_particles_idx);
    kernel_update_unique_restructure << <blocksPerGrid, threadsPerBlock >> > (d_map_2d, d_particles_x, d_particles_y, d_particles_idx,
        d_unique_in_particle, d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT);
    hipDeviceSynchronize();
    
    kernel_update_unique_sum << <1, 1 >> > (d_particles_idx, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_map_restructure = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_particles_x, d_particles_x, sz_particles_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_y, d_particles_y, sz_particles_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_idx, d_particles_idx, sz_particles_idx, hipMemcpyDeviceToHost));

    auto duration_create_map = std::chrono::duration_cast<std::chrono::milliseconds>(stop_create_map - start_create_map);
    auto duration_update_map = std::chrono::duration_cast<std::chrono::milliseconds>(stop_update_map - start_update_map);
    auto duration_cumulative_sum = std::chrono::duration_cast<std::chrono::microseconds>(stop_cumulative_sum - start_cumulative_sum);
    auto duration_map_restructure = std::chrono::duration_cast<std::chrono::milliseconds>(stop_map_restructure - start_map_restructure);

    std::cout << "Time taken by function (Create Map): " << duration_create_map.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Update Map): " << duration_update_map.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Cumulative Sum): " << duration_cumulative_sum.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Map Restructure): " << duration_map_restructure.count() << " milliseconds" << std::endl;

    printf("\nunique_in_particle: %d\n", NEW_LEN);
    printf("Measurement Length: %d\n", MEASURE_LEN);


    ASSERT_particles_pos_unique(res_particles_x, res_particles_y, h_particles_x_post, h_particles_y_post, NEW_LEN);

    //for (int i = 0; i < NUM_PARTICLES; i++) {
    //    int diff = (i == 0) ? 0 : (h_measure_idx[i] - h_measure_idx[i - 1]);
    //    printf("index %d --> value: %d, diff: %d\n", i, h_measure_idx[i], diff);
    //}

    //for (int i = 0; i < NUM_PARTICLES; i++) {
    //    printf("index %d: %d <> %d\n", i, res_particles_idx[i], h_particles_idx_post[i]);
    //}

    printf("All Passed\n");

}
#endif

#ifdef CORRELATION_EXEC
void host_correlation() {

    auto start_memory_copy = std::chrono::high_resolution_clock::now();

    /********************************************************************/
    /************************** PRIOR VARIABLES *************************/
    /********************************************************************/
    int* d_grid_map = NULL;
    int* d_particles_x = NULL;
    int* d_particles_y = NULL;
    int* d_particles_idx = NULL;
    int* d_extended_idx = NULL;

    const int num_elements_of_grid_map = GRID_WIDTH * GRID_HEIGHT;
    size_t sz_grid_map = num_elements_of_grid_map * sizeof(int);

    size_t sz_particles_pos = elems_particles * sizeof(int);
    size_t sz_particles_idx = NUM_PARTICLES * sizeof(int);
    size_t sz_extended_idx = elems_particles * sizeof(int);

    gpuErrchk(hipMalloc((void**)&d_grid_map, sz_grid_map));
    gpuErrchk(hipMalloc((void**)&d_particles_x, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_y, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_extended_idx, sz_extended_idx));
    gpuErrchk(hipMalloc((void**)&d_particles_idx, sz_particles_idx));

    hipMemcpy(d_grid_map, grid_map, sz_grid_map, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_x, h_particles_x, sz_particles_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_y, h_particles_y, sz_particles_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_idx, h_particles_idx, sz_particles_idx, hipMemcpyHostToDevice);

    size_t sz_weights = NUM_PARTICLES * sizeof(float);
    size_t sz_weights_raw = 25 * sz_weights;
    float* h_weights = (float*)malloc(sz_weights);
    int* h_extended_idx = (int*)malloc(sz_extended_idx);
    float* d_weights = NULL;
    float* d_weights_raw = NULL;
    memset(h_weights, 0, sz_weights);

    gpuErrchk(hipMalloc((void**)&d_weights, sz_weights));
    gpuErrchk(hipMalloc((void**)&d_weights_raw, sz_weights_raw));
    gpuErrchk(hipMemset(d_weights_raw, 0, sz_weights_raw));
    // gpuErrchk(hipMemcpy(d_all_correlation, h_correlation, sz_all_correlation, hipMemcpyHostToDevice));

    auto stop_memory_copy = std::chrono::high_resolution_clock::now();


    /********************************************************************/
    /*************************** PRINT SUMMARY **************************/
    /********************************************************************/
    printf("Elements of particles_x: %d,  Size of particles_x: %d\n", (int)elems_particles, (int)sz_particles_pos);
    printf("Elements of particles_y: %d,  Size of particles_y: %d\n", (int)elems_particles, (int)sz_particles_pos);
    printf("Elements of particles_idx: %d,  Size of particles_idx: %d\n", (int)elems_particles, (int)sz_extended_idx);

    printf("Elements of Grid_Map: %d,  Size of Grid_Map: %d\n", (int)num_elements_of_grid_map, (int)sz_grid_map);

    /********************************************************************/
    /************************* INDEX EXPANSION **************************/
    /********************************************************************/
    auto start_index_expansion = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = 100;
    int blocksPerGrid = NUM_PARTICLES;
    kernel_index_expansion << <blocksPerGrid, threadsPerBlock >> > (d_particles_idx, d_extended_idx, elems_particles);
    hipDeviceSynchronize();

    auto stop_index_expansion = std::chrono::high_resolution_clock::now();

    /********************************************************************/
    /************************ KERNEL CORRELATION ************************/
    /********************************************************************/
    threadsPerBlock = 256;
    blocksPerGrid = (elems_particles + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads, All Threads: %d\n", blocksPerGrid, threadsPerBlock, blocksPerGrid * threadsPerBlock);

    auto start_kernel = std::chrono::high_resolution_clock::now();

    kernel_correlation << <blocksPerGrid, threadsPerBlock >> > (d_grid_map, d_particles_x, d_particles_y, d_extended_idx, d_weights_raw, GRID_WIDTH, GRID_HEIGHT, elems_particles);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_correlation_max << <blocksPerGrid, threadsPerBlock >> > (d_weights_raw, d_weights, NUM_PARTICLES);

    auto stop_kernel = std::chrono::high_resolution_clock::now();


    gpuErrchk(hipMemcpy(h_weights, d_weights, sz_weights, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_extended_idx, d_extended_idx, sz_extended_idx, hipMemcpyDeviceToHost));


    bool all_equal = true;
    for (int i = 0; i < NUM_PARTICLES; i++) {
        // printf("index: %d --> %d, %d\n", i, final_result[i], new_weights[i]); 
        if (h_weights[i] != new_weights[i])
            all_equal = false;
    }


    std::cout << std::endl << "Execution Time: " << std::endl;
    auto duration_kernel = std::chrono::duration_cast<std::chrono::milliseconds>(stop_kernel - start_kernel);
    auto duration_memory_copy = std::chrono::duration_cast<std::chrono::milliseconds>(stop_memory_copy - start_memory_copy);
    auto duration_index_expansion = std::chrono::duration_cast<std::chrono::microseconds>(stop_index_expansion - start_index_expansion);
    std::cout << "Time taken by function (Correlation): " << duration_kernel.count() << " milliseconds" << std::endl;
    // std::cout << "Time taken by function (Memory Copy): " << duration_memory_copy.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Index Expansion): " << duration_index_expansion.count() << " microseconds" << std::endl;

    printf("All Equal: %s\n", all_equal ? "true" : "false");

    printf("Program Finished\n");

    gpuErrchk(hipFree(d_grid_map));
    gpuErrchk(hipFree(d_particles_x));
    gpuErrchk(hipFree(d_particles_y));
    gpuErrchk(hipFree(d_particles_idx));
    gpuErrchk(hipFree(d_extended_idx));
    gpuErrchk(hipFree(d_weights_raw));
}
#endif

#ifdef UPDATE_LOOP_EXEC
void host_update_loop() {

    // ✓

    int negative_before_counter  = getNegativeCounter(h_particles_x, h_particles_y, ELEMS_PARTICLES_START);
    int count_bigger_than_height = getGreaterThanCounter(h_particles_y, GRID_HEIGHT, ELEMS_PARTICLES_START);
    int negative_after_counter   = getNegativeCounter(h_particles_x_after_unique, h_particles_y_after_unique, ELEMS_PARTICLES_AFTER);;

    printf("GRID_WIDTH: %d, GRID_HEIGHT: %d\n", GRID_WIDTH, GRID_HEIGHT);
    printf("negative_before_counter: %d\n", negative_before_counter);
    printf("negative_after_counter: %d\n", negative_after_counter);
    printf("count_bigger_than_height: %d\n", count_bigger_than_height);


    // const int NUM_ELEMS     = NUM_PARTICLES + 1;
    const int UNIQUE_COUNTER_LEN = NUM_PARTICLES + 1;
    const int MEASURE_LEN   = NUM_PARTICLES * LIDAR_COORDS_LEN;

    printf("MEASURE_LEN: %d\n", MEASURE_LEN);

    /********************************************************************/
    /************************** PRIOR VARIABLES *************************/
    /********************************************************************/
    size_t sz_states_pos    = NUM_PARTICLES * sizeof(float);
    size_t sz_lidar_coords  = 2 * LIDAR_COORDS_LEN * sizeof(float);

    float* d_states_x           = NULL;
    float* d_states_y           = NULL;
    float* d_states_theta       = NULL;
    float* d_lidar_coords       = NULL;

    gpuErrchk(hipMalloc((void**)&d_states_x,       sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_states_y,       sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_states_theta,   sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_lidar_coords,   sz_lidar_coords));

    gpuErrchk(hipMemcpy(d_states_x,      h_states_x,     sz_states_pos,      hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_states_y,      h_states_y,     sz_states_pos,      hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_states_theta,  h_states_theta, sz_states_pos,      hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_lidar_coords,  lidar_coords,   sz_lidar_coords,    hipMemcpyHostToDevice));

    /********************************************************************/
    /**************************** MAP VARIABLES *************************/
    /********************************************************************/
    size_t sz_particles_pos = ELEMS_PARTICLES_START * sizeof(int);
    size_t sz_particles_idx = NUM_PARTICLES * sizeof(int);
    size_t sz_extended_idx  = ELEMS_PARTICLES_START * sizeof(int);
    size_t sz_grid_map      = GRID_WIDTH * GRID_HEIGHT * sizeof(int);

    int* d_grid_map         = NULL;
    int* d_particles_x      = NULL;
    int* d_particles_y      = NULL;
    int* d_particles_idx    = NULL;
    int* d_extended_idx     = NULL;

    int* res_particles_x    = (int*)malloc(sz_particles_pos);
    int* res_particles_y    = (int*)malloc(sz_particles_pos);
    int* res_particles_idx  = (int*)malloc(sz_particles_idx);
    int* res_extended_idx   = (int*)malloc(sz_extended_idx);

    gpuErrchk(hipMalloc((void**)&d_grid_map,       sz_grid_map));
    gpuErrchk(hipMalloc((void**)&d_particles_x,    sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_y,    sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_idx,  sz_particles_idx));
    gpuErrchk(hipMalloc((void**)&d_extended_idx,   sz_extended_idx));
    
    hipMemcpy(d_grid_map,      grid_map,         sz_grid_map,        hipMemcpyHostToDevice);
    hipMemcpy(d_particles_x,   h_particles_x,    sz_particles_pos,   hipMemcpyHostToDevice);
    hipMemcpy(d_particles_y,   h_particles_y,    sz_particles_pos,   hipMemcpyHostToDevice);
    hipMemcpy(d_particles_idx, h_particles_idx,  sz_particles_idx,   hipMemcpyHostToDevice);


    /********************************************************************/
    /********************** CORRELATION VARIABLES ***********************/
    /********************************************************************/
    size_t sz_weights       = NUM_PARTICLES * sizeof(float);
    size_t sz_correlation_raw   = 25 * sz_weights;

    float* h_weights      = (float*)malloc(sz_weights);
    int* h_extended_idx     = (int*)malloc(sz_extended_idx);
    float* res_weights    = (float*)malloc(sz_weights);
    float* d_weights      = NULL;
    float* d_weights_raw  = NULL;
    memset(h_weights, 0, sz_weights);

    gpuErrchk(hipMalloc((void**)&d_weights,        sz_weights));
    gpuErrchk(hipMalloc((void**)&d_weights_raw,    sz_correlation_raw));
    gpuErrchk(hipMemset(d_weights_raw,     0,      sz_correlation_raw));


    /********************************************************************/
    /*********************** TRANSITION VARIABLES ***********************/
    /********************************************************************/
    size_t sz_transition_body_frame     = 9 * NUM_PARTICLES * sizeof(float);
    size_t sz_transition_lidar_frame    = 9 * sizeof(float);
    size_t sz_transition_world_frame    = 9 * NUM_PARTICLES * sizeof(float);
    size_t sz_processed_measure_pos     = NUM_PARTICLES * LIDAR_COORDS_LEN * sizeof(int);
    size_t sz_measure_idx               = NUM_PARTICLES * LIDAR_COORDS_LEN * sizeof(int);

    float* d_transition_body_frame  = NULL;
    float* d_transition_lidar_frame = NULL;
    float* d_transition_world_frame = NULL;
    int*   d_processed_measure_x    = NULL;
    int*   d_processed_measure_y    = NULL;
    int*   d_measure_idx            = NULL;

    float* res_transition_body_frame    = (float*)malloc(sz_transition_body_frame);
    float* res_transition_world_frame   = (float*)malloc(sz_transition_world_frame);
    int*   res_processed_measure_x      = (int*)malloc(sz_processed_measure_pos);
    int*   res_processed_measure_y      = (int*)malloc(sz_processed_measure_pos);


    gpuErrchk(hipMalloc((void**)&d_transition_body_frame,  sz_transition_body_frame));
    gpuErrchk(hipMalloc((void**)&d_transition_lidar_frame, sz_transition_lidar_frame));
    gpuErrchk(hipMalloc((void**)&d_transition_world_frame, sz_transition_world_frame));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_x,    sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_y,    sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_measure_idx,            sz_measure_idx));


    gpuErrchk(hipMemset(d_transition_body_frame,   0, sz_transition_body_frame));
    gpuErrchk(hipMemset(d_transition_world_frame,  0, sz_transition_world_frame));
    gpuErrchk(hipMemset(d_processed_measure_x,     0, sz_processed_measure_pos));
    gpuErrchk(hipMemset(d_processed_measure_y,     0, sz_processed_measure_pos));
    gpuErrchk(hipMemset(d_measure_idx,             0, sz_measure_idx));

    hipMemcpy(d_transition_lidar_frame, h_transition_lidar_frame, sz_transition_lidar_frame, hipMemcpyHostToDevice);

    /********************************************************************/
    /**************************** MAP VARIABLES *************************/
    /********************************************************************/
    size_t   sz_map_2d                  = GRID_WIDTH * GRID_HEIGHT * NUM_PARTICLES * sizeof(uint8_t);
    size_t   sz_unique_in_particle      = UNIQUE_COUNTER_LEN * sizeof(int);
    size_t   sz_unique_in_particle_col  = UNIQUE_COUNTER_LEN * GRID_WIDTH * sizeof(int);

    uint8_t* d_map_2d                   = NULL;
    int*     d_unique_in_particle       = NULL;
    int*     d_unique_in_particle_col   = NULL;

    uint8_t* res_map_2d             = (uint8_t*)malloc(sz_map_2d);
    int* h_unique_in_particle       = (int*)malloc(sz_unique_in_particle);

    gpuErrchk(hipMalloc((void**)&d_map_2d,                 sz_map_2d));
    gpuErrchk(hipMalloc((void**)&d_unique_in_particle,     sz_unique_in_particle));
    gpuErrchk(hipMalloc((void**)&d_unique_in_particle_col, sz_unique_in_particle_col));
    
    gpuErrchk(hipMemset(d_map_2d,                    0,  sz_map_2d));
    gpuErrchk(hipMemset(d_unique_in_particle,        0,  sz_unique_in_particle));
    gpuErrchk(hipMemset(d_unique_in_particle_col,    0,  sz_unique_in_particle_col));

    /********************************************************************/
    /************************ TRANSITION KERNEL *************************/
    /********************************************************************/
    auto start_transition_kernel = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = NUM_PARTICLES;
    int blocksPerGrid = 1;

    kernel_update_particles_states << <blocksPerGrid, threadsPerBlock >> > (d_states_x, d_states_y, d_states_theta, d_transition_body_frame, d_transition_lidar_frame, d_transition_world_frame, NUM_PARTICLES);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = LIDAR_COORDS_LEN;
    kernel_update_particles_lidar << < blocksPerGrid, threadsPerBlock >> > (d_transition_world_frame, d_processed_measure_x, d_processed_measure_y, d_lidar_coords, res, xmin, ymax, LIDAR_COORDS_LEN, NUM_PARTICLES);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_index_init_const << < blocksPerGrid, threadsPerBlock >> > (d_measure_idx, LIDAR_COORDS_LEN);
    hipDeviceSynchronize();
    
    threadsPerBlock = 1;
    blocksPerGrid = 1;
    kernel_update_unique_sum << < blocksPerGrid, threadsPerBlock >> > (d_measure_idx, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_transition_kernel = std::chrono::high_resolution_clock::now();


    gpuErrchk(hipMemcpy(res_transition_body_frame, d_transition_body_frame, sz_transition_body_frame, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_transition_world_frame, d_transition_world_frame, sz_transition_world_frame, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_x, d_processed_measure_x, sz_processed_measure_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_y, d_processed_measure_y, sz_processed_measure_pos, hipMemcpyDeviceToHost));

    ASSERT_transition_frames(res_transition_body_frame, res_transition_world_frame, h_transition_body_frame, h_transition_world_frame, NUM_PARTICLES, false);
    ASSERT_processed_measurements(res_processed_measure_x, res_processed_measure_y, processed_measure, NUM_PARTICLES, LIDAR_COORDS_LEN);

    /********************************************************************/
    /************************** CREATE 2D MAP ***************************/
    /********************************************************************/
    threadsPerBlock = 100;
    blocksPerGrid = NUM_PARTICLES;

    auto start_create_map = std::chrono::high_resolution_clock::now();

    kernel_create_2d_map << <blocksPerGrid, threadsPerBlock >> > (d_particles_x, d_particles_y, d_particles_idx, ELEMS_PARTICLES_START, d_map_2d, d_unique_in_particle,
                                                                    d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_create_map = std::chrono::high_resolution_clock::now();

    //hipError_t err = hipPeekAtLastError();
    //printf("%s\n", hipGetErrorString(err));

    gpuErrchk(hipMemcpy(res_map_2d, d_map_2d, sz_map_2d, hipMemcpyDeviceToHost));

    ASSERT_create_2d_map_elements(res_map_2d, negative_before_counter, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES, ELEMS_PARTICLES_START);

    /********************************************************************/
    /**************************** UPDATE MAP ****************************/
    /********************************************************************/
    auto start_update_map = std::chrono::high_resolution_clock::now();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    
    kernel_update_2d_map_with_measure << <blocksPerGrid, threadsPerBlock >> > (d_processed_measure_x, d_processed_measure_y, d_measure_idx, 
        MEASURE_LEN, d_map_2d, d_unique_in_particle,
        d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_update_map = std::chrono::high_resolution_clock::now();

    /********************************************************************/
    /************************* CUMULATIVE SUM ***************************/
    /********************************************************************/
    threadsPerBlock = UNIQUE_COUNTER_LEN;
    blocksPerGrid = 1;

    auto start_cumulative_sum = std::chrono::high_resolution_clock::now();

    kernel_update_unique_sum << <1, 1 >> > (d_unique_in_particle, UNIQUE_COUNTER_LEN);
    kernel_update_unique_sum_col << <blocksPerGrid, threadsPerBlock >> > (d_unique_in_particle_col, GRID_WIDTH);
    hipDeviceSynchronize();

    auto stop_cumulative_sum = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(h_unique_in_particle, d_unique_in_particle, sz_unique_in_particle, hipMemcpyDeviceToHost));

    int NEW_LEN = h_unique_in_particle[UNIQUE_COUNTER_LEN - 1];
    ASSERT_new_len_calculation(NEW_LEN, ELEMS_PARTICLES_AFTER, negative_after_counter);


    /********************************************************************/
    /******************* REINITIALIZE MAP VARIABLES *********************/
    /********************************************************************/
    gpuErrchk(hipFree(d_particles_x));
    gpuErrchk(hipFree(d_particles_y));
    gpuErrchk(hipFree(d_extended_idx));
    free(res_particles_x);
    free(res_particles_y);

    sz_particles_pos = NEW_LEN * sizeof(int);
    sz_extended_idx  = NEW_LEN * sizeof(int);

    gpuErrchk(hipMalloc((void**)&d_particles_x, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_y, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_extended_idx, sz_extended_idx));

    res_particles_x = (int*)malloc(sz_particles_pos);
    res_particles_y = (int*)malloc(sz_particles_pos);


    /********************************************************************/
    /************************ MAP RESTRUCTURE ***************************/
    /********************************************************************/
    threadsPerBlock = GRID_WIDTH;
    blocksPerGrid = NUM_PARTICLES;

    auto start_map_restructure = std::chrono::high_resolution_clock::now();

    hipMemset(d_particles_idx, 0, sz_particles_idx);
    kernel_update_unique_restructure << <blocksPerGrid, threadsPerBlock >> > (d_map_2d, d_particles_x, d_particles_y, d_particles_idx,
                                                                                d_unique_in_particle, d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT);
    hipDeviceSynchronize();

    kernel_update_unique_sum << <1, 1 >> > (d_particles_idx, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_map_restructure = std::chrono::high_resolution_clock::now();

    auto start_copy_particles_pos = std::chrono::high_resolution_clock::now();
    gpuErrchk(hipMemcpy(res_particles_x, d_particles_x, sz_particles_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_y, d_particles_y, sz_particles_pos, hipMemcpyDeviceToHost));
    auto stop_copy_particles_pos = std::chrono::high_resolution_clock::now();

    ASSERT_particles_pos_unique(res_particles_x, res_particles_y, h_particles_x_after_unique, h_particles_y_after_unique, NEW_LEN);

    /********************************************************************/
    /************************* INDEX EXPANSION **************************/
    /********************************************************************/
    auto start_index_expansion = std::chrono::high_resolution_clock::now();

    threadsPerBlock = 100;
    blocksPerGrid = NUM_PARTICLES;

    kernel_index_expansion << <blocksPerGrid, threadsPerBlock >> > (d_particles_idx, d_extended_idx, NEW_LEN);
    hipDeviceSynchronize();

    auto stop_index_expansion = std::chrono::high_resolution_clock::now();

    res_extended_idx = (int*)malloc(sz_extended_idx);
    gpuErrchk(hipMemcpy(res_extended_idx, d_extended_idx, sz_extended_idx, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_idx, d_particles_idx, sz_particles_idx, hipMemcpyDeviceToHost));


    /********************************************************************/
    /************************ KERNEL CORRELATION ************************/
    /********************************************************************/
    threadsPerBlock = 256;
    blocksPerGrid = (NEW_LEN + threadsPerBlock - 1) / threadsPerBlock;
    printf("*** CUDA kernel launch with %d blocks of %d threads, All Threads: %d ***\n", blocksPerGrid, threadsPerBlock, blocksPerGrid * threadsPerBlock);

    auto start_correlation = std::chrono::high_resolution_clock::now();

    kernel_correlation << <blocksPerGrid, threadsPerBlock >> > (d_grid_map, d_particles_x, d_particles_y, d_extended_idx, d_weights_raw, GRID_WIDTH, GRID_HEIGHT, NEW_LEN);
    hipDeviceSynchronize();


    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_correlation_max << <blocksPerGrid, threadsPerBlock >> > (d_weights_raw, d_weights, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_correlation = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_weights,  d_weights,  sz_weights,  hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_extended_idx, d_extended_idx, sz_extended_idx, hipMemcpyDeviceToHost));

    ASSERT_correlation_Equality(res_weights, new_weights, NUM_PARTICLES);


    auto duration_create_map = std::chrono::duration_cast<std::chrono::milliseconds>(stop_create_map - start_create_map);
    auto duration_update_map = std::chrono::duration_cast<std::chrono::milliseconds>(stop_update_map - start_update_map);
    auto duration_cumulative_sum = std::chrono::duration_cast<std::chrono::milliseconds>(stop_cumulative_sum - start_cumulative_sum);
    auto duration_map_restructure = std::chrono::duration_cast<std::chrono::milliseconds>(stop_map_restructure - start_map_restructure);
    auto duration_copy_particles_pos = std::chrono::duration_cast<std::chrono::milliseconds>(stop_copy_particles_pos - start_copy_particles_pos);
    auto duration_transition_kernel = std::chrono::duration_cast<std::chrono::milliseconds>(stop_transition_kernel - start_transition_kernel);
    auto duration_correlation = std::chrono::duration_cast<std::chrono::milliseconds>(stop_correlation - start_correlation);
    auto duration_sum = duration_create_map + duration_update_map + duration_cumulative_sum + duration_map_restructure + duration_copy_particles_pos +
        duration_transition_kernel + duration_correlation;

    std::cout << "Time taken by function (Create Map): " << duration_create_map.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Update Map): " << duration_update_map.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Cumulative Sum): " << duration_cumulative_sum.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Map Restructure): " << duration_map_restructure.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Copy Particles): " << duration_copy_particles_pos.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Transition Kernel): " << duration_transition_kernel.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Correlation Kernel): " << duration_correlation.count() << " milliseconds" << std::endl;
    std::cout << "Time taken by function (Sum): " << duration_sum.count() << " milliseconds" << std::endl;

    printf("\nFinished All\n");

}
#endif

#ifdef UPDATE_FUNC_EXEC
void host_update_func() {

    thrust::device_vector<float> d_temp(h_states_x, h_states_x + NUM_PARTICLES);

    int negative_before_counter = getNegativeCounter(h_particles_x, h_particles_y, ELEMS_PARTICLES_START);
    int count_bigger_than_height = getGreaterThanCounter(h_particles_y, GRID_HEIGHT, ELEMS_PARTICLES_START);
    int negative_after_counter = getNegativeCounter(h_particles_x_after_resampling, h_particles_y_after_resampling, ELEMS_PARTICLES_AFTER);;

    printf("GRID_WIDTH: %d, GRID_HEIGHT: %d\n", GRID_WIDTH, GRID_HEIGHT);
    printf("negative_before_counter: %d\n", negative_before_counter);
    printf("negative_after_counter: %d\n", negative_after_counter);
    printf("count_bigger_than_height: %d\n", count_bigger_than_height);

    const int UNIQUE_COUNTER_LEN = NUM_PARTICLES + 1;
    const int MEASURE_LEN = NUM_PARTICLES * LIDAR_COORDS_LEN;

    printf("MEASURE_LEN: %d\n", MEASURE_LEN);

    /**************************************************************************************************************************************************/
    /**************************************************************** VARIABLES SCOPE *****************************************************************/
    /**************************************************************************************************************************************************/

    /********************************************************************/
    /************************** STATES VARIABLES ************************/
    /********************************************************************/
    size_t sz_states_pos = NUM_PARTICLES * sizeof(float);
    size_t sz_lidar_coords = 2 * LIDAR_COORDS_LEN * sizeof(float);

    float* d_states_x = NULL;
    float* d_states_y = NULL;
    float* d_states_theta = NULL;
    float* d_lidar_coords = NULL;

    float* res_states_x = (float*)malloc(sz_states_pos);
    float* res_states_y = (float*)malloc(sz_states_pos);
    float* res_states_theta = (float*)malloc(sz_states_pos);

    gpuErrchk(hipMalloc((void**)&d_states_x, sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_states_y, sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_states_theta, sz_states_pos));
    gpuErrchk(hipMalloc((void**)&d_lidar_coords, sz_lidar_coords));

    gpuErrchk(hipMemcpy(d_states_x, h_states_x, sz_states_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_states_y, h_states_y, sz_states_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_states_theta, h_states_theta, sz_states_pos, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_lidar_coords, lidar_coords, sz_lidar_coords, hipMemcpyHostToDevice));

    /********************************************************************/
    /**************************** MAP VARIABLES *************************/
    /********************************************************************/
    size_t sz_particles_pos = ELEMS_PARTICLES_START * sizeof(int);
    size_t sz_particles_idx = NUM_PARTICLES * sizeof(int);
    size_t sz_particles_weight = NUM_PARTICLES * sizeof(float);
    size_t sz_extended_idx = ELEMS_PARTICLES_START * sizeof(int);
    size_t sz_grid_map = GRID_WIDTH * GRID_HEIGHT * sizeof(int);

    int* d_grid_map = NULL;
    int* d_particles_x = NULL;
    int* d_particles_y = NULL;
    int* d_particles_idx = NULL;
    float* d_particles_weight = NULL;
    int* d_extended_idx = NULL;

    int* res_particles_x = (int*)malloc(sz_particles_pos);
    int* res_particles_y = (int*)malloc(sz_particles_pos);
    int* res_particles_idx = (int*)malloc(sz_particles_idx);
    float* res_particles_weight = (float*)malloc(sz_particles_weight);
    int* res_extended_idx = (int*)malloc(sz_extended_idx);

    gpuErrchk(hipMalloc((void**)&d_grid_map, sz_grid_map));
    gpuErrchk(hipMalloc((void**)&d_particles_x, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_y, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_idx, sz_particles_idx));
    gpuErrchk(hipMalloc((void**)&d_particles_weight, sz_particles_weight));
    gpuErrchk(hipMalloc((void**)&d_extended_idx, sz_extended_idx));

    hipMemcpy(d_grid_map, grid_map, sz_grid_map, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_x, h_particles_x, sz_particles_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_y, h_particles_y, sz_particles_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_idx, h_particles_idx, sz_particles_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_particles_weight, particles_weight_pre, sz_particles_weight, hipMemcpyHostToDevice);


    /********************************************************************/
    /******************** PARTICLES COPY VARIABLES **********************/
    /********************************************************************/
    int* dc_particles_x = NULL;
    int* dc_particles_y = NULL;
    int* dc_particles_idx = NULL;

    float* dc_states_x = NULL;
    float* dc_states_y = NULL;
    float* dc_states_theta = NULL;

    gpuErrchk(hipMalloc((void**)&dc_states_x, sz_states_pos));
    gpuErrchk(hipMalloc((void**)&dc_states_y, sz_states_pos));
    gpuErrchk(hipMalloc((void**)&dc_states_theta, sz_states_pos));

    /********************************************************************/
    /********************** CORRELATION VARIABLES ***********************/
    /********************************************************************/
    size_t sz_weights = NUM_PARTICLES * sizeof(float);
    size_t sz_correlation_raw = 25 * sz_weights;

    float* h_weights = (float*)malloc(sz_weights);
    int* h_extended_idx = (int*)malloc(sz_extended_idx);
    float* res_weights = (float*)malloc(sz_weights);
    float* d_weights = NULL;
    float* d_weights_raw = NULL;
    memset(h_weights, 0, sz_weights);

    gpuErrchk(hipMalloc((void**)&d_weights, sz_weights));
    gpuErrchk(hipMalloc((void**)&d_weights_raw, sz_correlation_raw));
    gpuErrchk(hipMemset(d_weights_raw, 0, sz_correlation_raw));


    /********************************************************************/
    /*********************** TRANSITION VARIABLES ***********************/
    /********************************************************************/
    size_t sz_transition_body_frame = 9 * NUM_PARTICLES * sizeof(float);
    size_t sz_transition_lidar_frame = 9 * sizeof(float);
    size_t sz_transition_world_frame = 9 * NUM_PARTICLES * sizeof(float);
    size_t sz_processed_measure_pos = NUM_PARTICLES * LIDAR_COORDS_LEN * sizeof(int);
    size_t sz_measure_idx = NUM_PARTICLES * LIDAR_COORDS_LEN * sizeof(int);

    float* d_transition_body_frame = NULL;
    float* d_transition_lidar_frame = NULL;
    float* d_transition_world_frame = NULL;
    int* d_processed_measure_x = NULL;
    int* d_processed_measure_y = NULL;
    int* d_measure_idx = NULL;

    float* res_transition_body_frame = (float*)malloc(sz_transition_body_frame);
    float* res_transition_world_frame = (float*)malloc(sz_transition_world_frame);
    int* res_processed_measure_x = (int*)malloc(sz_processed_measure_pos);
    int* res_processed_measure_y = (int*)malloc(sz_processed_measure_pos);


    gpuErrchk(hipMalloc((void**)&d_transition_body_frame, sz_transition_body_frame));
    gpuErrchk(hipMalloc((void**)&d_transition_lidar_frame, sz_transition_lidar_frame));
    gpuErrchk(hipMalloc((void**)&d_transition_world_frame, sz_transition_world_frame));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_x, sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_processed_measure_y, sz_processed_measure_pos));
    gpuErrchk(hipMalloc((void**)&d_measure_idx, sz_measure_idx));


    gpuErrchk(hipMemset(d_transition_body_frame, 0, sz_transition_body_frame));
    gpuErrchk(hipMemset(d_transition_world_frame, 0, sz_transition_world_frame));
    gpuErrchk(hipMemset(d_processed_measure_x, 0, sz_processed_measure_pos));
    gpuErrchk(hipMemset(d_processed_measure_y, 0, sz_processed_measure_pos));
    gpuErrchk(hipMemset(d_measure_idx, 0, sz_measure_idx));

    hipMemcpy(d_transition_lidar_frame, h_transition_lidar_frame, sz_transition_lidar_frame, hipMemcpyHostToDevice);

    /********************************************************************/
    /**************************** MAP VARIABLES *************************/
    /********************************************************************/
    size_t   sz_map_2d = GRID_WIDTH * GRID_HEIGHT * NUM_PARTICLES * sizeof(uint8_t);
    size_t   sz_unique_in_particle = UNIQUE_COUNTER_LEN * sizeof(int);
    size_t   sz_unique_in_particle_col = UNIQUE_COUNTER_LEN * GRID_WIDTH * sizeof(int);

    uint8_t* d_map_2d = NULL;
    int* d_unique_in_particle = NULL;
    int* d_unique_in_particle_col = NULL;

    uint8_t* res_map_2d = (uint8_t*)malloc(sz_map_2d);
    int* h_unique_in_particle = (int*)malloc(sz_unique_in_particle);

    gpuErrchk(hipMalloc((void**)&d_map_2d, sz_map_2d));
    gpuErrchk(hipMalloc((void**)&d_unique_in_particle, sz_unique_in_particle));
    gpuErrchk(hipMalloc((void**)&d_unique_in_particle_col, sz_unique_in_particle_col));

    gpuErrchk(hipMemset(d_map_2d, 0, sz_map_2d));
    gpuErrchk(hipMemset(d_unique_in_particle, 0, sz_unique_in_particle));
    gpuErrchk(hipMemset(d_unique_in_particle_col, 0, sz_unique_in_particle_col));

    /********************************************************************/
    /************************ WEIGHTS VARIABLES *************************/
    /********************************************************************/
    size_t sz_weights_max = sizeof(float);
    size_t sz_sum_exp = sizeof(double);

    float* d_weights_max = NULL;
    double* d_sum_exp = NULL;

    float* res_weights_max = (float*)malloc(sz_weights_max);
    double* res_sum_exp = (double*)malloc(sz_sum_exp);

    gpuErrchk(hipMalloc((void**)&d_weights_max, sz_weights_max));
    gpuErrchk(hipMalloc((void**)&d_sum_exp, sz_sum_exp));

    gpuErrchk(hipMemset(d_weights_max, 0, sz_weights_max));
    gpuErrchk(hipMemset(d_sum_exp, 0, sz_sum_exp));

    /********************************************************************/
    /*********************** RESAMPLING VARIABLES ***********************/
    /********************************************************************/
    int* d_js = NULL;
    float* d_rnd = NULL;

    size_t sz_js = NUM_PARTICLES * sizeof(int);
    size_t sz_rnd = NUM_PARTICLES * sizeof(float);

    int* res_js = (int*)malloc(sz_js);

    gpuErrchk(hipMalloc((void**)&d_js, sz_js));
    gpuErrchk(hipMalloc((void**)&d_rnd, sz_rnd));

    gpuErrchk(hipMemcpy(d_rnd, rnds, sz_rnd, hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(d_js, 0, sz_js));

    /********************************************************************/
    /********************* REARRANGEMENT VARIABLES **********************/
    /********************************************************************/
    std::vector<float> std_vec_states_x;
    std::vector<float> std_vec_states_y;
    std::vector<float> std_vec_states_theta;


    /**************************************************************************************************************************************************/
    /************************************************************* KERNEL EXECUTION SCOPE *************************************************************/
    /**************************************************************************************************************************************************/

    /********************************************************************/
    /************************ TRANSITION KERNEL *************************/
    /********************************************************************/
    auto start_transition_kernel = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = NUM_PARTICLES;
    int blocksPerGrid = 1;

    kernel_update_particles_states << <blocksPerGrid, threadsPerBlock >> > (d_states_x, d_states_y, d_states_theta, d_transition_body_frame, d_transition_lidar_frame, d_transition_world_frame, NUM_PARTICLES);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = LIDAR_COORDS_LEN;
    kernel_update_particles_lidar << < blocksPerGrid, threadsPerBlock >> > (d_transition_world_frame, d_processed_measure_x, d_processed_measure_y, d_lidar_coords, res, xmin, ymax, LIDAR_COORDS_LEN, NUM_PARTICLES);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_index_init_const << < blocksPerGrid, threadsPerBlock >> > (d_measure_idx, LIDAR_COORDS_LEN);
    hipDeviceSynchronize();

    threadsPerBlock = 1;
    blocksPerGrid = 1;
    kernel_update_unique_sum << < blocksPerGrid, threadsPerBlock >> > (d_measure_idx, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_transition_kernel = std::chrono::high_resolution_clock::now();


    gpuErrchk(hipMemcpy(res_transition_body_frame, d_transition_body_frame, sz_transition_body_frame, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_transition_world_frame, d_transition_world_frame, sz_transition_world_frame, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_x, d_processed_measure_x, sz_processed_measure_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_processed_measure_y, d_processed_measure_y, sz_processed_measure_pos, hipMemcpyDeviceToHost));

    // ASSERT_transition_frames(res_transition_body_frame, res_transition_world_frame, h_transition_body_frame, h_transition_world_frame, NUM_PARTICLES, false);
    // ASSERT_processed_measurements(res_processed_measure_x, res_processed_measure_y, processed_measure, NUM_PARTICLES, LIDAR_COORDS_LEN);

    /********************************************************************/
    /************************** CREATE 2D MAP ***************************/
    /********************************************************************/
    threadsPerBlock = 100;
    blocksPerGrid = NUM_PARTICLES;

    auto start_create_map = std::chrono::high_resolution_clock::now();

    kernel_create_2d_map << <blocksPerGrid, threadsPerBlock >> > (d_particles_x, d_particles_y, d_particles_idx, ELEMS_PARTICLES_START, d_map_2d, d_unique_in_particle,
        d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_create_map = std::chrono::high_resolution_clock::now();

    //hipError_t err = hipPeekAtLastError();
    //printf("%s\n", hipGetErrorString(err));

    gpuErrchk(hipMemcpy(res_map_2d, d_map_2d, sz_map_2d, hipMemcpyDeviceToHost));

    ASSERT_create_2d_map_elements(res_map_2d, negative_before_counter, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES, ELEMS_PARTICLES_START);

    /********************************************************************/
    /**************************** UPDATE MAP ****************************/
    /********************************************************************/
    auto start_update_map = std::chrono::high_resolution_clock::now();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;

    kernel_update_2d_map_with_measure << <blocksPerGrid, threadsPerBlock >> > (d_processed_measure_x, d_processed_measure_y, d_measure_idx,
        MEASURE_LEN, d_map_2d, d_unique_in_particle,
        d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_update_map = std::chrono::high_resolution_clock::now();

    /********************************************************************/
    /************************* CUMULATIVE SUM ***************************/
    /********************************************************************/
    auto start_cumulative_sum = std::chrono::high_resolution_clock::now();

    threadsPerBlock = UNIQUE_COUNTER_LEN;
    blocksPerGrid = 1;
    kernel_update_unique_sum << <1, 1 >> > (d_unique_in_particle, UNIQUE_COUNTER_LEN);
    kernel_update_unique_sum_col << <blocksPerGrid, threadsPerBlock >> > (d_unique_in_particle_col, GRID_WIDTH);
    hipDeviceSynchronize();

    auto stop_cumulative_sum = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(h_unique_in_particle, d_unique_in_particle, sz_unique_in_particle, hipMemcpyDeviceToHost));

    int NEW_LEN = h_unique_in_particle[UNIQUE_COUNTER_LEN - 1];
    int C_NEW_LEN = 0;
    // ASSERT_new_len_calculation(NEW_LEN, ELEMS_PARTICLES_AFTER, negative_after_counter);


    /*---------------------------------------------------------------------*/
    /*-------------------- REINITIALIZE MAP VARIABLES ---------------------*/
    /*---------------------------------------------------------------------*/
    /*---------------------------------------------------------------------*/
    gpuErrchk(hipFree(d_particles_x));
    gpuErrchk(hipFree(d_particles_y));
    gpuErrchk(hipFree(d_extended_idx));
    free(res_particles_x);
    free(res_particles_y);

    sz_particles_pos = NEW_LEN * sizeof(int);
    sz_extended_idx = NEW_LEN * sizeof(int);

    gpuErrchk(hipMalloc((void**)&d_particles_x, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_particles_y, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&d_extended_idx, sz_extended_idx));

    res_particles_x = (int*)malloc(sz_particles_pos);
    res_particles_y = (int*)malloc(sz_particles_pos);


    /********************************************************************/
    /************************ MAP RESTRUCTURE ***************************/
    /********************************************************************/
    threadsPerBlock = GRID_WIDTH;
    blocksPerGrid = NUM_PARTICLES;

    auto start_map_restructure = std::chrono::high_resolution_clock::now();

    hipMemset(d_particles_idx, 0, sz_particles_idx);
    kernel_update_unique_restructure << <blocksPerGrid, threadsPerBlock >> > (d_map_2d, d_particles_x, d_particles_y, d_particles_idx,
        d_unique_in_particle, d_unique_in_particle_col, GRID_WIDTH, GRID_HEIGHT);
    hipDeviceSynchronize();

    kernel_update_unique_sum << <1, 1 >> > (d_particles_idx, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_map_restructure = std::chrono::high_resolution_clock::now();

    auto start_copy_particles_pos = std::chrono::high_resolution_clock::now();
    gpuErrchk(hipMemcpy(res_particles_x, d_particles_x, sz_particles_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_y, d_particles_y, sz_particles_pos, hipMemcpyDeviceToHost));
    auto stop_copy_particles_pos = std::chrono::high_resolution_clock::now();

    // ASSERT_particles_pos_unique(res_particles_x, res_particles_y, h_particles_x_after_unique, h_particles_y_after_unique, NEW_LEN);

    /********************************************************************/
    /************************* INDEX EXPANSION **************************/
    /********************************************************************/
    auto start_index_expansion = std::chrono::high_resolution_clock::now();

    threadsPerBlock = 100;
    blocksPerGrid = NUM_PARTICLES;
    kernel_index_expansion << <blocksPerGrid, threadsPerBlock >> > (d_particles_idx, d_extended_idx, NEW_LEN);
    hipDeviceSynchronize();

    auto stop_index_expansion = std::chrono::high_resolution_clock::now();

    res_extended_idx = (int*)malloc(sz_extended_idx);
    gpuErrchk(hipMemcpy(res_extended_idx, d_extended_idx, sz_extended_idx, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_idx, d_particles_idx, sz_particles_idx, hipMemcpyDeviceToHost));


    /********************************************************************/
    /************************ KERNEL CORRELATION ************************/
    /********************************************************************/
    threadsPerBlock = 256;
    blocksPerGrid = (NEW_LEN + threadsPerBlock - 1) / threadsPerBlock;
    printf("*** CUDA kernel launch with %d blocks of %d threads, All Threads: %d ***\n", blocksPerGrid, threadsPerBlock, blocksPerGrid * threadsPerBlock);

    auto start_correlation = std::chrono::high_resolution_clock::now();

    kernel_correlation << <blocksPerGrid, threadsPerBlock >> > (d_grid_map, d_particles_x, d_particles_y, d_extended_idx, d_weights_raw, GRID_WIDTH, GRID_HEIGHT, NEW_LEN);
    hipDeviceSynchronize();


    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_correlation_max << <blocksPerGrid, threadsPerBlock >> > (d_weights_raw, d_weights, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_correlation = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_weights, d_weights, sz_weights, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_extended_idx, d_extended_idx, sz_extended_idx, hipMemcpyDeviceToHost));

    ASSERT_correlation_Equality(res_weights, pre_weights, NUM_PARTICLES);

    /********************************************************************/
    /********************** UPDATE WEIGHTS KERNEL ***********************/
    /********************************************************************/
    auto start_update_particle_weights = std::chrono::high_resolution_clock::now();

    threadsPerBlock = 1;
    blocksPerGrid = 1;

    kernel_arr_max << < blocksPerGrid, threadsPerBlock >> > (d_weights, d_weights_max, NUM_PARTICLES);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(res_weights_max, d_weights_max, sz_weights_max, hipMemcpyDeviceToHost));

    float norm_value = -res_weights_max[0] + 50;

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_arr_increase << < blocksPerGrid, threadsPerBlock >> > (d_weights, norm_value, 0);
    hipDeviceSynchronize();

    threadsPerBlock = 1;
    blocksPerGrid = 1;
    kernel_arr_sum_exp << < blocksPerGrid, threadsPerBlock >> > (d_weights, d_sum_exp, NUM_PARTICLES);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(res_sum_exp, d_sum_exp, sz_sum_exp, hipMemcpyDeviceToHost));

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_arr_normalize << < blocksPerGrid, threadsPerBlock >> > (d_weights, res_sum_exp[0]);
    hipDeviceSynchronize();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_arr_mult << < blocksPerGrid, threadsPerBlock >> > (d_particles_weight, d_weights);
    hipDeviceSynchronize();

    auto stop_update_particle_weights = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_weights, d_weights, sz_weights, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_weight, d_particles_weight, sz_particles_weight, hipMemcpyDeviceToHost));

    ASSERT_update_particle_weights(res_weights, new_weights, NUM_PARTICLES, false);
    ASSERT_update_particle_weights(res_particles_weight, particles_weight_post, NUM_PARTICLES, false);


    /********************************************************************/
    /************************ RESAMPLING KERNEL *************************/
    /********************************************************************/
    auto start_resampling = std::chrono::high_resolution_clock::now();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_resampling << <blocksPerGrid, threadsPerBlock >> > (d_weights, d_js, d_rnd, NUM_PARTICLES);
    hipDeviceSynchronize();

    auto stop_resampling = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_js, d_js, sz_js, hipMemcpyDeviceToHost));

    ASSERT_resampling_indices(res_js, js, NUM_PARTICLES, false);

    ASSERT_resampling_states(h_states_x, h_states_y, h_states_theta, h_states_x_updated, h_states_y_updated, h_states_theta_updated, res_js, NUM_PARTICLES, false);

    
    /*---------------------------------------------------------------------*/
    /*----------------- REINITIALIZE PARTICLES VARIABLES ------------------*/
    /*---------------------------------------------------------------------*/
    /*---------------------------------------------------------------------*/
    size_t sz_last_len = sizeof(int);
    int* d_last_len = NULL;
    int* res_last_len = (int*)malloc(sizeof(int));

    gpuErrchk(hipMalloc((void**)&d_last_len, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dc_particles_x, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&dc_particles_y, sz_particles_pos));
    gpuErrchk(hipMalloc((void**)&dc_particles_idx, sz_particles_idx));

    auto start_clone_particles = std::chrono::high_resolution_clock::now();
    gpuErrchk(hipMemcpy(dc_particles_x, d_particles_x, sz_particles_pos, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(dc_particles_y, d_particles_y, sz_particles_pos, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(dc_particles_idx, d_particles_idx, sz_particles_idx, hipMemcpyDeviceToDevice));

    gpuErrchk(hipMemcpy(dc_states_x, d_states_x, sz_states_pos, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(dc_states_y, d_states_y, sz_states_pos, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy(dc_states_theta, d_states_theta, sz_states_pos, hipMemcpyDeviceToDevice));
    auto stop_clone_particles = std::chrono::high_resolution_clock::now();

    threadsPerBlock = NUM_PARTICLES;
    blocksPerGrid = 1;
    kernel_rearrange_indecies << <blocksPerGrid, threadsPerBlock >> > (d_particles_idx, dc_particles_idx, d_js, d_last_len, NEW_LEN);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(res_particles_idx, d_particles_idx, sz_particles_idx, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_last_len, d_last_len, sz_last_len, hipMemcpyDeviceToHost));

    /********************************************************************/
    /********************** REARRANGEMENT KERNEL ************************/
    /********************************************************************/
    auto start_rearrange_index = std::chrono::high_resolution_clock::now();
    kernel_update_unique_sum << <1, 1 >> > (d_particles_idx, NUM_PARTICLES);
    hipDeviceSynchronize();
    auto stop_rearrange_index = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_particles_idx, d_particles_idx, sz_particles_idx, hipMemcpyDeviceToHost));
    C_NEW_LEN = NEW_LEN;
    NEW_LEN = res_particles_idx[NUM_PARTICLES - 1] + res_last_len[0];
    printf("--> NEW_LEN=%d <> ELEMS_PARTICLES_AFTER=%d\n", NEW_LEN, ELEMS_PARTICLES_AFTER);
    assert(NEW_LEN + negative_after_counter == ELEMS_PARTICLES_AFTER);

    free(res_particles_x);
    free(res_particles_y);
    sz_particles_pos = NEW_LEN * sizeof(int);
    res_particles_x = (int*)malloc(sz_particles_pos);
    res_particles_y = (int*)malloc(sz_particles_pos);

    ASSERT_resampling_particles_index(h_particles_idx_after_resampling, res_particles_idx, NUM_PARTICLES, false, negative_after_counter);

    auto start_rearrange_particles_states = std::chrono::high_resolution_clock::now();
    threadsPerBlock = 100;
    blocksPerGrid = NUM_PARTICLES;
    kernel_rearrange_particles << <blocksPerGrid, threadsPerBlock >> > (d_particles_x, d_particles_y, d_particles_idx,
        dc_particles_x, dc_particles_y, dc_particles_idx, d_js,
        GRID_WIDTH, GRID_HEIGHT, NUM_PARTICLES, NEW_LEN, C_NEW_LEN);

    kernel_rearrange_states << <blocksPerGrid, threadsPerBlock >> > (d_states_x, d_states_y, d_states_theta, 
        dc_states_x, dc_states_y, dc_states_theta, d_js);
    hipDeviceSynchronize();
    auto stop_rearrange_particles_states = std::chrono::high_resolution_clock::now();

    gpuErrchk(hipMemcpy(res_particles_x, d_particles_x, sz_particles_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_particles_y, d_particles_y, sz_particles_pos, hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(res_states_x, d_states_x, sz_states_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_states_y, d_states_y, sz_states_pos, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(res_states_theta, d_states_theta, sz_states_pos, hipMemcpyDeviceToHost));

    ASSERT_rearrange_particles_states(res_particles_x, res_particles_y, res_states_x, res_states_y, res_states_theta,
        h_particles_x_after_resampling, h_particles_y_after_resampling, h_states_x_updated, h_states_y_updated, h_states_theta_updated, 
        NEW_LEN, NUM_PARTICLES);


    /********************************************************************/
    /********************** REARRANGEMENT KERNEL ************************/
    /********************************************************************/
    auto start_update_states = std::chrono::high_resolution_clock::now();

    thrust::device_vector<float> d_vec_states_x(d_states_x, d_states_x + NUM_PARTICLES);
    thrust::device_vector<float> d_vec_states_y(d_states_y, d_states_y + NUM_PARTICLES);
    thrust::device_vector<float> d_vec_states_theta(d_states_theta, d_states_theta + NUM_PARTICLES);

    thrust::host_vector<float> h_vec_states_x(d_vec_states_x.begin(), d_vec_states_x.end());
    thrust::host_vector<float> h_vec_states_y(d_vec_states_y.begin(), d_vec_states_y.end());
    thrust::host_vector<float> h_vec_states_theta(d_vec_states_theta.begin(), d_vec_states_theta.end());

    std_vec_states_x.clear();
    std_vec_states_y.clear();
    std_vec_states_theta.clear();
    std_vec_states_x.resize(h_vec_states_x.size());
    std_vec_states_y.resize(h_vec_states_y.size());
    std_vec_states_theta.resize(h_vec_states_theta.size());

    std::copy(h_vec_states_x.begin(), h_vec_states_x.end(), std_vec_states_x.begin());
    std::copy(h_vec_states_y.begin(), h_vec_states_y.end(), std_vec_states_y.begin());
    std::copy(h_vec_states_theta.begin(), h_vec_states_theta.end(), std_vec_states_theta.begin());

    std::map<std::tuple<float, float, float>, int> states;

    for (int i = 0; i < NUM_PARTICLES; i++) {
        if (states.find(std::make_tuple(std_vec_states_x[i], std_vec_states_y[i], std_vec_states_theta[i])) == states.end())
            states.insert({ std::make_tuple(std_vec_states_x[i], std_vec_states_y[i], std_vec_states_theta[i]), 1 });
        else
            states[std::make_tuple(std_vec_states_x[i], std_vec_states_y[i], std_vec_states_theta[i])] += 1;
    }

    std::map<std::tuple<float, float, float>, int>::iterator best
        = std::max_element(states.begin(), states.end(), [](const std::pair<std::tuple<float, float, float>, int>& a,
            const std::pair<std::tuple<float, float, float>, int>& b)->bool { return a.second < b.second; });

    auto key = best->first;

    float theta = std::get<2>(key);
    float n_res_transition_body_frame[] = { cos(theta), -sin(theta), std::get<0>(key),
                        sin(theta),  cos(theta), std::get<1>(key),
                        0, 0, 1 };
    auto stop_update_states = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < 9; i++) {
        printf("%f  ", n_res_transition_body_frame[i]);
        // assert( abs(h_transition_body_frame[i] - n_res_transition_body_frame[i]) < 1e-2);
        if (abs(h_transition_body_frame[i] - n_res_transition_body_frame[i]) > 1e-4) {
            printf("(%f) ", h_transition_body_frame[i]);
        }
    }
    printf("\n");
    printf("%f, %f, %f\n", std::get<0>(key), std::get<1>(key), std::get<2>(key));


    /********************************************************************/
    /************************* EXECUTION TIMES **************************/
    /********************************************************************/
    auto duration_create_map = std::chrono::duration_cast<std::chrono::microseconds>(stop_create_map - start_create_map);
    auto duration_update_map = std::chrono::duration_cast<std::chrono::microseconds>(stop_update_map - start_update_map);
    auto duration_cumulative_sum = std::chrono::duration_cast<std::chrono::microseconds>(stop_cumulative_sum - start_cumulative_sum);
    auto duration_map_restructure = std::chrono::duration_cast<std::chrono::microseconds>(stop_map_restructure - start_map_restructure);
    auto duration_copy_particles_pos = std::chrono::duration_cast<std::chrono::microseconds>(stop_copy_particles_pos - start_copy_particles_pos);
    auto duration_transition_kernel = std::chrono::duration_cast<std::chrono::microseconds>(stop_transition_kernel - start_transition_kernel);
    auto duration_correlation = std::chrono::duration_cast<std::chrono::microseconds>(stop_correlation - start_correlation);
    auto duration_update_particle_weights = std::chrono::duration_cast<std::chrono::microseconds>(stop_update_particle_weights - start_update_particle_weights);
    auto duration_resampling = std::chrono::duration_cast<std::chrono::microseconds>(stop_resampling - start_resampling);
    auto duration_clone_particles = std::chrono::duration_cast<std::chrono::microseconds>(stop_clone_particles - start_clone_particles);
    auto duration_rearrange_particles_states = std::chrono::duration_cast<std::chrono::microseconds>(stop_rearrange_particles_states - start_rearrange_particles_states);
    auto duration_rearrange_index = std::chrono::duration_cast<std::chrono::microseconds>(stop_rearrange_index - start_rearrange_index);
    auto duration_update_states = std::chrono::duration_cast<std::chrono::microseconds>(stop_update_states - start_update_states);

    auto duration_sum = duration_create_map + duration_update_map + duration_cumulative_sum + duration_map_restructure + duration_copy_particles_pos +
        duration_transition_kernel + duration_correlation + duration_update_particle_weights + duration_resampling + duration_clone_particles + 
        duration_rearrange_particles_states + duration_rearrange_index + duration_update_states;

    std::cout << "Time taken by function (Create Map): " << duration_create_map.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Update Map): " << duration_update_map.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Cumulative Sum): " << duration_cumulative_sum.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Map Restructure): " << duration_map_restructure.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Copy Particles): " << duration_copy_particles_pos.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Transition Kernel): " << duration_transition_kernel.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Correlation Kernel): " << duration_correlation.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Update Particle Weights): " << duration_update_particle_weights.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Kernel Resampling): " << duration_resampling.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Clone Particles): " << duration_clone_particles.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Rearrange Particles States): " << duration_rearrange_particles_states.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Rearrange Index): " << duration_rearrange_index.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Update States): " << duration_update_states.count() << " microseconds" << std::endl;
    std::cout << "Time taken by function (Sum): " << duration_sum.count() << " microseconds" << std::endl;

    printf("\nFinished All\n");

}
#endif

/*
* Kernel Functions
*/

__global__ void kernel_bresenham_rearrange(int* particles_free_x, int* particles_free_y, int* particles_free_x_max, int* particles_free_y_max, 
   int* particles_free_idx, const int MAX_DIST_IN_MAP, const int NUM_ELEMS) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < NUM_ELEMS) {

        int start_idx = particles_free_idx[i];
        int end_idx = particles_free_idx[i + 1];
        int curr_particles_len = end_idx - start_idx;
        int start_idx_max = i * MAX_DIST_IN_MAP;

        for (int j = 0; j < curr_particles_len; j++) {

            particles_free_x[start_idx + j] = particles_free_x_max[start_idx_max + j];
            particles_free_y[start_idx + j] = particles_free_y_max[start_idx_max + j];
        }
    }
}

__global__ void kernel_bresenham(const int* particles_occupied_x, const int* particles_occupied_y, const int* position_image_body, 
    int* particles_free_x, int* particles_free_y, int* particles_free_counter, const int PARTICLES_LEN, const int MAX_DIST_IN_MAP) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < PARTICLES_LEN) {

        int pointsCounter = 0;
        int x = particles_occupied_x[i];
        int y = particles_occupied_y[i];
        int x1 = x;
        int y1 = y;
        int position_image_body_x = position_image_body[0];
        int position_image_body_y = position_image_body[1];
        int x2 = position_image_body_x;
        int y2 = position_image_body_y;

        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);

        int start_idx = i * MAX_DIST_IN_MAP;

        if (dx == 0) {

            int sign = (y2 - y1) > 0 ? 1 : -1;
            particles_free_x[start_idx] = x;
            particles_free_y[start_idx] = y;
            pointsCounter += 1;

            for (int j = 1; j <= dy; j++) {
                particles_free_x[start_idx + j] = x;
                particles_free_y[start_idx + j] = y + sign * j;
                pointsCounter += 1;
            }
        }
        else {

            float gradient = dy / float(dx);
            bool should_reverse = false;

            if (gradient > 1) {

                swap(dx, dy);
                swap(x, y);
                swap(x1, y1);
                swap(x2, y2);
                should_reverse = true;
            }

            int p = 2 * dy - dx;
            if (should_reverse == false) {
                particles_free_x[start_idx] = x;
                particles_free_y[start_idx] = y;
                pointsCounter += 1;
            }
            else {
                particles_free_x[start_idx] = y;
                particles_free_y[start_idx] = x;
                pointsCounter += 1;
            }

            for (int j = 1; j <= dx; j++) {

                if (p > 0) {
                    y = (y < y2) ? y + 1 : y - 1;
                    p = p + 2 * (dy - dx);
                }
                else {
                    p = p + 2 * dy;
                }

                x = (x < x2) ? x + 1 : x - 1;

                if (should_reverse == false) {
                    particles_free_x[start_idx + j] = x;
                    particles_free_y[start_idx + j] = y;
                    pointsCounter += 1;
                }
                else {
                    particles_free_x[start_idx + j] = y;
                    particles_free_y[start_idx + j] = x;
                    pointsCounter += 1;
                }
            }
        }
        particles_free_counter[i] = pointsCounter;
    }
}


__global__ void kernel_bresenham(const int* particles_occupied_x, const int* particles_occupied_y, 
    const int* position_image_body, int* particles_free_x, int* particles_free_y, const int* particles_free_idx, const int PARTICLES_LEN) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < PARTICLES_LEN) {

        int x = particles_occupied_x[i];
        int y = particles_occupied_y[i];
        int x1 = x;
        int y1 = y;
        int position_image_body_x = position_image_body[0];
        int position_image_body_y = position_image_body[1];
        int x2 = position_image_body_x;
        int y2 = position_image_body_y;

        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);

        int start_index = particles_free_idx[i];

        if (dx == 0) {
        
            int sign = (y2 - y1) > 0 ? 1 : -1;
            particles_free_x[start_index] = x;
            particles_free_y[start_index] = y;

            for (int j = 1; j <= dy; j++) {
                particles_free_x[start_index + j] = x;
                particles_free_y[start_index + j] = y + sign * j;
            }
        }
        else {

            float gradient = dy / float(dx);
            bool should_reverse = false;

            if (gradient > 1) {

                swap(dx, dy);
                swap(x, y);
                swap(x1, y1);
                swap(x2, y2);
                should_reverse = true;
            }

            int p = 2 * dy - dx;
            if (should_reverse == false) {
                particles_free_x[start_index] = x;
                particles_free_y[start_index] = y;
            }
            else {
                particles_free_x[start_index] = y;
                particles_free_y[start_index] = x;
            }

            for (int j = 1; j <= dx; j++) {

                if (p > 0) {
                    y = (y < y2) ? y + 1 : y - 1;
                    p = p + 2 * (dy - dx);
                }
                else {
                    p = p + 2 * dy;
                }

                x = (x < x2) ? x + 1 : x - 1;

                if (should_reverse == false) {
                    particles_free_x[start_index + j] = x;
                    particles_free_y[start_index + j] = y;
                }
                else {
                    particles_free_x[start_index + j] = y;
                    particles_free_y[start_index + j] = x;
                }
            }
        }
    }
}

__global__ void kernel_index_init_const(int* indices, const int value) {
    
    int i = threadIdx.x;
    if (i > 0) {
        indices[i] = value;
    }
}

__global__ void kernel_index_expansion(const int *idx, int *extended_idx, const int numElements) {

    int i = blockIdx.x;
    int k = threadIdx.x;
    const int numThreads = blockDim.x;

    if (i < numThreads) {

        int first_idx = idx[i];
        int last_idx = (i < numThreads - 1) ? idx[i + 1] : numElements;
        int arr_len = last_idx - first_idx;

        int arr_end = last_idx;

        int start_idx = ((arr_len / blockDim.x) * k) + first_idx;
        int end_idx = ((arr_len / blockDim.x) * (k + 1)) + first_idx;
        end_idx = (k < blockDim.x - 1) ? end_idx : arr_end;

        for (int j = start_idx; j < end_idx; j++)
            extended_idx[j] = i;
    }
}

__global__ void kernel_correlation_max(const float* weights_raw, float* weights, const int _NUM_PARTICLES) {

    int i = threadIdx.x;

    float curr_max_value = weights_raw[i];
    for (int j = 0; j < 25; j++) {
        float curr_value = weights_raw[j * _NUM_PARTICLES + i];
        if (curr_value > curr_max_value) {
            curr_max_value = curr_value;
        }
    }
    weights[i] = curr_max_value;
}

__global__ void kernel_correlation(const int* grid_map, const int* states_x, const int* states_y,
                                    const int* states_idx, float* weights, const int _GRID_WIDTH, const int _GRID_HEIGHT, int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        // int start_current_map_idx = i * _GRID_WIDTH * _GRID_HEIGHT;
        int loop_counter = 0;
        for (int x_offset = -2; x_offset <= 2; x_offset++) {

            for (int y_offset = -2; y_offset <= 2; y_offset++) {

                int idx = states_idx[i];
                int x = states_x[i] + x_offset;
                int y = states_y[i] + y_offset;

                if (x >= 0 && y >= 0 && x < _GRID_WIDTH && y < _GRID_HEIGHT) {

                    int curr_idx = x * _GRID_HEIGHT + y;
                    // int curr_idx = start_current_map_idx + (x * _GRID_HEIGHT) + y;
                    float value = grid_map[curr_idx];

                    if (value != 0)
                        atomicAdd(&weights[loop_counter * 100 + idx], value);
                }
                loop_counter++;
            }
        }
    }
}

__global__ void kernel_update_map(int* grid_map, const float* log_odds, const float _LOG_ODD_PRIOR, const int _WALL, const int _FREE, const int numElements) {


    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        if (log_odds[i] > 0)
            grid_map[i] = _WALL;

        if (log_odds[i] < _LOG_ODD_PRIOR)
            grid_map[i] = _FREE;
    }
}


__global__ void kernel_update_log_odds(float* log_odds, int* f_x, int* f_y, const float _log_t,
    const int _GRID_WIDTH, const int _GRID_HEIGHT, const int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        int x = f_x[i];
        int y = f_y[i];

        if (x >= 0 && y >= 0 && x < _GRID_WIDTH && y < _GRID_HEIGHT) {

            int grid_map_idx = x * _GRID_HEIGHT + y;

            log_odds[grid_map_idx] = log_odds[grid_map_idx] + _log_t;
        }
    }
}

__global__ void kernel_resampling(const float* weights, int* js, const float* rnd, const int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        float u = rnd[i] / numElements;
        int j = 0;
        float beta = u + float(i) / numElements;

        float accum = 0;
        for (int idx = 0; idx <= i; idx++) {
            accum += weights[idx];

            while (beta > accum) {
                j += 1;
                accum += weights[j];
            }
        }
        js[i] = j;
    }
}

__global__ void kernel_update_particles_states(const float* states_x, const float* states_y, const float* states_theta,
                                        float* transition_body_frame, const float* transition_lidar_frame, float* transition_world_frame, const int numElements) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {

        int T_idx = i * 9;

        float p_wb_0 = states_x[i];
        float p_wb_1 = states_y[i];

        float R_wb_0 =  cos(states_theta[i]);
        float R_wb_1 = -sin(states_theta[i]);
        float R_wb_2 =  sin(states_theta[i]);
        float R_wb_3 =  cos(states_theta[i]);

        transition_body_frame[T_idx + 0] = R_wb_0;   transition_body_frame[T_idx + 1] = R_wb_1;   transition_body_frame[T_idx + 2] = p_wb_0;
        transition_body_frame[T_idx + 3] = R_wb_2;   transition_body_frame[T_idx + 4] = R_wb_3;   transition_body_frame[T_idx + 5] = p_wb_1;
        transition_body_frame[T_idx + 6] = 0;        transition_body_frame[T_idx + 7] = 0;        transition_body_frame[T_idx + 8] = 1;

        kernel_matrix_mul_3x3(transition_body_frame, transition_lidar_frame, transition_world_frame, T_idx);
    }
}

__global__ void kernel_update_particles_lidar(float* transition_world_frame, int* processed_measure_x, int* processed_measure_y,
    float* particles_wframe_x, float* particles_wframe_y, const float* _lidar_coords, float _res, int _xmin, int _ymax, const int _LIDAR_COORDS_LEN) {

    int k = blockIdx.x;

    for (int j = 0; j < 2; j++) {

        double currVal = 0;
        currVal += transition_world_frame[j * 3 + 0] * _lidar_coords[(0 * _LIDAR_COORDS_LEN) + k];
        currVal += transition_world_frame[j * 3 + 1] * _lidar_coords[(1 * _LIDAR_COORDS_LEN) + k];
        currVal += transition_world_frame[j * 3 + 2];

        if (j == 0) {
            particles_wframe_x[k] = currVal;
            processed_measure_y[k] = (int)ceil((currVal - _xmin) / _res);
        }
        else {
            particles_wframe_y[k] = currVal; 
            processed_measure_x[k] = (int)ceil((_ymax - currVal) / _res);
        }
    }
}

__global__ void kernel_update_particles_lidar(float* transition_world_frame, int* processed_measure_x, int* processed_measure_y, const float* _lidar_coords, float _res, int _xmin, int _ymax,
                                                const int _lidar_coords_LEN, const int numElements) {

    int T_idx = threadIdx.x * 9;
    // int wo_idx = 2 * _lidar_coords_LEN * threadIdx.x;
    int wo_idx = _lidar_coords_LEN * threadIdx.x;
    int k = blockIdx.x;

    for (int j = 0; j < 2; j++) {

        double currVal = 0;
        currVal += transition_world_frame[T_idx + j * 3 + 0] * _lidar_coords[(0 * _lidar_coords_LEN) + k];
        currVal += transition_world_frame[T_idx + j * 3 + 1] * _lidar_coords[(1 * _lidar_coords_LEN) + k];
        currVal += transition_world_frame[T_idx + j * 3 + 2];

        // _Y_wo[wo_idx + (j * _lidar_coords_LEN) + k] = currVal; // ceil((currVal - _xmin) / _res);

        if (j == 0) {
            // processed_measure_y[wo_idx + (1 * _lidar_coords_LEN) + k] = ceil((currVal - _xmin) / _res);
            processed_measure_y[wo_idx + k] = (int) ceil((currVal - _xmin) / _res);
        }
        else {
            // processed_measure_x[wo_idx + (0 * _lidar_coords_LEN) + k] = ceil((_ymax - currVal) / _res);
            processed_measure_x[wo_idx + k] = (int) ceil((_ymax - currVal) / _res);
        }
    }
}

__device__ void kernel_matrix_mul_3x3(const float* A, const float* B, float* C, int start_i) {

    // A[i, j] --> A[i*3 + j]

    for (int i = 0; i < 3; i++) {

        for (int j = 0; j < 3; j++) {

            float currVal = 0;
            for (int k = 0; k < 3; k++) {
                currVal += A[start_i + (i * 3) + k] * B[k * 3 + j];
            }
            C[start_i + (i * 3) + j] = currVal;
        }
    }
}

__global__ void kernel_matrix_mul_3x3(const float* A, const float* B, float* C) {

    for (int i = 0; i < 3; i++) {

        for (int j = 0; j < 3; j++) {

            float currVal = 0;
            for (int k = 0; k < 3; k++) {
                currVal += A[(i * 3) + k] * B[k * 3 + j];
            }
            C[(i * 3) + j] = currVal;
        }
    }
}

__global__ void kernel_2d_map_counter(uint8_t* map_2d, int* unique_counter, int* unique_counter_col, const int _GRID_WIDHT, const int _GRID_HEIGHT) {

    int i = blockIdx.x;
    int k = threadIdx.x;

    int start_idx = k * _GRID_HEIGHT;
    int end_idx = (k + 1) * _GRID_HEIGHT;

    for (int j = start_idx; j < end_idx; j++) {

        if (map_2d[j] != 0) {
            atomicAdd(&unique_counter[i], 1);
            atomicAdd(&unique_counter_col[k+1], 1);
        }
    }
}

__global__ void kernel_create_2d_map(const int* particles_x, const int* particles_y, const int* particles_idx, const int IDX_LEN, uint8_t* map_2d,
    const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS) {

    int i = blockIdx.x;
    int k = threadIdx.x;

    if (i < _NUM_ELEMS) {

        int first_idx = particles_idx[i];
        int last_idx = (i < _NUM_ELEMS - 1) ? particles_idx[i + 1] : IDX_LEN;
        int arr_len = last_idx - first_idx;

        int arr_end = last_idx;

        int start_idx = ((arr_len / blockDim.x) * k) + first_idx;
        int end_idx = ((arr_len / blockDim.x) * (k + 1)) + first_idx;
        end_idx = (k < blockDim.x - 1) ? end_idx : arr_end;

        int start_of_current_map = i * _GRID_WIDTH * _GRID_HEIGHT;

        for (int j = start_idx; j < end_idx; j++) {

            int x = particles_x[j];
            int y = particles_y[j];

            int curr_idx = start_of_current_map + (x * _GRID_HEIGHT) + y;

            if (x >= 0 && y >= 0 && map_2d[curr_idx] == 0) {
                map_2d[curr_idx] = 1;
            }
        }
    }
}

__global__ void kernel_create_2d_map(const int *particles_x, const int *particles_y, const int *particles_idx, const int IDX_LEN, uint8_t *map_2d,
                                        int *unique_in_particle, int *unique_in_particle_col, const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS) {

    int i = blockIdx.x;
    int k = threadIdx.x;

    if (i < _NUM_ELEMS) {

        int first_idx = particles_idx[i];
        int last_idx = (i < _NUM_ELEMS - 1) ? particles_idx[i + 1] : IDX_LEN;
        int arr_len = last_idx - first_idx;

        int arr_end = last_idx;

        int start_idx   = ((arr_len / blockDim.x) * k) + first_idx;
        int end_idx     = ((arr_len / blockDim.x) * (k + 1)) + first_idx;
        end_idx = (k < blockDim.x - 1) ? end_idx : arr_end;

        int start_of_current_map = i * _GRID_WIDTH * _GRID_HEIGHT;
        int start_of_col = i * _GRID_WIDTH;


        for (int j = start_idx; j < end_idx; j++) {

            int x = particles_x[j];
            int y = particles_y[j];

            int curr_idx = start_of_current_map + (x * _GRID_HEIGHT) + y;

            if (x >= 0 && y >= 0 && map_2d[curr_idx] == 0) {
                map_2d[curr_idx] = 1;
                atomicAdd(&unique_in_particle[i + 1], 1);
                atomicAdd(&unique_in_particle_col[start_of_col + x + 1], 1);
                // unique_in_particle_col[start_of_col + x + 1] = unique_in_particle_col[start_of_col + x + 1] + 1;
            }
        }
    }    
}

__global__ void kernel_update_2d_map_with_measure(const int* measure_x, const int* measure_y, const int* measure_idx, const int IDX_LEN, uint8_t* map_2d,
                    int* unique_in_particle, int* unique_in_particle_col, const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS) {

    int i = threadIdx.x;

    if (i < _NUM_ELEMS) {

        int start_idx = measure_idx[i];
        int end_idx = (i < _NUM_ELEMS - 1) ? measure_idx[i + 1] : IDX_LEN;

        int start_of_current_map = i * _GRID_WIDTH * _GRID_HEIGHT;
        int start_of_col = i * _GRID_WIDTH;

        for (int j = start_idx; j < end_idx; j++) {

            int x = measure_x[j];
            int y = measure_y[j];

            int curr_idx = start_of_current_map + (x * _GRID_HEIGHT) + y;

            if (x >= 0 && y >= 0 && map_2d[curr_idx] == 0) {
                map_2d[curr_idx] = 1;
                atomicAdd(&unique_in_particle[i + 1], 1);
                atomicAdd(&unique_in_particle_col[start_of_col + x + 1], 1);
                // unique_in_particle_col[start_of_col + x + 1] = unique_in_particle_col[start_of_col + x + 1] + 1;
            }
        }
    }
}

__global__ void kernel_update_unique_restructure2(uint8_t* map_2d, int* particles_x, int* particles_y, int* unique_in_particle, int* unique_in_particle_col,
    const int _GRID_WIDTH, const int _GRID_HEIGHT) {

    int i = threadIdx.x;

    int start_idx = (i * _GRID_HEIGHT);
    int end_idx = ((i + 1) * _GRID_HEIGHT);
    int key = unique_in_particle_col[i];
    int first_key = key;

    for (int j = start_idx; j < end_idx; j++) {

        if (map_2d[j] == 1) {

            int y = j % _GRID_HEIGHT;
            int x = j / _GRID_HEIGHT;

            particles_x[key] = x;
            particles_y[key] = y;
            key += 1;
        }
    }
}

__global__ void kernel_update_unique_restructure(uint8_t* map_2d, int* particles_x, int* particles_y, int* particles_idx, int* unique_in_particle, int* unique_in_particle_col,
                                                    const int _GRID_WIDTH, const int _GRID_HEIGHT) {

    int i = blockIdx.x;
    int l = threadIdx.x;

    int start_of_current_map    =  i * _GRID_WIDTH * _GRID_HEIGHT;
    int start_idx               = (i * _GRID_WIDTH * _GRID_HEIGHT) + (l * _GRID_HEIGHT);
    int end_idx                 = (i * _GRID_WIDTH * _GRID_HEIGHT) + ((l + 1) * _GRID_HEIGHT);
    int key                     = unique_in_particle_col[i * _GRID_WIDTH + l] + unique_in_particle[i];

    printf("key=%d\n", key);

    for (int j = start_idx; j < end_idx; j++) {

        if (map_2d[j] == 1) {

            int y = (j - start_of_current_map) % _GRID_HEIGHT;
            int x = (j - start_of_current_map) / _GRID_HEIGHT;

            particles_x[key] = x;
            particles_y[key] = y;
            key += 1;
            atomicAdd(&particles_idx[i + 1], 1);
        }
    }
}

__global__ void kernel_rearrange_particles(int* particles_x, int* particles_y, const int* particles_idx, 
    const int* c_particles_x, const int* c_particles_y, const int* c_particles_idx, const int* js,
    const int _GRID_WIDTH, const int _GRID_HEIGHT, const int _NUM_ELEMS, const int IDX_LEN, const int C_IDX_LEN) {

    int i = blockIdx.x;
    int k = threadIdx.x;
    int m = js[i];

    if (i < _NUM_ELEMS) {

        int first_idx = particles_idx[i];
        int last_idx = (i < _NUM_ELEMS - 1) ? particles_idx[i + 1] : IDX_LEN;
        int arr_end = last_idx;
        int arr_len = last_idx - first_idx;

        int c_first_idx = c_particles_idx[m];
        int c_last_idx = (m < _NUM_ELEMS - 1) ? c_particles_idx[m + 1] : C_IDX_LEN;
        int c_arr_end = c_last_idx;
        int c_arr_len = c_last_idx - c_first_idx;

        //if (arr_len != c_arr_len)
        //    printf("%d <> %d | i=%d, k=%d, m=%d\n", arr_len, c_arr_len, i, k, m);

        int start_idx = ((arr_len / blockDim.x) * k) + first_idx;
        int end_idx = ((arr_len / blockDim.x) * (k + 1)) + first_idx;
        end_idx = (k < blockDim.x - 1) ? end_idx : arr_end;

        int c_start_idx = ((c_arr_len / blockDim.x) * k) + c_first_idx;
        int c_end_idx = ((c_arr_len / blockDim.x) * (k + 1)) + c_first_idx;
        c_end_idx = (k < blockDim.x - 1) ? c_end_idx : c_arr_end;

        for (int j = start_idx, c_j = c_start_idx; j < end_idx && c_j < c_end_idx; j++, c_j++) {
            particles_x[j] = c_particles_x[c_j];
            particles_y[j] = c_particles_y[c_j];
        }
    }
}

__global__ void kernel_rearrange_states(float* states_x, float* states_y, float* states_theta, 
    float* c_states_x, float* c_states_y, float* c_states_theta, int* js) {

    int i = threadIdx.x;
    int j = js[i];

    states_x[i] = c_states_x[j];
    states_y[i] = c_states_y[j];
    states_theta[i] = c_states_theta[j];
}

__global__ void kernel_position_to_image(int* position_image_body, float* transition_world_lidar, float _res, int _xmin, int _ymax) {

    float a = transition_world_lidar[2];
    float b = transition_world_lidar[5];

    position_image_body[0] = (int)ceil((_ymax - b) / _res);
    position_image_body[1] = (int)ceil((a - _xmin) / _res);
}

__global__ void kernel_rearrange_indecies(int* particles_idx, int* c_particles_idx, int* js, int* last_len, const int ARR_LEN) {

    int i = threadIdx.x;
    int j = js[i];
    int idx_value = 0;

    if (j == blockDim.x - 1)
        idx_value = ARR_LEN - c_particles_idx[j];
    else 
        idx_value = c_particles_idx[j + 1] - c_particles_idx[j];

    if (i == 0)
        particles_idx[i] = 0;
     
    if (i < blockDim.x - 1)
        particles_idx[i+1] = idx_value;
    else
       last_len[0] = idx_value;
}

__global__ void kernel_arr_increase(int* arr, const int increase_value, const int start_index) {

    int i = threadIdx.x;
    if (i >= start_index) {
        arr[i] += increase_value;
    }
}

__global__ void kernel_arr_increase(float* arr, const float increase_value, const int start_index) {

    int i = threadIdx.x;
    if (i >= start_index) {
        arr[i] += increase_value;
    }
}

__global__ void kernel_arr_mult(float* arr, const float mult_value) {

    int i = threadIdx.x;
    arr[i] = arr[i] * mult_value;
}

__global__ void kernel_arr_mult(float* arr, float* mult_arr) {

    int i = threadIdx.x;
    arr[i] = arr[i] * mult_arr[i];
}

__global__ void kernel_arr_max(float* arr, float* result, const int LEN) {

    float rs = arr[0];
    for (int i = 1; i < LEN; i++) {
        if (rs < arr[i])
            rs = arr[i];
    }
    result[0] = rs;
}

__global__ void kernel_arr_sum_exp(float* arr, double* result, const int LEN) {

    double s = 0;
    for (int i = 0; i < LEN; i++) {
        s += exp(arr[i]);
    }
    result[0] = s;
}

__global__ void kernel_arr_normalize(float* arr, const double norm) {

    int i = threadIdx.x;
    arr[i] = exp(arr[i]) / norm;
}

__global__ void kernel_update_unique_sum(int* unique_in_particle, const int _NUM_ELEMS) {

    for (int j = 1; j < _NUM_ELEMS; j++)
        unique_in_particle[j] = unique_in_particle[j] + unique_in_particle[j - 1];
}

__global__ void kernel_update_unique_sum_col(int * unique_in_particle_col, const int _GRID_WIDTH) {

    int i = threadIdx.x;

    for (int j = (i * _GRID_WIDTH) + 1; j < (i + 1) * _GRID_WIDTH; j++)
        unique_in_particle_col[j] = unique_in_particle_col[j] + unique_in_particle_col[j - 1];
}