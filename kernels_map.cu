#include "hip/hip_runtime.h"

#include "kernels_map.cuh"

__global__ void kernel_bresenham_rearrange(int* particles_free_x, int* particles_free_y, int* particles_free_x_max, int* particles_free_y_max,
    int* particles_free_idx, const int MAX_DIST_IN_MAP, const int NUM_ELEMS) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < NUM_ELEMS) {

        int start_idx = particles_free_idx[i];
        int end_idx = particles_free_idx[i + 1];
        int curr_particles_len = end_idx - start_idx;
        int start_idx_max = i * MAX_DIST_IN_MAP;

        for (int j = 0; j < curr_particles_len; j++) {

            particles_free_x[start_idx + j] = particles_free_x_max[start_idx_max + j];
            particles_free_y[start_idx + j] = particles_free_y_max[start_idx_max + j];
        }
    }
}

__global__ void kernel_bresenham(const int* particles_occupied_x, const int* particles_occupied_y, const int* position_image_body,
    int* particles_free_x, int* particles_free_y, int* particles_free_counter, const int PARTICLES_LEN, const int MAX_DIST_IN_MAP) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < PARTICLES_LEN) {

        int pointsCounter = 0;
        int x = particles_occupied_x[i];
        int y = particles_occupied_y[i];
        int x1 = x;
        int y1 = y;
        int position_image_body_x = position_image_body[0];
        int position_image_body_y = position_image_body[1];
        int x2 = position_image_body_x;
        int y2 = position_image_body_y;

        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);

        int start_idx = i * MAX_DIST_IN_MAP;

        if (dx == 0) {

            int sign = (y2 - y1) > 0 ? 1 : -1;
            particles_free_x[start_idx] = x;
            particles_free_y[start_idx] = y;
            pointsCounter += 1;

            for (int j = 1; j <= dy; j++) {
                particles_free_x[start_idx + j] = x;
                particles_free_y[start_idx + j] = y + sign * j;
                pointsCounter += 1;
            }
        }
        else {

            float gradient = dy / float(dx);
            bool should_reverse = false;

            if (gradient > 1) {

                swap(dx, dy);
                swap(x, y);
                swap(x1, y1);
                swap(x2, y2);
                should_reverse = true;
            }

            int p = 2 * dy - dx;
            if (should_reverse == false) {
                particles_free_x[start_idx] = x;
                particles_free_y[start_idx] = y;
                pointsCounter += 1;
            }
            else {
                particles_free_x[start_idx] = y;
                particles_free_y[start_idx] = x;
                pointsCounter += 1;
            }

            for (int j = 1; j <= dx; j++) {

                if (p > 0) {
                    y = (y < y2) ? y + 1 : y - 1;
                    p = p + 2 * (dy - dx);
                }
                else {
                    p = p + 2 * dy;
                }

                x = (x < x2) ? x + 1 : x - 1;

                if (should_reverse == false) {
                    particles_free_x[start_idx + j] = x;
                    particles_free_y[start_idx + j] = y;
                    pointsCounter += 1;
                }
                else {
                    particles_free_x[start_idx + j] = y;
                    particles_free_y[start_idx + j] = x;
                    pointsCounter += 1;
                }
            }
        }
        particles_free_counter[i] = pointsCounter;
    }
}

__global__ void kernel_bresenham(const int* particles_occupied_x, const int* particles_occupied_y,
    const int* position_image_body, int* particles_free_x, int* particles_free_y, const int* particles_free_idx, const int PARTICLES_LEN) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < PARTICLES_LEN) {

        int x = particles_occupied_x[i];
        int y = particles_occupied_y[i];
        int x1 = x;
        int y1 = y;
        int position_image_body_x = position_image_body[0];
        int position_image_body_y = position_image_body[1];
        int x2 = position_image_body_x;
        int y2 = position_image_body_y;

        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);

        int start_index = particles_free_idx[i];

        if (dx == 0) {

            int sign = (y2 - y1) > 0 ? 1 : -1;
            particles_free_x[start_index] = x;
            particles_free_y[start_index] = y;

            for (int j = 1; j <= dy; j++) {
                particles_free_x[start_index + j] = x;
                particles_free_y[start_index + j] = y + sign * j;
            }
        }
        else {

            float gradient = dy / float(dx);
            bool should_reverse = false;

            if (gradient > 1) {

                swap(dx, dy);
                swap(x, y);
                swap(x1, y1);
                swap(x2, y2);
                should_reverse = true;
            }

            int p = 2 * dy - dx;
            if (should_reverse == false) {
                particles_free_x[start_index] = x;
                particles_free_y[start_index] = y;
            }
            else {
                particles_free_x[start_index] = y;
                particles_free_y[start_index] = x;
            }

            for (int j = 1; j <= dx; j++) {

                if (p > 0) {
                    y = (y < y2) ? y + 1 : y - 1;
                    p = p + 2 * (dy - dx);
                }
                else {
                    p = p + 2 * dy;
                }

                x = (x < x2) ? x + 1 : x - 1;

                if (should_reverse == false) {
                    particles_free_x[start_index + j] = x;
                    particles_free_y[start_index + j] = y;
                }
                else {
                    particles_free_x[start_index + j] = y;
                    particles_free_y[start_index + j] = x;
                }
            }
        }
    }
}

__global__ void kernel_2d_map_counter(uint8_t* map_2d, int* unique_counter, int* unique_counter_col, const int GRID_WIDHT, const int GRID_HEIGHT) {

    int i = blockIdx.x;
    int k = threadIdx.x;

    int start_idx = k * GRID_HEIGHT;
    int end_idx = (k + 1) * GRID_HEIGHT;

    for (int j = start_idx; j < end_idx; j++) {

        if (map_2d[j] != 0) {
            atomicAdd(&unique_counter[i], 1);
            atomicAdd(&unique_counter_col[k + 1], 1);
        }
    }
}

__global__ void kernel_update_log_odds(float* log_odds, int* f_x, int* f_y, const float _log_t,
    const int GRID_WIDTH, const int GRID_HEIGHT, const int NUM_ELEMS) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < NUM_ELEMS) {

        int x = f_x[i];
        int y = f_y[i];

        if (x >= 0 && y >= 0 && x < GRID_WIDTH && y < GRID_HEIGHT) {

            int grid_map_idx = x * GRID_HEIGHT + y;

            log_odds[grid_map_idx] = log_odds[grid_map_idx] + _log_t;
        }
    }
}

__global__ void kernel_position_to_image(int* position_image_body,
    const float transition_world_lidar_x, const float transition_world_lidar_y,
    const float res, const int xmin, const int ymax) {

    position_image_body[0] = (int)ceil((ymax - transition_world_lidar_y) / res);
    position_image_body[1] = (int)ceil((transition_world_lidar_x - xmin) / res);
}

__global__ void kernel_position_to_image(int* position_image_body,
    const float transition_world_lidar_x, const float transition_world_lidar_y,
    const float res, const float xmin, const float ymax) {

    position_image_body[0] = (int)ceil((ymax - transition_world_lidar_y) / res);
    position_image_body[1] = (int)ceil((transition_world_lidar_x - xmin) / res);
}

__global__ void kernel_position_to_image(int* position_image_body, float* transition_world_lidar, float res, int xmin, int ymax) {

    float a = transition_world_lidar[2];
    float b = transition_world_lidar[5];

    position_image_body[0] = (int)ceil((ymax - b) / res);
    position_image_body[1] = (int)ceil((a - xmin) / res);
}

__global__ void kernel_update_map(int* grid_map, const float* log_odds, const float _LOG_ODD_PRIOR, const int _WALL, const int _FREE, const int NUM_ELEMS) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < NUM_ELEMS) {

        if (log_odds[i] > 0)
            grid_map[i] = _WALL;

        if (log_odds[i] < _LOG_ODD_PRIOR)
            grid_map[i] = _FREE;
    }
}

__global__ void kernel_update_particles_lidar(float* transition_world_lidar, int* processed_measure_x, int* processed_measure_y,
    float* particles_world_frame_x, float* particles_world_frame_y, const float* _lidar_coords, float _res, int _xmin, int _ymax, const int LIDAR_COORDS_LEN) {

    int k = blockIdx.x;

    for (int j = 0; j < 2; j++) {

        double currVal = 0;
        currVal += transition_world_lidar[j * 3 + 0] * _lidar_coords[(0 * LIDAR_COORDS_LEN) + k];
        currVal += transition_world_lidar[j * 3 + 1] * _lidar_coords[(1 * LIDAR_COORDS_LEN) + k];
        currVal += transition_world_lidar[j * 3 + 2];

        if (j == 0) {
            particles_world_frame_x[k] = currVal;
            processed_measure_y[k] = (int)ceil((currVal - _xmin) / _res);
        }
        else {
            particles_world_frame_y[k] = currVal;
            processed_measure_x[k] = (int)ceil((_ymax - currVal) / _res);
        }
    }
}

__global__ void kernel_create_2d_map(const int* particles_x, const int* particles_y, const int* particles_idx, const int IDX_LEN, uint8_t* map_2d,
    const int GRID_WIDTH, const int GRID_HEIGHT, const int NUM_ELEMS) {

    int i = blockIdx.x;
    int k = threadIdx.x;

    if (i < NUM_ELEMS) {

        int first_idx = particles_idx[i];
        int last_idx = (i < NUM_ELEMS - 1) ? particles_idx[i + 1] : IDX_LEN;
        int arr_len = last_idx - first_idx;

        int arr_end = last_idx;

        int start_idx = ((arr_len / blockDim.x) * k) + first_idx;
        int end_idx = ((arr_len / blockDim.x) * (k + 1)) + first_idx;
        end_idx = (k < blockDim.x - 1) ? end_idx : arr_end;

        int start_of_current_map = i * GRID_WIDTH * GRID_HEIGHT;

        for (int j = start_idx; j < end_idx; j++) {

            int x = particles_x[j];
            int y = particles_y[j];

            int curr_idx = start_of_current_map + (x * GRID_HEIGHT) + y;

            if (x >= 0 && y >= 0 && map_2d[curr_idx] == 0) {
                map_2d[curr_idx] = 1;
            }
        }
    }
}

__global__ void kernel_update_unique_restructure(uint8_t* map_2d, int* particles_x, int* particles_y, int* unique_in_particle_col,
    const int GRID_WIDTH, const int GRID_HEIGHT) {

    int i = threadIdx.x;

    int start_idx = (i * GRID_HEIGHT);
    int end_idx = ((i + 1) * GRID_HEIGHT);
    int key = unique_in_particle_col[i];
    int first_key = key;

    for (int j = start_idx; j < end_idx; j++) {

        if (map_2d[j] == 1) {

            int y = j % GRID_HEIGHT;
            int x = j / GRID_HEIGHT;

            particles_x[key] = x;
            particles_y[key] = y;
            key += 1;
        }
    }
}


